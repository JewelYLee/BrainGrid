/*
 * AllSpikingNeurons_d.cu
 *
 */

#include "AllSpikingNeurons.h"
#include "Book.h"

void AllSpikingNeurons::copyDeviceSpikeHistoryToHost( AllSpikingNeurons& allNeurons, const SimulationInfo *sim_info ) 
{
        int numNeurons = sim_info->totalNeurons;
        uint64_t* pSpikeHistory[numNeurons];
        HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeurons.spike_history, numNeurons * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );

        int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);
        for (int i = 0; i < numNeurons; i++) {
                HANDLE_ERROR( hipMemcpy ( spike_history[i], pSpikeHistory[i],
                        max_spikes * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        }
}


void AllSpikingNeurons::copyDeviceSpikeCountsToHost( AllSpikingNeurons& allNeurons, const SimulationInfo *sim_info ) 
{
        int numNeurons = sim_info->totalNeurons;

        HANDLE_ERROR( hipMemcpy ( spikeCount, allNeurons.spikeCount, numNeurons * sizeof( int ), hipMemcpyDeviceToHost ) );
}


void AllSpikingNeurons::clearDeviceSpikeCounts( AllSpikingNeurons& allNeurons, const SimulationInfo *sim_info ) 
{
        int numNeurons = sim_info->totalNeurons;

        HANDLE_ERROR( hipMemset( allNeurons.spikeCount, 0, numNeurons * sizeof( int ) ) );
}
