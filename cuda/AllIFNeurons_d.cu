/*
 * AllIFNeurons_d.cu
 *
 */

#include "AllIFNeurons.h"
#include "Book.h"

void AllIFNeurons::allocNeuronDeviceStruct( void** allNeuronsDevice, SimulationInfo *sim_info ) {
	AllIFNeurons allNeurons;

	allocDeviceStruct( allNeurons, sim_info );

        HANDLE_ERROR( hipMalloc( allNeuronsDevice, sizeof( AllIFNeurons ) ) );
        HANDLE_ERROR( hipMemcpy ( *allNeuronsDevice, &allNeurons, sizeof( AllIFNeurons ), hipMemcpyHostToDevice ) );
}

void AllIFNeurons::allocDeviceStruct( AllIFNeurons &allNeurons, SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;
	int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);
 
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.C1, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.C2, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Cm, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.I0, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Iinject, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Inoise, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Isyn, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Rm, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Tau, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Trefract, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vinit, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vm, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vreset, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vrest, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vthresh, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.hasFired, count * sizeof( bool ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.nStepsInRefr, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.neuron_type_map, count * sizeof( neuronType ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.spikeCount, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.starter_map, count * sizeof( bool ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.summation_map, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.spike_history, count * sizeof( uint64_t* ) ) );
	
	uint64_t* pSpikeHistory[count];
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR( hipMalloc( ( void ** ) &pSpikeHistory[i], max_spikes * sizeof( uint64_t ) ) );
	}
	HANDLE_ERROR( hipMemcpy ( allNeurons.spike_history, pSpikeHistory,
		count * sizeof( uint64_t* ), hipMemcpyHostToDevice ) );

	// get device summation point address and set it to sim info
	sim_info->pSummationMap = allNeurons.summation_map;
}

void AllIFNeurons::deleteNeuronDeviceStruct( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
	AllIFNeurons allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeurons ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allNeurons, sim_info );

	HANDLE_ERROR( hipFree( allNeuronsDevice ) );
}

void AllIFNeurons::deleteDeviceStruct( AllIFNeurons& allNeurons, const SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;

	uint64_t* pSpikeHistory[count];
	HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeurons.spike_history,
		count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR( hipFree( pSpikeHistory[i] ) );
	}

	HANDLE_ERROR( hipFree( allNeurons.C1 ) );
	HANDLE_ERROR( hipFree( allNeurons.C2 ) );
	HANDLE_ERROR( hipFree( allNeurons.Cm ) );
	HANDLE_ERROR( hipFree( allNeurons.I0 ) );
	HANDLE_ERROR( hipFree( allNeurons.Iinject ) );
	HANDLE_ERROR( hipFree( allNeurons.Inoise ) );
	HANDLE_ERROR( hipFree( allNeurons.Isyn ) );
	HANDLE_ERROR( hipFree( allNeurons.Rm ) );
	HANDLE_ERROR( hipFree( allNeurons.Tau ) );
	HANDLE_ERROR( hipFree( allNeurons.Trefract ) );
	HANDLE_ERROR( hipFree( allNeurons.Vinit ) );
	HANDLE_ERROR( hipFree( allNeurons.Vm ) );
	HANDLE_ERROR( hipFree( allNeurons.Vreset ) );
	HANDLE_ERROR( hipFree( allNeurons.Vrest ) );
	HANDLE_ERROR( hipFree( allNeurons.Vthresh ) );
	HANDLE_ERROR( hipFree( allNeurons.hasFired ) );
	HANDLE_ERROR( hipFree( allNeurons.nStepsInRefr ) );
	HANDLE_ERROR( hipFree( allNeurons.neuron_type_map) );
	HANDLE_ERROR( hipFree( allNeurons.spikeCount ) );
	HANDLE_ERROR( hipFree( allNeurons.starter_map ) );
	HANDLE_ERROR( hipFree( allNeurons.summation_map ) );
	HANDLE_ERROR( hipFree( allNeurons.spike_history ) );
}

void AllIFNeurons::copyNeuronHostToDevice( void* allNeuronsDevice, const SimulationInfo *sim_info ) { 
	AllIFNeurons allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeurons ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allNeurons, sim_info );
}

void AllIFNeurons::copyHostToDevice( AllIFNeurons& allNeurons, const SimulationInfo *sim_info ) { 
	int count = sim_info->totalNeurons;

	HANDLE_ERROR( hipMemcpy ( allNeurons.C1, C1, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.C2, C2, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Cm, Cm, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.I0, I0, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Iinject, Iinject, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Inoise, Inoise, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Isyn, Isyn, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Rm, Rm, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Tau, Tau, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Trefract, Trefract, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vinit, Vinit, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vm, Vm, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vreset, Vreset, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vrest, Vrest, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vthresh, Vthresh, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.hasFired, hasFired, count * sizeof( bool ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.nStepsInRefr, nStepsInRefr, count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.neuron_type_map, neuron_type_map, count * sizeof( neuronType ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.spikeCount, spikeCount, count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.starter_map, starter_map, count * sizeof( bool ), hipMemcpyHostToDevice ) );
	//HANDLE_ERROR( hipMemcpy ( allNeurons.spike_history, spike_history, count * sizeof( uint64_t* ), hipMemcpyHostToDevice ) );
}

void AllIFNeurons::copyNeuronDeviceToHost( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
	AllIFNeurons allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeurons ), hipMemcpyDeviceToHost ) );
	copyDeviceToHost( allNeurons, sim_info );
}

void AllIFNeurons::copyDeviceToHost( AllIFNeurons& allNeurons, const SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;

	HANDLE_ERROR( hipMemcpy ( C1, allNeurons.C1, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( C2, allNeurons.C2, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Cm, allNeurons.C1, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( I0, allNeurons.I0, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Iinject, allNeurons.Iinject, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Inoise, allNeurons.Inoise, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Isyn, allNeurons.Isyn, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Rm, allNeurons.Rm, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Tau, allNeurons.Tau, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Trefract, allNeurons.Trefract, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vinit, allNeurons.Vinit, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vm, allNeurons.Vm, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vreset, allNeurons.Vreset, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vrest, allNeurons.Vrest, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vthresh, allNeurons.Vthresh, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( hasFired, allNeurons.hasFired, count * sizeof( bool ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( nStepsInRefr, allNeurons.nStepsInRefr, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( neuron_type_map, allNeurons.neuron_type_map, count * sizeof( neuronType ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( spikeCount, allNeurons.spikeCount, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( starter_map, allNeurons.starter_map, count * sizeof( bool ), hipMemcpyDeviceToHost ) );
	//HANDLE_ERROR( hipMemcpy ( spike_history, allNeurons.spike_history, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
}
