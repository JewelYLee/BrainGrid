/*
 * AllIFNeurons_d.cu
 *
 */

#include "AllIFNeurons.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all neurons' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allNeuronsDevice   Reference to the allNeurons struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::allocNeuronDeviceStruct( void** allNeuronsDevice, SimulationInfo *sim_info ) {
	AllIFNeurons allNeurons;
   for(int i =0; i < sim_info->numGPU; i++){
      hipSetDevice(i);
      allocDeviceStruct( allNeurons, &sim_info->individualGPUInfo[i] );
      HANDLE_ERROR( hipMalloc( allNeuronsDevice, sizeof( AllIFNeurons ) ) );
      HANDLE_ERROR( hipMemcpy ( allNeuronsDevice[i], &allNeurons, sizeof( AllIFNeurons ), hipMemcpyHostToDevice ) );\
   }
}

/*
 *  Allocate GPU memories to store all neurons' states.
 *  (Helper function of allocNeuronDeviceStruct)
 *
 *  @param  allNeurons         Reference to the allIFNeurons struct.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::allocDeviceStruct( AllIFNeurons &allNeurons, SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;
	int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);
 
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.C1, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.C2, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Cm, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.I0, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Iinject, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Inoise, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Isyn, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Rm, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Tau, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Trefract, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vinit, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vm, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vreset, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vrest, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vthresh, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.hasFired, count * sizeof( bool ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.nStepsInRefr, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.spikeCount, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.spikeCountOffset, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.summation_map, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.spike_history, count * sizeof( uint64_t* ) ) );
	
	uint64_t* pSpikeHistory[count]; //make a "count" length array of uint64_t pointers
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR( hipMalloc( ( void ** ) &pSpikeHistory[i], max_spikes * sizeof( uint64_t ) ) );
	}
	HANDLE_ERROR( hipMemcpy ( allNeurons.spike_history, pSpikeHistory,
		count * sizeof( uint64_t* ), hipMemcpyHostToDevice ) );

	// get device summation point address and set it to sim info
	sim_info->pSummationMap = allNeurons.summation_map;
}

/*
 *  Delete GPU memories.
 *
 *  @param  allNeuronsDevice   Reference to the allNeurons struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::deleteNeuronDeviceStruct( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
	AllIFNeurons allNeurons;
   
   for(int i =0; i < sim_info->numGPU; i++){
      hipSetDevice(i);
	   HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeurons ), hipMemcpyDeviceToHost ) );
	   deleteDeviceStruct( allNeurons, sim_info );
      HANDLE_ERROR( hipFree( allNeuronsDevice ) );
   }
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteNeuronDeviceStruct)
 *
 *  @param  allNeurons         Reference to the allIFNeurons struct.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::deleteDeviceStruct( AllIFNeurons& allNeurons, const SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;

	uint64_t* pSpikeHistory[count];
	HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeurons.spike_history,
		count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR( hipFree( pSpikeHistory[i] ) );
	}

	HANDLE_ERROR( hipFree( allNeurons.C1 ) );
	HANDLE_ERROR( hipFree( allNeurons.C2 ) );
	HANDLE_ERROR( hipFree( allNeurons.Cm ) );
	HANDLE_ERROR( hipFree( allNeurons.I0 ) );
	HANDLE_ERROR( hipFree( allNeurons.Iinject ) );
	HANDLE_ERROR( hipFree( allNeurons.Inoise ) );
	HANDLE_ERROR( hipFree( allNeurons.Isyn ) );
	HANDLE_ERROR( hipFree( allNeurons.Rm ) );
	HANDLE_ERROR( hipFree( allNeurons.Tau ) );
	HANDLE_ERROR( hipFree( allNeurons.Trefract ) );
	HANDLE_ERROR( hipFree( allNeurons.Vinit ) );
	HANDLE_ERROR( hipFree( allNeurons.Vm ) );
	HANDLE_ERROR( hipFree( allNeurons.Vreset ) );
	HANDLE_ERROR( hipFree( allNeurons.Vrest ) );
	HANDLE_ERROR( hipFree( allNeurons.Vthresh ) );
	HANDLE_ERROR( hipFree( allNeurons.hasFired ) );
	HANDLE_ERROR( hipFree( allNeurons.nStepsInRefr ) );
	HANDLE_ERROR( hipFree( allNeurons.spikeCount ) );
	HANDLE_ERROR( hipFree( allNeurons.spikeCountOffset ) );
	HANDLE_ERROR( hipFree( allNeurons.summation_map ) );
	HANDLE_ERROR( hipFree( allNeurons.spike_history ) );
}

/*
 *  Copy all neurons' data from host to device.
 *
 *  @param  allNeuronsDevice   Reference to the allNeurons struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::copyNeuronHostToDevice( void** allNeuronsDevice, const SimulationInfo *sim_info ) { 
	AllIFNeurons allNeurons;
   int offsetFromFirstNeuron = 0;
   for(int i =0; i < sim_info->numGPU; i++){
      hipSetDevice(i);
      //copy the base addresses for all the arrays on the device
      HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice[i], sizeof( AllIFNeurons ), hipMemcpyDeviceToHost ) );
      
      copyHostToDevice( allNeurons, &sim_info->individualGPUInfo[i], offsetFromFirstNeuron );
      offsetFromFirstNeuron += sim_info->individualGPUInfo[i].totalNeurons;
   }
}

/*
 *  Copy all neurons' data from host to device.
 *  (Helper function of copyNeuronHostToDevice)
 *
 *  @param  allNeurons            Reference to the allIFNeurons struct.
 *  @param  sim_info              SimulationInfo to refer from.
 *  @param  offsetFromFirstNeuron 
 */
void AllIFNeurons::copyHostToDevice( AllIFNeurons& allNeurons, const SimulationInfo *sim_info, const int offsetFromFirstNeuron ) { 

   int count = sim_info->totalNeurons;

   HANDLE_ERROR( hipMemcpy ( allNeurons.C1, C1+offsetFromFirstNeuron, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.C2, C2+offsetFromFirstNeuron, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.Cm, Cm+offsetFromFirstNeuron, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.I0, I0+offsetFromFirstNeuron, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.Iinject, Iinject+offsetFromFirstNeuron, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.Inoise, Inoise+offsetFromFirstNeuron, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.Isyn, Isyn+offsetFromFirstNeuron, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.Rm, Rm+offsetFromFirstNeuron, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.Tau, Tau+offsetFromFirstNeuron, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.Trefract, Trefract+offsetFromFirstNeuron, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.Vinit, Vinit+offsetFromFirstNeuron, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.Vm, Vm+offsetFromFirstNeuron, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.Vreset, Vreset+offsetFromFirstNeuron, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.Vrest, Vrest+offsetFromFirstNeuron, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.Vthresh, Vthresh+offsetFromFirstNeuron, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.hasFired, hasFired+offsetFromFirstNeuron, count * sizeof( bool ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.nStepsInRefr, nStepsInRefr+offsetFromFirstNeuron, count * sizeof( int ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.spikeCount, spikeCount+offsetFromFirstNeuron, count * sizeof( int ), hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy ( allNeurons.spikeCountOffset, spikeCountOffset+offsetFromFirstNeuron, count * sizeof( int ), hipMemcpyHostToDevice ) );

   int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);
   uint64_t* pSpikeHistory[count];
   HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeurons.spike_history, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
   for (int i = 0; i < count; i++) {
          HANDLE_ERROR( hipMemcpy ( pSpikeHistory[i], spike_history[i + offsetFromFirstNeuron], max_spikes * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
   }
}

/*
 *  Copy all neurons' data from device to host.
 *
 *  @param  allNeuronsDevice   Reference to the allNeurons struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::copyNeuronDeviceToHost( void** allNeuronsDevice, const SimulationInfo *sim_info ) {
	AllIFNeurons allNeurons;

   int offsetFromFirstNeuron = 0;
   for(int i =0; i < sim_info->numGPU; i++){
      hipSetDevice(i);
      //get pointers to arrays on device
      HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice[i], sizeof( AllIFNeurons ), hipMemcpyDeviceToHost ) );
      copyDeviceToHost( allNeurons, &sim_info->individualGPUInfo[i], offsetFromFirstNeuron );
      offsetFromFirstNeuron += sim_info->individualGPUInfo[i].totalNeurons;
   }
}

/*
 *  Copy all neurons' data from device to host.
 *  (Helper function of copyNeuronDeviceToHost)
 *
 *  @param  allNeurons         Reference to the allIFNeurons struct.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::copyDeviceToHost( AllIFNeurons& allNeurons, const SimulationInfo *sim_info, const int offsetFromFirstNeuron ) {
	int count = sim_info->totalNeurons;

	HANDLE_ERROR( hipMemcpy ( C1+offsetFromFirstNeuron, allNeurons.C1, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( C2+offsetFromFirstNeuron, allNeurons.C2, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Cm+offsetFromFirstNeuron, allNeurons.C1, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( I0+offsetFromFirstNeuron, allNeurons.I0, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Iinject+offsetFromFirstNeuron, allNeurons.Iinject, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Inoise+offsetFromFirstNeuron, allNeurons.Inoise, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Isyn+offsetFromFirstNeuron, allNeurons.Isyn, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Rm+offsetFromFirstNeuron, allNeurons.Rm, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Tau+offsetFromFirstNeuron, allNeurons.Tau, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Trefract+offsetFromFirstNeuron, allNeurons.Trefract, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vinit+offsetFromFirstNeuron, allNeurons.Vinit, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vm+offsetFromFirstNeuron, allNeurons.Vm, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vreset+offsetFromFirstNeuron, allNeurons.Vreset, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vrest+offsetFromFirstNeuron, allNeurons.Vrest, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vthresh+offsetFromFirstNeuron, allNeurons.Vthresh, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( hasFired+offsetFromFirstNeuron, allNeurons.hasFired, count * sizeof( bool ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( nStepsInRefr+offsetFromFirstNeuron, allNeurons.nStepsInRefr, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( spikeCount+offsetFromFirstNeuron, allNeurons.spikeCount, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( spikeCountOffset+offsetFromFirstNeuron, allNeurons.spikeCountOffset, count * sizeof( int ), hipMemcpyDeviceToHost ) );

   int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);
   uint64_t* pSpikeHistory[count];
   HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeurons.spike_history, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
   for (int i = 0; i < count; i++) {
      HANDLE_ERROR( hipMemcpy ( spike_history[i+offsetFromFirstNeuron], pSpikeHistory[i], max_spikes * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
   }
}

/*
 *  Copy spike history data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the allNeurons struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::copyNeuronDeviceSpikeHistoryToHost( void* allNeuronsDevice, const SimulationInfo *sim_info ) 
{        
        AllIFNeurons allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeurons ), hipMemcpyDeviceToHost ) );        
        AllSpikingNeurons::copyDeviceSpikeHistoryToHost( allNeurons, sim_info );
}

/*
 *  Copy spike counts data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the allNeurons struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::copyNeuronDeviceSpikeCountsToHost( void* allNeuronsDevice, const SimulationInfo *sim_info )
{
        AllIFNeurons allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeurons ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeCountsToHost( allNeurons, sim_info );
}

/*
 *  Clear the spike counts out of all neurons.
 *
 *  @param  allNeuronsDevice   Reference to the allNeurons struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::clearNeuronSpikeCounts( void* allNeuronsDevice, const SimulationInfo *sim_info )
{
        AllIFNeurons allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeurons ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::clearDeviceSpikeCounts( allNeurons, sim_info );
}

/*
 *  Update the state of all neurons for a time step
 *  Notify outgoing synapses if neuron has fired.
 *
 *  @param  synapses               Reference to the allSynapses struct on host memory.
 *  @param  allNeuronsDevice       Reference to the allNeurons struct on device memory.
 *  @param  allSynapsesDevice      Reference to the allSynapses struct on device memory.
 *  @param  sim_info               SimulationInfo to refer from.
 *  @param  randNoise              Reference to the random noise array.
 *  @param  synapseIndexMapDevice  Reference to the SynapseIndexMap on device memory.
 */
void AllIFNeurons::advanceNeurons( IAllSynapses &synapses, IAllNeurons** allNeuronsDevice, IAllSynapses** allSynapsesDevice, const SimulationInfo *sim_info, float* randNoise, SynapseIndexMap** synapseIndexMapDevice )
{
}
