/*
 * LifNeuron_struct_d.cu
 *
 */

#include "LIFGPUModel.h"

void LIFGPUModel::allocNeuronDeviceStruct( int count, int max_spikes ) {
	AllNeurons allNeurons;

	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.C1, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.C2, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Cm, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.I0, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Iinject, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Inoise, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Isyn, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Rm, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Tau, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Trefract, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vinit, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vm, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vreset, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vrest, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vthresh, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.hasFired, count * sizeof( bool ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.nStepsInRefr, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.neuron_type_map, count * sizeof( neuronType ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.spikeCount, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.starter_map, count * sizeof( bool ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.summation_map, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.spike_history, count * sizeof( uint64_t* ) ) );
	
	uint64_t* pSpikeHistory[count];
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR( hipMalloc( ( void ** ) &pSpikeHistory[i], max_spikes * sizeof( uint64_t ) ) );
	}
	HANDLE_ERROR( hipMemcpy ( allNeurons.spike_history, pSpikeHistory,
		count * sizeof( uint64_t* ), hipMemcpyHostToDevice ) );

	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeuronsDevice, sizeof( AllNeurons ) ) );
	HANDLE_ERROR( hipMemcpy( allNeuronsDevice, &allNeurons, sizeof( AllNeurons ), hipMemcpyHostToDevice ) );
}

void LIFGPUModel::deleteNeuronDeviceStruct( int count ) {
	AllNeurons allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllNeurons ), hipMemcpyDeviceToHost ) );

	uint64_t* pSpikeHistory[count];
	HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeurons.spike_history,
		count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR( hipFree( pSpikeHistory[i] ) );
	}

	HANDLE_ERROR( hipFree( allNeurons.C1 ) );
	HANDLE_ERROR( hipFree( allNeurons.C2 ) );
	HANDLE_ERROR( hipFree( allNeurons.Cm ) );
	HANDLE_ERROR( hipFree( allNeurons.I0 ) );
	HANDLE_ERROR( hipFree( allNeurons.Iinject ) );
	HANDLE_ERROR( hipFree( allNeurons.Inoise ) );
	HANDLE_ERROR( hipFree( allNeurons.Isyn ) );
	HANDLE_ERROR( hipFree( allNeurons.Rm ) );
	HANDLE_ERROR( hipFree( allNeurons.Tau ) );
	HANDLE_ERROR( hipFree( allNeurons.Trefract ) );
	HANDLE_ERROR( hipFree( allNeurons.Vinit ) );
	HANDLE_ERROR( hipFree( allNeurons.Vm ) );
	HANDLE_ERROR( hipFree( allNeurons.Vreset ) );
	HANDLE_ERROR( hipFree( allNeurons.Vrest ) );
	HANDLE_ERROR( hipFree( allNeurons.Vthresh ) );
	HANDLE_ERROR( hipFree( allNeurons.hasFired ) );
	HANDLE_ERROR( hipFree( allNeurons.nStepsInRefr ) );
	HANDLE_ERROR( hipFree( allNeurons.neuron_type_map) );
	HANDLE_ERROR( hipFree( allNeurons.spikeCount ) );
	HANDLE_ERROR( hipFree( allNeurons.starter_map ) );
	HANDLE_ERROR( hipFree( allNeurons.summation_map ) );
	HANDLE_ERROR( hipFree( allNeurons.spike_history ) );

	HANDLE_ERROR( hipFree( allNeuronsDevice ) );
}

void LIFGPUModel::copyNeuronHostToDevice( const AllNeurons& allNeuronsHost, int count ) { 
	AllNeurons allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllNeurons ), hipMemcpyDeviceToHost ) );

	HANDLE_ERROR( hipMemcpy ( allNeurons.C1, allNeuronsHost.C1, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.C2, allNeuronsHost.C2, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Cm, allNeuronsHost.Cm, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.I0, allNeuronsHost.I0, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Iinject, allNeuronsHost.Iinject, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Inoise, allNeuronsHost.Inoise, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Isyn, allNeuronsHost.Isyn, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Rm, allNeuronsHost.Rm, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Tau, allNeuronsHost.Tau, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Trefract, allNeuronsHost.Trefract, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vinit, allNeuronsHost.Vinit, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vm, allNeuronsHost.Vm, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vreset, allNeuronsHost.Vreset, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vrest, allNeuronsHost.Vrest, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vthresh, allNeuronsHost.Vthresh, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.hasFired, allNeuronsHost.hasFired, count * sizeof( bool ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.nStepsInRefr, allNeuronsHost.nStepsInRefr, count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.neuron_type_map, allNeuronsHost.neuron_type_map, count * sizeof( neuronType ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.spikeCount, allNeuronsHost.spikeCount, count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.starter_map, allNeuronsHost.starter_map, count * sizeof( bool ), hipMemcpyHostToDevice ) );
	//HANDLE_ERROR( hipMemcpy ( allNeurons.spike_history, allNeuronsHost.spike_history, count * sizeof( uint64_t* ), hipMemcpyHostToDevice ) );
}

void LIFGPUModel::copyNeuronDeviceToHost( AllNeurons& allNeuronsHost, int count ) {
	AllNeurons allNeurons;
	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllNeurons ), hipMemcpyDeviceToHost ) );

	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.C1, allNeurons.C1, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.C2, allNeurons.C2, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.Cm, allNeurons.C1, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.I0, allNeurons.I0, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.Iinject, allNeurons.Iinject, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.Inoise, allNeurons.Inoise, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.Isyn, allNeurons.Isyn, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.Rm, allNeurons.Rm, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.Tau, allNeurons.Tau, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.Trefract, allNeurons.Trefract, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.Vinit, allNeurons.Vinit, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.Vm, allNeurons.Vm, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.Vreset, allNeurons.Vreset, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.Vrest, allNeurons.Vrest, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.Vthresh, allNeurons.Vthresh, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.hasFired, allNeurons.hasFired, count * sizeof( bool ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.nStepsInRefr, allNeurons.nStepsInRefr, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.neuron_type_map, allNeurons.neuron_type_map, count * sizeof( neuronType ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.spikeCount, allNeurons.spikeCount, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allNeuronsHost.starter_map, allNeurons.starter_map, count * sizeof( bool ), hipMemcpyDeviceToHost ) );
	//HANDLE_ERROR( hipMemcpy ( allNeuronsHost.spike_history, allNeurons.spike_history, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
}
