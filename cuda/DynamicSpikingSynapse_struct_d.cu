/*
 * DynamicSpikingSynapse_struct_d.cu
 *
 */

#include "LIFGPUModel.h"

void LIFGPUModel::allocSynapseDeviceStruct( int num_neurons, int max_synapses ) {
	AllSynapses allSynapses_0;
	AllSynapses allSynapses_1(num_neurons, 0);

	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.summationCoord, num_neurons * sizeof( Coordinate* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.W, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.summationPoint, num_neurons * sizeof( BGFLOAT** ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.synapseCoord, num_neurons * sizeof( Coordinate* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.psr, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.decay, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.total_delay, num_neurons * sizeof( int* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.delayQueue, num_neurons * sizeof( uint32_t** ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.delayIdx, num_neurons * sizeof( int* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.ldelayQueue, num_neurons * sizeof( int* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.type, num_neurons * sizeof( synapseType* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.tau, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.r, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.u, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.D, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.U, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.F, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.lastSpike, num_neurons * sizeof( uint64_t* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.in_use, num_neurons * sizeof( bool* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.synapse_counts, num_neurons * sizeof( size_t ) ) );

	for (int i = 0; i < num_neurons; i++) {
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.summationCoord[i], max_synapses * sizeof( Coordinate ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.W[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.summationPoint[i], max_synapses * sizeof( BGFLOAT* ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.synapseCoord[i], max_synapses * sizeof( Coordinate ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.psr[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.decay[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.total_delay[i], max_synapses * sizeof( int ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.delayQueue[i], max_synapses * sizeof( uint32_t* ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.delayIdx[i], max_synapses * sizeof( int ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.ldelayQueue[i], max_synapses * sizeof( int ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.type[i], max_synapses * sizeof( synapseType ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.tau[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.r[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.u[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.D[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.U[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.F[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.lastSpike[i], max_synapses * sizeof( uint64_t ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.in_use[i], max_synapses * sizeof( bool ) ) );

		uint32_t* pDelayQueue[max_synapses];
		for (int j = 0; j < max_synapses; j++) {
			HANDLE_ERROR( hipMalloc( ( void ** ) &pDelayQueue[j], sizeof( uint32_t ) ) );
		}
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayQueue[i], pDelayQueue, 
			max_synapses * sizeof( uint32_t ), hipMemcpyHostToDevice ) );
	}
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.summationCoord, allSynapses_1.summationCoord, 
		num_neurons * sizeof( Coordinate* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.W, allSynapses_1.W, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) ); 
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.summationPoint, allSynapses_1.summationPoint, 
		num_neurons * sizeof( BGFLOAT** ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.synapseCoord, allSynapses_1.synapseCoord, 
		num_neurons * sizeof( Coordinate* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.psr, allSynapses_1.psr, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.decay, allSynapses_1.decay, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.total_delay, allSynapses_1.total_delay, 
		num_neurons * sizeof( int* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.delayQueue, allSynapses_1.delayQueue, 
		num_neurons * sizeof( uint32_t** ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.delayIdx, allSynapses_1.delayIdx, 
		num_neurons * sizeof( int* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.ldelayQueue, allSynapses_1.ldelayQueue, 
		num_neurons * sizeof( int* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.type, allSynapses_1.type, 
		num_neurons * sizeof( synapseType* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.tau, allSynapses_1.tau, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.r, allSynapses_1.r, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.u, allSynapses_1.u, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.D, allSynapses_1.D, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.U, allSynapses_1.U, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.F, allSynapses_1.F, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.lastSpike, allSynapses_1.lastSpike, 
		num_neurons * sizeof( uint64_t* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.in_use, allSynapses_1.in_use, 
		num_neurons * sizeof( bool* ), hipMemcpyHostToDevice ) );

	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice, sizeof( AllSynapses ) ) );
	HANDLE_ERROR( hipMemcpy ( allSynapsesDevice, &allSynapses_0, sizeof( AllSynapses ), hipMemcpyHostToDevice ) );
}

void LIFGPUModel::deleteSynapseDeviceStruct( int num_neurons, int max_synapses ) {
	AllSynapses allSynapses_0;
	AllSynapses allSynapses_1(num_neurons, 0);

	HANDLE_ERROR( hipMemcpy ( &allSynapses_0, allSynapsesDevice, sizeof( AllSynapses ), hipMemcpyDeviceToHost ) );

	HANDLE_ERROR( hipMemcpy ( allSynapses_1.summationCoord, allSynapses_0.summationCoord, 
		num_neurons * sizeof( Coordinate* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.W, allSynapses_0.W, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) ); 
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.summationPoint, allSynapses_0.summationPoint, 
		num_neurons * sizeof( BGFLOAT** ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.synapseCoord, allSynapses_0.synapseCoord, 
		num_neurons * sizeof( Coordinate* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.psr, allSynapses_0.psr, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.decay, allSynapses_0.decay, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.total_delay, allSynapses_0.total_delay, 
		num_neurons * sizeof( int* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayQueue, allSynapses_0.delayQueue, 
		num_neurons * sizeof( uint32_t** ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayIdx, allSynapses_0.delayIdx, 
		num_neurons * sizeof( int* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.ldelayQueue, allSynapses_0.ldelayQueue, 
		num_neurons * sizeof( int* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.type, allSynapses_0.type, 
		num_neurons * sizeof( synapseType* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.tau, allSynapses_0.tau, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.r, allSynapses_0.r, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.u, allSynapses_0.u, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.D, allSynapses_0.D, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.U, allSynapses_0.U, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.F, allSynapses_0.F, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.lastSpike, allSynapses_0.lastSpike, 
		num_neurons * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.in_use, allSynapses_0.in_use, 
		num_neurons * sizeof( bool* ), hipMemcpyDeviceToHost ) );

	for (int i = 0; i < num_neurons; i++) {
		uint32_t* pDelayQueue[max_synapses];
		HANDLE_ERROR( hipMemcpy ( pDelayQueue, allSynapses_1.delayQueue[i],
			max_synapses * sizeof( uint32_t* ), hipMemcpyDeviceToHost ) );
		for (int j = 0; j < max_synapses; j++) {
			HANDLE_ERROR( hipFree( pDelayQueue[j] ) );
		}

		HANDLE_ERROR( hipFree( allSynapses_1.summationCoord[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.W[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.summationPoint[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.synapseCoord[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.psr[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.decay[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.total_delay[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.delayQueue[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.delayIdx[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.ldelayQueue[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.type[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.tau[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.r[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.u[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.D[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.U[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.F[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.lastSpike[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.in_use[i] ) );
	}

	HANDLE_ERROR( hipFree( allSynapses_0.summationCoord ) );
	HANDLE_ERROR( hipFree( allSynapses_0.W ) );
	HANDLE_ERROR( hipFree( allSynapses_0.summationPoint ) );
	HANDLE_ERROR( hipFree( allSynapses_0.synapseCoord ) );
	HANDLE_ERROR( hipFree( allSynapses_0.psr ) );
	HANDLE_ERROR( hipFree( allSynapses_0.decay ) );
	HANDLE_ERROR( hipFree( allSynapses_0.total_delay ) );
	HANDLE_ERROR( hipFree( allSynapses_0.delayQueue ) );
	HANDLE_ERROR( hipFree( allSynapses_0.delayIdx ) );
	HANDLE_ERROR( hipFree( allSynapses_0.ldelayQueue ) );
	HANDLE_ERROR( hipFree( allSynapses_0.type ) );
	HANDLE_ERROR( hipFree( allSynapses_0.tau ) );
	HANDLE_ERROR( hipFree( allSynapses_0.r ) );
	HANDLE_ERROR( hipFree( allSynapses_0.u ) );
	HANDLE_ERROR( hipFree( allSynapses_0.D ) );
	HANDLE_ERROR( hipFree( allSynapses_0.U ) );
	HANDLE_ERROR( hipFree( allSynapses_0.F ) );
	HANDLE_ERROR( hipFree( allSynapses_0.lastSpike ) );
	HANDLE_ERROR( hipFree( allSynapses_0.in_use ) );
	HANDLE_ERROR( hipFree( allSynapses_0.synapse_counts ) );

	HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

void LIFGPUModel::copySynapseHostToDevice( const AllSynapses& allSynapsesHost, int num_neurons, int max_synapses ) { // copy everything necessary
	AllSynapses allSynapses_0;
	AllSynapses allSynapses_1(num_neurons, 0);

        HANDLE_ERROR( hipMemcpy ( &allSynapses_0, allSynapsesDevice, sizeof( AllSynapses ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.synapse_counts, allSynapsesHost.synapse_counts, 
			num_neurons * sizeof( size_t ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( &allSynapsesDevice->max_synapses, &allSynapsesHost.max_synapses, 
			sizeof( size_t ), hipMemcpyHostToDevice ) );

	HANDLE_ERROR( hipMemcpy ( allSynapses_1.summationCoord, allSynapses_0.summationCoord, 
		num_neurons * sizeof( Coordinate* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.W, allSynapses_0.W,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	//HANDLE_ERROR( hipMemcpy ( allSynapses_1.summationPoint, allSynapses_0.summationPoint,
	//	num_neurons * sizeof( BGFLOAT** ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.synapseCoord, allSynapses_0.synapseCoord,
		num_neurons * sizeof( Coordinate* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.psr, allSynapses_0.psr,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.decay, allSynapses_0.decay,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.total_delay, allSynapses_0.total_delay,
		num_neurons * sizeof( int* ), hipMemcpyDeviceToHost ) );
	//HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayQueue, allSynapses_0.delayQueue,
	//	num_neurons * sizeof( uint32_t** ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayIdx, allSynapses_0.delayIdx,
		num_neurons * sizeof( int* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.ldelayQueue, allSynapses_0.ldelayQueue,
		num_neurons * sizeof( int* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.type, allSynapses_0.type,
		num_neurons * sizeof( synapseType* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.tau, allSynapses_0.tau,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.r, allSynapses_0.r,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.u, allSynapses_0.u,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.D, allSynapses_0.D,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.U, allSynapses_0.U,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.F, allSynapses_0.F,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.lastSpike, allSynapses_0.lastSpike,
		num_neurons * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.in_use, allSynapses_0.in_use,
		num_neurons * sizeof( bool* ), hipMemcpyDeviceToHost ) );

	for (int i = 0; i < num_neurons; i++) {
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.summationCoord[i], allSynapsesHost.summationCoord[i], 
			max_synapses * sizeof( Coordinate ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.W[i], allSynapsesHost.W[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		//HANDLE_ERROR( hipMemcpy ( allSynapses_1.summationPoint[i], allSynapsesHost.summationPoint[i],
		//	max_synapses * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.synapseCoord[i], allSynapsesHost.synapseCoord[i],
			max_synapses * sizeof( Coordinate ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.psr[i], allSynapsesHost.psr[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.decay[i], allSynapsesHost.decay[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.total_delay[i], allSynapsesHost.total_delay[i],
			max_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
		//HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayQueue[i], allSynapsesHost.delayQueue[i],
		//	max_synapses * sizeof( uint32_t* ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayIdx[i], allSynapsesHost.delayIdx[i],
			max_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.ldelayQueue[i], allSynapsesHost.ldelayQueue[i],
			max_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.type[i], allSynapsesHost.type[i],
			max_synapses * sizeof( synapseType ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.tau[i], allSynapsesHost.tau[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.r[i], allSynapsesHost.r[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.u[i], allSynapsesHost.u[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.D[i], allSynapsesHost.D[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.U[i], allSynapsesHost.U[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.F[i], allSynapsesHost.F[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.lastSpike[i], allSynapsesHost.lastSpike[i],
			max_synapses * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.in_use[i], allSynapsesHost.in_use[i],
			max_synapses * sizeof( bool ), hipMemcpyHostToDevice ) );
	}
}

void LIFGPUModel::copySynapseDeviceToHost( AllSynapses& allSynapsesHost, int num_neurons, int max_synapses ) {
	// copy everything necessary
	AllSynapses allSynapses;

#if 0
        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSynapses ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapsesHost.synapse_counts, allSynapses.synapse_counts, 
		num_neurons * sizeof( size_t ), hipMemcpyDeviceToHost ) );

	for (int i = 0; i < num_neurons; i++) {
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.summationCoord[i], allSynapses.summationCoord[i],
			max_synapses * sizeof( Coordinate ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.W[i], allSynapses.W[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		//HANDLE_ERROR( hipMemcpy ( allSynapsesHost.summationPoint[i], allSynapses.summationPoint[i],
		//	max_synapses * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.synapseCoord[i], allSynapses.synapseCoord[i],
			max_synapses * sizeof( Coordinate ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.psr[i], allSynapses.psr[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.decay[i], allSynapses.decay[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.total_delay[i], allSynapses.total_delay[i],
			max_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
		//HANDLE_ERROR( hipMemcpy ( allSynapsesHost.delayQueue[i], allSynapses.delayQueue[i],
		//	max_synapses * sizeof( uint32_t* ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.delayIdx[i], allSynapses.delayIdx[i],
			max_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.ldelayQueue[i], allSynapses.ldelayQueue[i],
			max_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.type[i], allSynapses.type[i],
			max_synapses * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.tau[i], allSynapses.tau[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.r[i], allSynapses.r[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.u[i], allSynapses.u[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.D[i], allSynapses.D[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.U[i], allSynapses.U[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.F[i], allSynapses.F[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.lastSpike[i], allSynapses.lastSpike[i],
			max_synapses * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.in_use[i], allSynapses.in_use[i],
			max_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
	}
#endif
}

#if 0
void copySynapseSumCoordDeviceToHost( DynamicSpikingSynapse_struct& synapse_h, int count ) {
	// copy everything necessary
	DynamicSpikingSynapse_struct synapse;

	if ( count > 0 ) {
        	HANDLE_ERROR( hipMemcpyFromSymbol( &synapse, HIP_SYMBOL(synapse_st_d), sizeof( DynamicSpikingSynapse_struct ) ) );

		HANDLE_ERROR( hipMemcpy ( synapse_h.in_use, synapse.in_use, count * sizeof( bool ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( synapse_h.summationCoord, synapse.summationCoord, count * sizeof( Coordinate ), hipMemcpyDeviceToHost ) );
	}
}
#endif
