#include "hip/hip_runtime.h"
/*
 * DynamicSpikingSynapse_struct_d.cu
 *
 */

#include "LIFGPUModel.h"

__global__ void setSynapseSummationPointDevice(int num_neurons, AllNeurons* allNeuronsDevice, AllSynapses* allSynapsesDevice, int width);

void LIFGPUModel::allocSynapseDeviceStruct( int num_neurons, int max_synapses ) {
	AllSynapses allSynapses_0;
	AllSynapses allSynapses_1(num_neurons, 0);

	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.summationCoord, num_neurons * sizeof( Coordinate* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.W, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.summationPoint, num_neurons * sizeof( BGFLOAT** ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.synapseCoord, num_neurons * sizeof( Coordinate* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.psr, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.decay, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.total_delay, num_neurons * sizeof( int* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.delayQueue, num_neurons * sizeof( uint32_t** ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.delayIdx, num_neurons * sizeof( int* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.ldelayQueue, num_neurons * sizeof( int* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.type, num_neurons * sizeof( synapseType* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.tau, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.r, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.u, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.D, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.U, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.F, num_neurons * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.lastSpike, num_neurons * sizeof( uint64_t* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.in_use, num_neurons * sizeof( bool* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_0.synapse_counts, num_neurons * sizeof( size_t ) ) );

	for (int i = 0; i < num_neurons; i++) {
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.summationCoord[i], max_synapses * sizeof( Coordinate ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.W[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.summationPoint[i], max_synapses * sizeof( BGFLOAT* ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.synapseCoord[i], max_synapses * sizeof( Coordinate ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.psr[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.decay[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.total_delay[i], max_synapses * sizeof( int ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.delayQueue[i], max_synapses * sizeof( uint32_t* ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.delayIdx[i], max_synapses * sizeof( int ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.ldelayQueue[i], max_synapses * sizeof( int ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.type[i], max_synapses * sizeof( synapseType ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.tau[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.r[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.u[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.D[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.U[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.F[i], max_synapses * sizeof( BGFLOAT ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.lastSpike[i], max_synapses * sizeof( uint64_t ) ) );
		HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses_1.in_use[i], max_synapses * sizeof( bool ) ) );

		uint32_t* pDelayQueue[max_synapses];
		for (int j = 0; j < max_synapses; j++) {
			HANDLE_ERROR( hipMalloc( ( void ** ) &pDelayQueue[j], sizeof( uint32_t ) ) );
		}
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayQueue[i], pDelayQueue, 
			max_synapses * sizeof( uint32_t* ), hipMemcpyHostToDevice ) );
	}
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.summationCoord, allSynapses_1.summationCoord, 
		num_neurons * sizeof( Coordinate* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.W, allSynapses_1.W, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) ); 
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.summationPoint, allSynapses_1.summationPoint, 
		num_neurons * sizeof( BGFLOAT** ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.synapseCoord, allSynapses_1.synapseCoord, 
		num_neurons * sizeof( Coordinate* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.psr, allSynapses_1.psr, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.decay, allSynapses_1.decay, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.total_delay, allSynapses_1.total_delay, 
		num_neurons * sizeof( int* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.delayQueue, allSynapses_1.delayQueue, 
		num_neurons * sizeof( uint32_t** ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.delayIdx, allSynapses_1.delayIdx, 
		num_neurons * sizeof( int* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.ldelayQueue, allSynapses_1.ldelayQueue, 
		num_neurons * sizeof( int* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.type, allSynapses_1.type, 
		num_neurons * sizeof( synapseType* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.tau, allSynapses_1.tau, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.r, allSynapses_1.r, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.u, allSynapses_1.u, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.D, allSynapses_1.D, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.U, allSynapses_1.U, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.F, allSynapses_1.F, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.lastSpike, allSynapses_1.lastSpike, 
		num_neurons * sizeof( uint64_t* ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.in_use, allSynapses_1.in_use, 
		num_neurons * sizeof( bool* ), hipMemcpyHostToDevice ) );

	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice, sizeof( AllSynapses ) ) );
	HANDLE_ERROR( hipMemcpy ( allSynapsesDevice, &allSynapses_0, sizeof( AllSynapses ), hipMemcpyHostToDevice ) );
}

void LIFGPUModel::deleteSynapseDeviceStruct( int num_neurons, int max_synapses ) {
	AllSynapses allSynapses_0;
	AllSynapses allSynapses_1(num_neurons, 0);

	HANDLE_ERROR( hipMemcpy ( &allSynapses_0, allSynapsesDevice, sizeof( AllSynapses ), hipMemcpyDeviceToHost ) );

	HANDLE_ERROR( hipMemcpy ( allSynapses_1.summationCoord, allSynapses_0.summationCoord, 
		num_neurons * sizeof( Coordinate* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.W, allSynapses_0.W, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) ); 
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.summationPoint, allSynapses_0.summationPoint, 
		num_neurons * sizeof( BGFLOAT** ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.synapseCoord, allSynapses_0.synapseCoord, 
		num_neurons * sizeof( Coordinate* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.psr, allSynapses_0.psr, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.decay, allSynapses_0.decay, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.total_delay, allSynapses_0.total_delay, 
		num_neurons * sizeof( int* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayQueue, allSynapses_0.delayQueue, 
		num_neurons * sizeof( uint32_t** ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayIdx, allSynapses_0.delayIdx, 
		num_neurons * sizeof( int* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.ldelayQueue, allSynapses_0.ldelayQueue, 
		num_neurons * sizeof( int* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.type, allSynapses_0.type, 
		num_neurons * sizeof( synapseType* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.tau, allSynapses_0.tau, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.r, allSynapses_0.r, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.u, allSynapses_0.u, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.D, allSynapses_0.D, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.U, allSynapses_0.U, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.F, allSynapses_0.F, 
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.lastSpike, allSynapses_0.lastSpike, 
		num_neurons * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.in_use, allSynapses_0.in_use, 
		num_neurons * sizeof( bool* ), hipMemcpyDeviceToHost ) );

	for (int i = 0; i < num_neurons; i++) {
		uint32_t* pDelayQueue[max_synapses];
		HANDLE_ERROR( hipMemcpy ( pDelayQueue, allSynapses_1.delayQueue[i],
			max_synapses * sizeof( uint32_t* ), hipMemcpyDeviceToHost ) );
		for (int j = 0; j < max_synapses; j++) {
			HANDLE_ERROR( hipFree( pDelayQueue[j] ) );
		}

		HANDLE_ERROR( hipFree( allSynapses_1.summationCoord[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.W[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.summationPoint[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.synapseCoord[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.psr[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.decay[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.total_delay[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.delayQueue[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.delayIdx[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.ldelayQueue[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.type[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.tau[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.r[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.u[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.D[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.U[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.F[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.lastSpike[i] ) );
		HANDLE_ERROR( hipFree( allSynapses_1.in_use[i] ) );
	}

	HANDLE_ERROR( hipFree( allSynapses_0.summationCoord ) );
	HANDLE_ERROR( hipFree( allSynapses_0.W ) );
	HANDLE_ERROR( hipFree( allSynapses_0.summationPoint ) );
	HANDLE_ERROR( hipFree( allSynapses_0.synapseCoord ) );
	HANDLE_ERROR( hipFree( allSynapses_0.psr ) );
	HANDLE_ERROR( hipFree( allSynapses_0.decay ) );
	HANDLE_ERROR( hipFree( allSynapses_0.total_delay ) );
	HANDLE_ERROR( hipFree( allSynapses_0.delayQueue ) );
	HANDLE_ERROR( hipFree( allSynapses_0.delayIdx ) );
	HANDLE_ERROR( hipFree( allSynapses_0.ldelayQueue ) );
	HANDLE_ERROR( hipFree( allSynapses_0.type ) );
	HANDLE_ERROR( hipFree( allSynapses_0.tau ) );
	HANDLE_ERROR( hipFree( allSynapses_0.r ) );
	HANDLE_ERROR( hipFree( allSynapses_0.u ) );
	HANDLE_ERROR( hipFree( allSynapses_0.D ) );
	HANDLE_ERROR( hipFree( allSynapses_0.U ) );
	HANDLE_ERROR( hipFree( allSynapses_0.F ) );
	HANDLE_ERROR( hipFree( allSynapses_0.lastSpike ) );
	HANDLE_ERROR( hipFree( allSynapses_0.in_use ) );
	HANDLE_ERROR( hipFree( allSynapses_0.synapse_counts ) );

	HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

void LIFGPUModel::copySynapseHostToDevice( const AllSynapses& allSynapsesHost, const SimulationInfo *sim_info ) { // copy everything necessary
	int num_neurons = sim_info->totalNeurons;
	int max_synapses = sim_info->maxSynapsesPerNeuron;
	AllSynapses allSynapses_0;
	AllSynapses allSynapses_1(num_neurons, 0);

        HANDLE_ERROR( hipMemcpy ( &allSynapses_0, allSynapsesDevice, sizeof( AllSynapses ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_0.synapse_counts, allSynapsesHost.synapse_counts, 
			num_neurons * sizeof( size_t ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( &allSynapsesDevice->max_synapses, &allSynapsesHost.max_synapses, 
			sizeof( size_t ), hipMemcpyHostToDevice ) );

	HANDLE_ERROR( hipMemcpy ( allSynapses_1.summationCoord, allSynapses_0.summationCoord, 
		num_neurons * sizeof( Coordinate* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.W, allSynapses_0.W,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	//HANDLE_ERROR( hipMemcpy ( allSynapses_1.summationPoint, allSynapses_0.summationPoint,
	//	num_neurons * sizeof( BGFLOAT** ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.synapseCoord, allSynapses_0.synapseCoord,
		num_neurons * sizeof( Coordinate* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.psr, allSynapses_0.psr,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.decay, allSynapses_0.decay,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.total_delay, allSynapses_0.total_delay,
		num_neurons * sizeof( int* ), hipMemcpyDeviceToHost ) );
	//HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayQueue, allSynapses_0.delayQueue,
	//	num_neurons * sizeof( uint32_t** ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayIdx, allSynapses_0.delayIdx,
		num_neurons * sizeof( int* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.ldelayQueue, allSynapses_0.ldelayQueue,
		num_neurons * sizeof( int* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.type, allSynapses_0.type,
		num_neurons * sizeof( synapseType* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.tau, allSynapses_0.tau,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.r, allSynapses_0.r,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.u, allSynapses_0.u,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.D, allSynapses_0.D,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.U, allSynapses_0.U,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.F, allSynapses_0.F,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.lastSpike, allSynapses_0.lastSpike,
		num_neurons * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.in_use, allSynapses_0.in_use,
		num_neurons * sizeof( bool* ), hipMemcpyDeviceToHost ) );

	for (int i = 0; i < num_neurons; i++) {
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.summationCoord[i], allSynapsesHost.summationCoord[i], 
			max_synapses * sizeof( Coordinate ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.W[i], allSynapsesHost.W[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		//HANDLE_ERROR( hipMemcpy ( allSynapses_1.summationPoint[i], allSynapsesHost.summationPoint[i],
		//	max_synapses * sizeof( BGFLOAT* ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.synapseCoord[i], allSynapsesHost.synapseCoord[i],
			max_synapses * sizeof( Coordinate ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.psr[i], allSynapsesHost.psr[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.decay[i], allSynapsesHost.decay[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.total_delay[i], allSynapsesHost.total_delay[i],
			max_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
		//HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayQueue[i], allSynapsesHost.delayQueue[i],
		//	max_synapses * sizeof( uint32_t* ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayIdx[i], allSynapsesHost.delayIdx[i],
			max_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.ldelayQueue[i], allSynapsesHost.ldelayQueue[i],
			max_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.type[i], allSynapsesHost.type[i],
			max_synapses * sizeof( synapseType ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.tau[i], allSynapsesHost.tau[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.r[i], allSynapsesHost.r[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.u[i], allSynapsesHost.u[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.D[i], allSynapsesHost.D[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.U[i], allSynapsesHost.U[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.F[i], allSynapsesHost.F[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.lastSpike[i], allSynapsesHost.lastSpike[i],
			max_synapses * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy ( allSynapses_1.in_use[i], allSynapsesHost.in_use[i],
			max_synapses * sizeof( bool ), hipMemcpyHostToDevice ) );

                // set summation points
                const int threadsPerBlock = 256;
                int blocksPerGrid = ( sim_info->totalNeurons + threadsPerBlock - 1 ) / threadsPerBlock;       
                setSynapseSummationPointDevice <<< blocksPerGrid, threadsPerBlock >>> (sim_info->totalNeurons, allNeuronsDevice, allSynapsesDevice, sim_info->width);
	}
}

/**
 * Set the summation points in device memory
 * @param[in] num_neurons        Number of neurons.
 * @param[in] allNeuronsDevice   Pointer to the Neuron structures in device memory.
 * @param[in] allSynapsesDevice  Pointer to the Synapse structures in device memory.
 * @param[in] width              Width of neuron map (assumes square).
 */
__global__ void setSynapseSummationPointDevice(int num_neurons, AllNeurons* allNeuronsDevice, AllSynapses* allSynapsesDevice, int width)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= num_neurons )
        return;

    int src_neuron = idx; 
    for (int syn_index = 0; syn_index < allSynapsesDevice->synapse_counts[src_neuron]; syn_index++) {
        int dest_neuron = allSynapsesDevice->summationCoord[src_neuron][syn_index].x 
            + allSynapsesDevice->summationCoord[src_neuron][syn_index].y * width;
        allSynapsesDevice->summationPoint[src_neuron][syn_index] = &( allNeuronsDevice->summation_map[dest_neuron] );
    }
}

void LIFGPUModel::copySynapseDeviceToHost( AllSynapses& allSynapsesHost, int num_neurons, int max_synapses ) {
	// copy everything necessary
	AllSynapses allSynapses_0;
	AllSynapses allSynapses_1(num_neurons, 0);

        HANDLE_ERROR( hipMemcpy ( &allSynapses_0, allSynapsesDevice, sizeof( AllSynapses ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapsesHost.synapse_counts, allSynapses_0.synapse_counts, 
		num_neurons * sizeof( size_t ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &allSynapsesHost.max_synapses, &allSynapsesDevice->max_synapses, 
		sizeof( size_t ), hipMemcpyDeviceToHost ) );

	HANDLE_ERROR( hipMemcpy ( allSynapses_1.summationCoord, allSynapses_0.summationCoord, 
		num_neurons * sizeof( Coordinate* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.W, allSynapses_0.W,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	//HANDLE_ERROR( hipMemcpy ( allSynapses_1.summationPoint, allSynapses_0.summationPoint,
	//	num_neurons * sizeof( BGFLOAT** ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.synapseCoord, allSynapses_0.synapseCoord,
		num_neurons * sizeof( Coordinate* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.psr, allSynapses_0.psr,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.decay, allSynapses_0.decay,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.total_delay, allSynapses_0.total_delay,
		num_neurons * sizeof( int* ), hipMemcpyDeviceToHost ) );
	//HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayQueue, allSynapses_0.delayQueue,
	//	num_neurons * sizeof( uint32_t** ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.delayIdx, allSynapses_0.delayIdx,
		num_neurons * sizeof( int* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.ldelayQueue, allSynapses_0.ldelayQueue,
		num_neurons * sizeof( int* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.type, allSynapses_0.type,
		num_neurons * sizeof( synapseType* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.tau, allSynapses_0.tau,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.r, allSynapses_0.r,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.u, allSynapses_0.u,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.D, allSynapses_0.D,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.U, allSynapses_0.U,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.F, allSynapses_0.F,
		num_neurons * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.lastSpike, allSynapses_0.lastSpike,
		num_neurons * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( allSynapses_1.in_use, allSynapses_0.in_use,
		num_neurons * sizeof( bool* ), hipMemcpyDeviceToHost ) );

	for (int i = 0; i < num_neurons; i++) {
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.summationCoord[i], allSynapses_1.summationCoord[i],
			max_synapses * sizeof( Coordinate ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.W[i], allSynapses_1.W[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		//HANDLE_ERROR( hipMemcpy ( allSynapsesHost.summationPoint[i], allSynapses_1.summationPoint[i],
		//	max_synapses * sizeof( BGFLOAT* ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.synapseCoord[i], allSynapses_1.synapseCoord[i],
			max_synapses * sizeof( Coordinate ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.psr[i], allSynapses_1.psr[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.decay[i], allSynapses_1.decay[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.total_delay[i], allSynapses_1.total_delay[i],
			max_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
		//HANDLE_ERROR( hipMemcpy ( allSynapsesHost.delayQueue[i], allSynapses_1.delayQueue[i],
		//	max_synapses * sizeof( uint32_t* ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.delayIdx[i], allSynapses_1.delayIdx[i],
			max_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.ldelayQueue[i], allSynapses_1.ldelayQueue[i],
			max_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.type[i], allSynapses_1.type[i],
			max_synapses * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.tau[i], allSynapses_1.tau[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.r[i], allSynapses_1.r[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.u[i], allSynapses_1.u[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.D[i], allSynapses_1.D[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.U[i], allSynapses_1.U[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.F[i], allSynapses_1.F[i],
			max_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.lastSpike[i], allSynapses_1.lastSpike[i],
			max_synapses * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy ( allSynapsesHost.in_use[i], allSynapses_1.in_use[i],
			max_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
	}
}
