/*
 * DynamicSpikingSynapse_struct_d.cu
 *
 */

#include "LIFGPUModel.h"

void LIFGPUModel::allocSynapseDeviceStruct( AllDSSynapses*& allSynapsesDevice, int num_neurons, int max_synapses ) {
	AllDSSynapses allSynapses;
	uint32_t max_total_synapses = max_synapses * num_neurons;

	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.summationCoord, max_total_synapses * sizeof( Coordinate ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.W, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.summationPoint, max_total_synapses * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.synapseCoord, max_total_synapses * sizeof( Coordinate ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.psr, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.decay, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.total_delay, max_total_synapses * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.delayQueue, max_total_synapses * sizeof( uint32_t ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.delayIdx, max_total_synapses * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.ldelayQueue, max_total_synapses * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.type, max_total_synapses * sizeof( synapseType ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.tau, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.r, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.u, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.D, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.U, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.F, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.lastSpike, max_total_synapses * sizeof( uint64_t ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.in_use, max_total_synapses * sizeof( bool ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.synapse_counts, num_neurons * sizeof( size_t ) ) );

	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapsesDevice, sizeof( AllDSSynapses ) ) );
	HANDLE_ERROR( hipMemcpy ( allSynapsesDevice, &allSynapses, sizeof( AllDSSynapses ), hipMemcpyHostToDevice ) );
}

void LIFGPUModel::deleteSynapseDeviceStruct( AllDSSynapses* allSynapsesDevice, int num_neurons, int max_synapses ) {
	AllDSSynapses allSynapses;

	HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDSSynapses ), hipMemcpyDeviceToHost ) );

	HANDLE_ERROR( hipFree( allSynapses.summationCoord ) );
	HANDLE_ERROR( hipFree( allSynapses.W ) );
	HANDLE_ERROR( hipFree( allSynapses.summationPoint ) );
	HANDLE_ERROR( hipFree( allSynapses.synapseCoord ) );
	HANDLE_ERROR( hipFree( allSynapses.psr ) );
	HANDLE_ERROR( hipFree( allSynapses.decay ) );
	HANDLE_ERROR( hipFree( allSynapses.total_delay ) );
	HANDLE_ERROR( hipFree( allSynapses.delayQueue ) );
	HANDLE_ERROR( hipFree( allSynapses.delayIdx ) );
	HANDLE_ERROR( hipFree( allSynapses.ldelayQueue ) );
	HANDLE_ERROR( hipFree( allSynapses.type ) );
	HANDLE_ERROR( hipFree( allSynapses.tau ) );
	HANDLE_ERROR( hipFree( allSynapses.r ) );
	HANDLE_ERROR( hipFree( allSynapses.u ) );
	HANDLE_ERROR( hipFree( allSynapses.D ) );
	HANDLE_ERROR( hipFree( allSynapses.U ) );
	HANDLE_ERROR( hipFree( allSynapses.F ) );
	HANDLE_ERROR( hipFree( allSynapses.lastSpike ) );
	HANDLE_ERROR( hipFree( allSynapses.in_use ) );
	HANDLE_ERROR( hipFree( allSynapses.synapse_counts ) );

	HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

void LIFGPUModel::copySynapseHostToDevice( AllDSSynapses* allSynapsesDevice, const AllDSSynapses& allSynapsesHost, int num_neurons, int max_synapses ) { // copy everything necessary
	uint32_t max_total_synapses = max_synapses * num_neurons;
	AllDSSynapses allSynapses_0;

        HANDLE_ERROR( hipMemcpy ( &allSynapses_0, allSynapsesDevice, sizeof( AllDSSynapses ), hipMemcpyDeviceToHost ) );

	HANDLE_ERROR( hipMemcpy ( allSynapses_0.synapse_counts, allSynapsesHost.synapse_counts, 
			num_neurons * sizeof( size_t ), hipMemcpyHostToDevice ) );
	allSynapses_0.maxSynapsesPerNeuron = allSynapsesHost.maxSynapsesPerNeuron;	
	allSynapses_0.total_synapse_counts = allSynapsesHost.total_synapse_counts;	
	HANDLE_ERROR( hipMemcpy ( allSynapsesDevice, &allSynapses_0, sizeof( AllDSSynapses ), hipMemcpyHostToDevice ) );

        HANDLE_ERROR( hipMemcpy ( allSynapses_0.summationCoord, allSynapsesHost.summationCoord,
                max_total_synapses * sizeof( Coordinate ),  hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.W, allSynapses_0.W,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.synapseCoord, allSynapsesHost.synapseCoord,
                max_total_synapses * sizeof( Coordinate ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.psr, allSynapses_0.psr,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.decay, allSynapsesHost.decay,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.total_delay, allSynapsesHost.total_delay,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.delayQueue, allSynapsesHost.delayQueue,
                max_total_synapses * sizeof( uint32_t ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.delayIdx, allSynapsesHost.delayIdx,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.ldelayQueue, allSynapsesHost.ldelayQueue,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.type, allSynapsesHost.type,
                max_total_synapses * sizeof( synapseType ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.tau, allSynapsesHost.tau,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.r, allSynapsesHost.r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.u, allSynapsesHost.u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.D, allSynapsesHost.D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.U, allSynapsesHost.U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.F, allSynapsesHost.F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.lastSpike, allSynapsesHost.lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.in_use, allSynapsesHost.in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyHostToDevice ) );
}

void LIFGPUModel::copySynapseDeviceToHost( AllDSSynapses* allSynapsesDevice, AllDSSynapses& allSynapsesHost, int num_neurons, int max_synapses ) {
	// copy everything necessary
	AllDSSynapses allSynapses_0;
	uint32_t max_total_synapses = max_synapses * num_neurons;

        HANDLE_ERROR( hipMemcpy ( &allSynapses_0, allSynapsesDevice, sizeof( AllDSSynapses ), hipMemcpyDeviceToHost ) );

	HANDLE_ERROR( hipMemcpy ( allSynapsesHost.synapse_counts, allSynapses_0.synapse_counts, 
		num_neurons * sizeof( size_t ), hipMemcpyDeviceToHost ) );
	allSynapsesHost.maxSynapsesPerNeuron = allSynapses_0.maxSynapsesPerNeuron;
	allSynapsesHost.total_synapse_counts = allSynapses_0.total_synapse_counts;

        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.summationCoord, allSynapses_0.summationCoord,
                max_total_synapses * sizeof( Coordinate ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.W, allSynapses_0.W,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.synapseCoord, allSynapses_0.synapseCoord,
                max_total_synapses * sizeof( Coordinate ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.psr, allSynapses_0.psr,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.decay, allSynapses_0.decay,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.total_delay, allSynapses_0.total_delay,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.delayQueue, allSynapses_0.delayQueue,
                max_total_synapses * sizeof( uint32_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.delayIdx, allSynapses_0.delayIdx,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.ldelayQueue, allSynapses_0.ldelayQueue,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.type, allSynapses_0.type,
                max_total_synapses * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.tau, allSynapses_0.tau,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.r, allSynapses_0.r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.u, allSynapses_0.u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.D, allSynapses_0.D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.U, allSynapses_0.U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.F, allSynapses_0.F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.lastSpike, allSynapses_0.lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.in_use, allSynapses_0.in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
}
