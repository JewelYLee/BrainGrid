#include "hip/hip_runtime.h"
#include "AllLIFNeurons.h"
#include "AllDSSynapses.h"
#include "Book.h"

//! Perform updating neurons for one time step.
__global__ void advanceNeuronsDevice( int totalNeurons, int maxSynapses, int maxSpikes, const BGFLOAT deltaT, uint64_t simulationStep, float* randNoise, AllIFNeurons* allNeuronsDevice, AllDSSynapses* allSynapsesDevice ); 

/**
 *  Notify outgoing synapses if neuron has fired.
 *  @param  sim_info    SimulationInfo class to read information from.
 */
void AllLIFNeurons::advanceNeurons( AllNeurons* allNeuronsDevice, AllSynapses* allSynapsesDevice, const SimulationInfo *sim_info, float* randNoise)
{
    int neuron_count = sim_info->totalNeurons;
    int maxSpikes = (int)((sim_info->epochDuration * sim_info->maxFiringRate));

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( neuron_count + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance neurons ------------->
    advanceNeuronsDevice <<< blocksPerGrid, threadsPerBlock >>> ( neuron_count, sim_info->maxSynapsesPerNeuron, maxSpikes, sim_info->deltaT, g_simulationStep, randNoise, (AllIFNeurons *)allNeuronsDevice, (AllDSSynapses*)allSynapsesDevice );
}

/* ------------------*\
|* # Global Functions
\* ------------------*/

// CUDA code for advancing neurons
/**
* @param[in] totalNeurons       Number of neurons.
* @param[in] maxSynapses        Maximum number of synapses per neuron.
* @param[in] maxSpikes
* @param[in] deltaT             Inner simulation step duration.
* @param[in] simulationStep     The current simulation step.
* @param[in] randNoise          Pointer to device random noise array.
* @param[in] allNeuronsDevice   Pointer to Neuron structures in device memory.
* @param[in] allSynapsesDevice  Pointer to Synapse structures in device memory.
*/
__global__ void advanceNeuronsDevice( int totalNeurons, int maxSynapses, int maxSpikes, const BGFLOAT deltaT, uint64_t simulationStep, float* randNoise, AllIFNeurons* allNeuronsDevice, AllDSSynapses* allSynapsesDevice ) {
        // determine which neuron this thread is processing
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if ( idx >= totalNeurons )
                return;

        allNeuronsDevice->hasFired[idx] = false;
        BGFLOAT& sp = allNeuronsDevice->summation_map[idx];
        BGFLOAT& vm = allNeuronsDevice->Vm[idx];
        BGFLOAT r_sp = sp;
        BGFLOAT r_vm = vm;

        if ( allNeuronsDevice->nStepsInRefr[idx] > 0 ) { // is neuron refractory?
                --allNeuronsDevice->nStepsInRefr[idx];
        } else if ( r_vm >= allNeuronsDevice->Vthresh[idx] ) { // should it fire?
                int& spikeCount = allNeuronsDevice->spikeCount[idx];
                int& spikeCountOffset = allNeuronsDevice->spikeCountOffset[idx];

                // Note that the neuron has fired!
                allNeuronsDevice->hasFired[idx] = true;

                // record spike time
                int idxSp = (spikeCount + spikeCountOffset) % maxSpikes;
                allNeuronsDevice->spike_history[idx][idxSp] = simulationStep;
                spikeCount++;

                // calculate the number of steps in the absolute refractory period
                allNeuronsDevice->nStepsInRefr[idx] = static_cast<int> ( allNeuronsDevice->Trefract[idx] / deltaT + 0.5 );

                // reset to 'Vreset'
                vm = allNeuronsDevice->Vreset[idx];

                // notify synapses of spike
                size_t synapse_counts = allSynapsesDevice->synapse_counts[idx];
                int synapse_notified = 0;
                for (int i = 0; synapse_notified < synapse_counts; i++) {
                        uint32_t iSyn = maxSynapses * idx + i;
                        if (allSynapsesDevice->in_use[iSyn] == true) {
                                uint32_t &delay_queue = allSynapsesDevice->delayQueue[iSyn];
                                int delayIdx = allSynapsesDevice->delayIdx[iSyn];
                                int ldelayQueue = allSynapsesDevice->ldelayQueue[iSyn];
                                int total_delay = allSynapsesDevice->total_delay[iSyn];

                                // Add to spike queue

                                // calculate index where to insert the spike into delayQueue
                                int idx = delayIdx +  total_delay;
                                if ( idx >= ldelayQueue ) {
                                        idx -= ldelayQueue;
                                }

                                // set a spike
                                //assert( !(delay_queue[0] & (0x1 << idx)) );
                                delay_queue |= (0x1 << idx);

                                synapse_notified++;
                        }
                }
        } else {
                r_sp += allNeuronsDevice->I0[idx]; // add IO

                // Random number alg. goes here
                r_sp += (randNoise[idx] * allNeuronsDevice->Inoise[idx]); // add cheap noise
                vm = allNeuronsDevice->C1[idx] * r_vm + allNeuronsDevice->C2[idx] * ( r_sp ); // decay Vm and add inputs
        }

        // clear synaptic input for next time step
        sp = 0;
}

