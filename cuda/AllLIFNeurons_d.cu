#include "hip/hip_runtime.h"
#include "AllLIFNeurons.h"
#include "AllDSSynapses.h"
#include "Book.h"

//! Perform updating neurons for one time step.
__global__ void advanceNeuronsDevice( int totalNeurons, uint64_t simulationStep, int maxSynapses, const BGFLOAT deltaT, float* randNoise, AllIFNeurons* allNeuronsDevice, AllDSSynapses* allSynapsesDevice );

/**
 *  Notify outgoing synapses if neuron has fired.
 *  @param  sim_info    SimulationInfo class to read information from.
 */
void AllLIFNeurons::advanceNeurons( AllNeurons* allNeuronsDevice, AllSynapses* allSynapsesDevice, const SimulationInfo *sim_info, float* randNoise)
{
    int neuron_count = sim_info->totalNeurons;

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( neuron_count + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance neurons ------------->
    advanceNeuronsDevice <<< blocksPerGrid, threadsPerBlock >>> ( neuron_count, g_simulationStep, sim_info->maxSynapsesPerNeuron, sim_info->deltaT, randNoise, (AllIFNeurons *)allNeuronsDevice, (AllDSSynapses*)allSynapsesDevice );
}

/* ------------------*\
|* # Global Functions
\* ------------------*/

// CUDA code for advancing neurons
/**
* @param[in] totalNeurons       Number of neurons.
* @param[in] simulationStep     The current simulation step.
* @param[in] maxSynapses        Maximum number of synapses per neuron.
* @param[in] deltaT             Inner simulation step duration.
* @param[in] randNoise          Pointer to device random noise array.
* @param[in] allNeuronsDevice   Pointer to Neuron structures in device memory.
* @param[in] allSynapsesDevice  Pointer to Synapse structures in device memory.
*/
__global__ void advanceNeuronsDevice( int totalNeurons, uint64_t simulationStep, int maxSynapses, const BGFLOAT deltaT, float* randNoise, AllIFNeurons* allNeuronsDevice, AllDSSynapses* allSynapsesDevice ) {
        // determine which neuron this thread is processing
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if ( idx >= totalNeurons )
                return;

        allNeuronsDevice->hasFired[idx] = false;
        BGFLOAT& sp = allNeuronsDevice->summation_map[idx];
        BGFLOAT& vm = allNeuronsDevice->Vm[idx];
        BGFLOAT r_sp = sp;
        BGFLOAT r_vm = vm;

        if ( allNeuronsDevice->nStepsInRefr[idx] > 0 ) { // is neuron refractory?
                --allNeuronsDevice->nStepsInRefr[idx];
        } else if ( r_vm >= allNeuronsDevice->Vthresh[idx] ) { // should it fire?
                // Note that the neuron has fired!
                allNeuronsDevice->hasFired[idx] = true;

                // record spike time
                allNeuronsDevice->spike_history[idx][allNeuronsDevice->spikeCount[idx]] = simulationStep;
                allNeuronsDevice->spikeCount[idx]++;

                // calculate the number of steps in the absolute refractory period
                allNeuronsDevice->nStepsInRefr[idx] = static_cast<int> ( allNeuronsDevice->Trefract[idx] / deltaT + 0.5 );

                // reset to 'Vreset'
                vm = allNeuronsDevice->Vreset[idx];

                // notify synapses of spike
                size_t synapse_counts = allSynapsesDevice->synapse_counts[idx];
                int synapse_notified = 0;
                for (int i = 0; synapse_notified < synapse_counts; i++) {
                        uint32_t iSyn = maxSynapses * idx + i;
                        if (allSynapsesDevice->in_use[iSyn] == true) {
                                uint32_t &delay_queue = allSynapsesDevice->delayQueue[iSyn];
                                int delayIdx = allSynapsesDevice->delayIdx[iSyn];
                                int ldelayQueue = allSynapsesDevice->ldelayQueue[iSyn];
                                int total_delay = allSynapsesDevice->total_delay[iSyn];

                                // Add to spike queue

                                // calculate index where to insert the spike into delayQueue
                                int idx = delayIdx +  total_delay;
                                if ( idx >= ldelayQueue ) {
                                        idx -= ldelayQueue;
                                }

                                // set a spike
                                //assert( !(delay_queue[0] & (0x1 << idx)) );
                                delay_queue |= (0x1 << idx);

                                synapse_notified++;
                        }
                }
        } else {
                r_sp += allNeuronsDevice->I0[idx]; // add IO

                // Random number alg. goes here
                r_sp += (randNoise[idx] * allNeuronsDevice->Inoise[idx]); // add cheap noise
                vm = allNeuronsDevice->C1[idx] * r_vm + allNeuronsDevice->C2[idx] * ( r_sp ); // decay Vm and add inputs
        }

        // clear synaptic input for next time step
        sp = 0;
}

