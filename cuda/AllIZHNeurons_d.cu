#include "hip/hip_runtime.h"
/*
 * AllIZHNeurons.cu
 *
 */

#include "AllSpikingSynapses.h"
#include "AllIZHNeurons.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all neurons' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allNeuronsDevice   Reference to the allNeurons struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::allocNeuronDeviceStruct( void** allNeuronsDevice, SimulationInfo *sim_info ) {
	AllIZHNeurons allNeurons;

	allocDeviceStruct( allNeurons, sim_info );

        HANDLE_ERROR( hipMalloc( allNeuronsDevice, sizeof( AllIZHNeurons ) ) );
        HANDLE_ERROR( hipMemcpy ( *allNeuronsDevice, &allNeurons, sizeof( AllIZHNeurons ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all neurons' states.
 *  (Helper function of allocNeuronDeviceStruct)
 *
 *  @param  allNeurons         Reference to the allIFNeurons struct.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::allocDeviceStruct( AllIZHNeurons &allNeurons, SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;

	AllIFNeurons::allocDeviceStruct( allNeurons, sim_info );
 
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Aconst, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Bconst, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Cconst, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Dconst, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.u, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.C3, count * sizeof( BGFLOAT ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allNeuronsDevice   Reference to the allNeurons struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::deleteNeuronDeviceStruct( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
	AllIZHNeurons allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allNeurons, sim_info );

	HANDLE_ERROR( hipFree( allNeuronsDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteNeuronDeviceStruct)
 *
 *  @param  allNeurons         Reference to the allIFNeurons struct.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::deleteDeviceStruct( AllIZHNeurons& allNeurons, const SimulationInfo *sim_info ) {
	HANDLE_ERROR( hipFree( allNeurons.Aconst ) );
	HANDLE_ERROR( hipFree( allNeurons.Bconst ) );
	HANDLE_ERROR( hipFree( allNeurons.Cconst ) );
	HANDLE_ERROR( hipFree( allNeurons.Dconst ) );
	HANDLE_ERROR( hipFree( allNeurons.u ) );
	HANDLE_ERROR( hipFree( allNeurons.C3 ) );

	AllIFNeurons::deleteDeviceStruct( allNeurons, sim_info );
}

/*
 *  Copy all neurons' data from host to device.
 *
 *  @param  allNeuronsDevice   Reference to the allNeurons struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::copyNeuronHostToDevice( void* allNeuronsDevice, const SimulationInfo *sim_info ) { 
	AllIZHNeurons allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allNeurons, sim_info );
}

/*
 *  Copy all neurons' data from host to device.
 *  (Helper function of copyNeuronHostToDevice)
 *
 *  @param  allNeurons         Reference to the allIFNeurons struct.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::copyHostToDevice( AllIZHNeurons& allNeurons, const SimulationInfo *sim_info ) { 
	int count = sim_info->totalNeurons;

	AllIFNeurons::copyHostToDevice( allNeurons, sim_info );

	HANDLE_ERROR( hipMemcpy ( allNeurons.Aconst, Aconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Bconst, Bconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Cconst, Cconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Dconst, Dconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.u, u, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.C3, C3, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
}

/*
 *  Copy all neurons' data from device to host.
 *
 *  @param  allNeuronsDevice   Reference to the allNeurons struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::copyNeuronDeviceToHost( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
	AllIZHNeurons allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );
	copyDeviceToHost( allNeurons, sim_info );
}

/*
 *  Copy all neurons' data from device to host.
 *  (Helper function of copyNeuronDeviceToHost)
 *
 *  @param  allNeurons         Reference to the allIFNeurons struct.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::copyDeviceToHost( AllIZHNeurons& allNeurons, const SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;

	AllIFNeurons::copyDeviceToHost( allNeurons, sim_info );

	HANDLE_ERROR( hipMemcpy ( Aconst, allNeurons.Aconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Bconst, allNeurons.Bconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Cconst, allNeurons.Cconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Dconst, allNeurons.Dconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( u, allNeurons.u, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( C3, allNeurons.C3, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

/*
 *  Copy spike history data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the allNeurons struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::copyNeuronDeviceSpikeHistoryToHost( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
        AllIZHNeurons allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeHistoryToHost( allNeurons, sim_info );
}

/*
 *  Copy spike counts data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the allNeurons struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::copyNeuronDeviceSpikeCountsToHost( void* allNeuronsDevice, const SimulationInfo *sim_info )
{
        AllIZHNeurons allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeCountsToHost( allNeurons, sim_info );
}

/*
 *  Clear the spike counts out of all neurons.
 *
 *  @param  allNeuronsDevice   Reference to the allNeurons struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIZHNeurons::clearNeuronSpikeCounts( void* allNeuronsDevice, const SimulationInfo *sim_info )
{
        AllIZHNeurons allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::clearDeviceSpikeCounts( allNeurons, sim_info );
}

/*
 *  Notify outgoing synapses if neuron has fired.
 *
 *  @param  sim_info    SimulationInfo class to read information from.
 */
void AllIZHNeurons::advanceNeurons( IAllSynapses &synapses, IAllNeurons* allNeuronsDevice, IAllSynapses* allSynapsesDevice, const SimulationInfo *sim_info, float* randNoise, SynapseIndexMap* synapseIndexMapDevice)
{
    int neuron_count = sim_info->totalNeurons;
    int maxSpikes = (int)((sim_info->epochDuration * sim_info->maxFiringRate));

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( neuron_count + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance neurons ------------->
    advanceIZHNeuronsDevice <<< blocksPerGrid, threadsPerBlock >>> ( neuron_count, sim_info->maxSynapsesPerNeuron, maxSpikes, sim_info->deltaT, g_simulationStep, randNoise, (AllIZHNeurons *)allNeuronsDevice, (AllSpikingSynapses*)allSynapsesDevice, synapseIndexMapDevice, (void (*)(const uint32_t, AllSpikingSynapses*))m_fpPreSpikeHit_h, (void (*)(const uint32_t, AllSpikingSynapses*))m_fpPostSpikeHit_h, m_fAllowBackPropagation );
}

/* ------------------*\
|* # Global Functions
\* ------------------*/

/*
 *  CUDA code for advancing izhikevich neurons
 *
 *  @param[in] totalNeurons          Number of neurons.
 *  @param[in] maxSynapses           Maximum number of synapses per neuron.
 *  @param[in] maxSpikes             Maximum number of spikes per neuron per epoch.
 *  @param[in] deltaT                Inner simulation step duration.
 *  @param[in] simulationStep        The current simulation step.
 *  @param[in] randNoise             Pointer to device random noise array.
 *  @param[in] allNeuronsDevice      Pointer to Neuron structures in device memory.
 *  @param[in] allSynapsesDevice     Pointer to Synapse structures in device memory.
 *  @param[in] synapseIndexMap       Inverse map, which is a table indexed by an input neuron and maps to the synapses that provide input to that neuron.
 *  @param[in] fpPreSpikeHit         Pointer to the device function preSpikeHit() function.
 *  @param[in] fpPostSpikeHit        Pointer to the device function postSpikeHit() function.
 *  @param[in] fAllowBackPropagation True if back propagaion is allowed.
 */
__global__ void advanceIZHNeuronsDevice( int totalNeurons, int maxSynapses, int maxSpikes, const BGFLOAT deltaT, uint64_t simulationStep, float* randNoise, AllIZHNeurons* allNeuronsDevice, AllSpikingSynapses* allSynapsesDevice, SynapseIndexMap* synapseIndexMapDevice, void (*fpPreSpikeHit)(const uint32_t, AllSpikingSynapses*), void (*fpPostSpikeHit)(const uint32_t, AllSpikingSynapses*), bool fAllowBackPropagation ) {
        // determine which neuron this thread is processing
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if ( idx >= totalNeurons )
                return;

        allNeuronsDevice->hasFired[idx] = false;
        BGFLOAT& sp = allNeuronsDevice->summation_map[idx];
        BGFLOAT& vm = allNeuronsDevice->Vm[idx];
        BGFLOAT& a = allNeuronsDevice->Aconst[idx];
        BGFLOAT& b = allNeuronsDevice->Bconst[idx];
        BGFLOAT& u = allNeuronsDevice->u[idx];
        BGFLOAT r_sp = sp;
        BGFLOAT r_vm = vm;
        BGFLOAT r_a = a;
        BGFLOAT r_b = b;
        BGFLOAT r_u = u;

        if ( allNeuronsDevice->nStepsInRefr[idx] > 0 ) { // is neuron refractory?
                --allNeuronsDevice->nStepsInRefr[idx];
        } else if ( r_vm >= allNeuronsDevice->Vthresh[idx] ) { // should it fire?
                int& spikeCount = allNeuronsDevice->spikeCount[idx];
                int& spikeCountOffset = allNeuronsDevice->spikeCountOffset[idx];

                // Note that the neuron has fired!
                allNeuronsDevice->hasFired[idx] = true;

                // record spike time
                int idxSp = (spikeCount + spikeCountOffset) % maxSpikes;
                allNeuronsDevice->spike_history[idx][idxSp] = simulationStep;
                spikeCount++;

                // calculate the number of steps in the absolute refractory period
                allNeuronsDevice->nStepsInRefr[idx] = static_cast<int> ( allNeuronsDevice->Trefract[idx] / deltaT + 0.5 );

                // reset to 'Vreset'
                vm = allNeuronsDevice->Cconst[idx] * 0.001;
                u = r_u + allNeuronsDevice->Dconst[idx];

                //notify incomming synapses of spike
                size_t synapse_counts = allSynapsesDevice->synapse_counts[idx];
                uint32_t synapse_notified = 0;
                if(fAllowBackPropagation && synapse_counts != 0){
                   for(uint32_t synapse_index = maxSynapses * idx ; synapse_notified < synapse_counts; synapse_index++){
                      if (allSynapsesDevice->in_use[synapse_index] == true) {
                         fpPreSpikeHit(synapse_index, allSynapsesDevice); 
                         synapse_notified++;
                      }
                   }
                }

                // notify outgoing synapses of spike
                synapse_counts = synapseIndexMapDevice->synapseCount[idx];
                if(synapse_counts != 0){
                   int beginIndex = synapseIndexMapDevice->outgoingSynapse_begin[idx]; //get the index of where this neuron's list of synapses are 
                   uint32_t * forwardMap_begin = &(synapseIndexMapDevice->forwardIndex[beginIndex]); //get the memory location of where that list begins
                   
                   //for each synapse, let them know we have fired
                   for(uint32_t i = 0; i < synapse_counts; i++){
                      fpPreSpikeHit(forwardMap_begin[i], allSynapsesDevice);
                   }
                   //synapse_notified += synapse_counts; //we could increment this every time we notified a synapse, but we know how many we are going to notify, and there currently isn't a way notification could fail so this seems better
                }
        } else {
                r_sp += allNeuronsDevice->I0[idx]; // add IO

                // Random number alg. goes here
                r_sp += (randNoise[idx] * allNeuronsDevice->Inoise[idx]); // add cheap noise

                BGFLOAT Vint = r_vm * 1000;

                // Izhikevich model integration step
                BGFLOAT Vb = Vint + allNeuronsDevice->C3[idx] * (0.04 * Vint * Vint + 5 * Vint + 140 - u);
                u = r_u + allNeuronsDevice->C3[idx] * r_a * (r_b * Vint - r_u);

                vm = Vb * 0.001 + allNeuronsDevice->C2[idx] * r_sp;  // add inputs
        }

        // clear synaptic input for next time step
        sp = 0;
}

