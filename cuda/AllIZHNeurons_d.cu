#include "hip/hip_runtime.h"
/*
 * AllIZHNeurons.cu
 *
 */

#include "AllIZHNeurons.h"
#include "AllDSSynapses.h"
#include "Book.h"

void AllIZHNeurons::allocNeuronDeviceStruct( void** allNeuronsDevice, SimulationInfo *sim_info ) {
	AllIZHNeurons allNeurons;

	allocDeviceStruct( allNeurons, sim_info );

        HANDLE_ERROR( hipMalloc( allNeuronsDevice, sizeof( AllIZHNeurons ) ) );
        HANDLE_ERROR( hipMemcpy ( *allNeuronsDevice, &allNeurons, sizeof( AllIZHNeurons ), hipMemcpyHostToDevice ) );
}

void AllIZHNeurons::allocDeviceStruct( AllIZHNeurons &allNeurons, SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;
	int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);

	AllIFNeurons::allocDeviceStruct( allNeurons, sim_info );
 
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Aconst, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Bconst, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Cconst, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Dconst, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.u, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.C3, count * sizeof( BGFLOAT ) ) );
}

void AllIZHNeurons::deleteNeuronDeviceStruct( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
	AllIZHNeurons allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allNeurons, sim_info );

	HANDLE_ERROR( hipFree( allNeuronsDevice ) );
}

void AllIZHNeurons::deleteDeviceStruct( AllIZHNeurons& allNeurons, const SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;

	HANDLE_ERROR( hipFree( allNeurons.Aconst ) );
	HANDLE_ERROR( hipFree( allNeurons.Bconst ) );
	HANDLE_ERROR( hipFree( allNeurons.Cconst ) );
	HANDLE_ERROR( hipFree( allNeurons.Dconst ) );
	HANDLE_ERROR( hipFree( allNeurons.u ) );
	HANDLE_ERROR( hipFree( allNeurons.C3 ) );

	AllIFNeurons::deleteDeviceStruct( allNeurons, sim_info );
}

void AllIZHNeurons::copyNeuronHostToDevice( void* allNeuronsDevice, const SimulationInfo *sim_info ) { 
	AllIZHNeurons allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allNeurons, sim_info );
}

void AllIZHNeurons::copyHostToDevice( AllIZHNeurons& allNeurons, const SimulationInfo *sim_info ) { 
	int count = sim_info->totalNeurons;

	AllIFNeurons::copyHostToDevice( allNeurons, sim_info );

	HANDLE_ERROR( hipMemcpy ( allNeurons.Aconst, Aconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Bconst, Bconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Cconst, Cconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Dconst, Dconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.u, u, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.C3, C3, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
}

void AllIZHNeurons::copyNeuronDeviceToHost( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
	AllIZHNeurons allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );
	copyDeviceToHost( allNeurons, sim_info );
}

void AllIZHNeurons::copyDeviceToHost( AllIZHNeurons& allNeurons, const SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;

	AllIFNeurons::copyDeviceToHost( allNeurons, sim_info );

	HANDLE_ERROR( hipMemcpy ( Aconst, allNeurons.Aconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Bconst, allNeurons.Bconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Cconst, allNeurons.Cconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Dconst, allNeurons.Dconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( u, allNeurons.u, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( C3, allNeurons.C3, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

/**
 *  Get spike history in AllIZHNeurons struct on device memory.
 *  @param  allNeuronsDevice      Reference to the allNeurons struct on device memory.
 *  @param  sim_info    SimulationInfo to refer from.
 */
void AllIZHNeurons::copyNeuronDeviceSpikeHistoryToHost( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
        AllIZHNeurons allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeHistoryToHost( allNeurons, sim_info );
}

/**
 *  Get spikeCount in AllIZHNeurons struct on device memory.
 *  @param  allNeuronsDevice      Reference to the allNeurons struct on device memory.
 *  @param  sim_info    SimulationInfo to refer from.
 */
void AllIZHNeurons::copyNeuronDeviceSpikeCountsToHost( void* allNeuronsDevice, const SimulationInfo *sim_info )
{
        AllIZHNeurons allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeCountsToHost( allNeurons, sim_info );
}

/** 
*  Clear the spike counts out of all Neurons.
 *  @param  allNeuronsDevice      Reference to the allNeurons struct on device memory.
 *  @param  sim_info    SimulationInfo to refer from.
*/
void AllIZHNeurons::clearNeuronSpikeCounts( void* allNeuronsDevice, const SimulationInfo *sim_info )
{
        AllIZHNeurons allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::clearDeviceSpikeCounts( allNeurons, sim_info );
}

__global__ void advanceNeuronsDevice( int totalNeurons, uint64_t simulationStep, int maxSynapses, const BGFLOAT deltaT, float* randNoise, AllIZHNeurons* allNeuronsDevice, AllDSSynapses* allSynapsesDevice );

/**
 *  Notify outgoing synapses if neuron has fired.
 *  @param  sim_info    SimulationInfo class to read information from.
 */
void AllIZHNeurons::advanceNeurons( AllNeurons* allNeuronsDevice, AllSynapses* allSynapsesDevice, const SimulationInfo *sim_info, float* randNoise)
{
    int neuron_count = sim_info->totalNeurons;

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( neuron_count + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance neurons ------------->
    advanceNeuronsDevice <<< blocksPerGrid, threadsPerBlock >>> ( neuron_count, g_simulationStep, sim_info->maxSynapsesPerNeuron, sim_info->deltaT, randNoise, (AllIZHNeurons *)allNeuronsDevice, (AllDSSynapses*)allSynapsesDevice );
}

/* ------------------*\
|* # Global Functions
\* ------------------*/

// CUDA code for advancing neurons
/**
* @param[in] totalNeurons       Number of neurons.
* @param[in] simulationStep     The current simulation step.
* @param[in] maxSynapses        Maximum number of synapses per neuron.
* @param[in] deltaT             Inner simulation step duration.
* @param[in] randNoise          Pointer to device random noise array.
* @param[in] allNeuronsDevice   Pointer to Neuron structures in device memory.
* @param[in] allSynapsesDevice  Pointer to Synapse structures in device memory.
*/
__global__ void advanceNeuronsDevice( int totalNeurons, uint64_t simulationStep, int maxSynapses, const BGFLOAT deltaT, float* randNoise, AllIZHNeurons* allNeuronsDevice, AllDSSynapses* allSynapsesDevice ) {
        // determine which neuron this thread is processing
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if ( idx >= totalNeurons )
                return;

        allNeuronsDevice->hasFired[idx] = false;
        BGFLOAT& sp = allNeuronsDevice->summation_map[idx];
        BGFLOAT& vm = allNeuronsDevice->Vm[idx];
        BGFLOAT& a = allNeuronsDevice->Aconst[idx];
        BGFLOAT& b = allNeuronsDevice->Bconst[idx];
        BGFLOAT& u = allNeuronsDevice->u[idx];
        BGFLOAT r_sp = sp;
        BGFLOAT r_vm = vm;
        BGFLOAT r_a = a;
        BGFLOAT r_b = b;
        BGFLOAT r_u = u;

        if ( allNeuronsDevice->nStepsInRefr[idx] > 0 ) { // is neuron refractory?
                --allNeuronsDevice->nStepsInRefr[idx];
        } else if ( r_vm >= allNeuronsDevice->Vthresh[idx] ) { // should it fire?
                // Note that the neuron has fired!
                allNeuronsDevice->hasFired[idx] = true;

                // record spike time
                allNeuronsDevice->spike_history[idx][allNeuronsDevice->spikeCount[idx]] = simulationStep;
                allNeuronsDevice->spikeCount[idx]++;

                // calculate the number of steps in the absolute refractory period
                allNeuronsDevice->nStepsInRefr[idx] = static_cast<int> ( allNeuronsDevice->Trefract[idx] / deltaT + 0.5 );

                // reset to 'Vreset'
                vm = allNeuronsDevice->Cconst[idx] * 0.001;
                u = r_u + allNeuronsDevice->Dconst[idx];

                // notify synapses of spike
                size_t synapse_counts = allSynapsesDevice->synapse_counts[idx];
                int synapse_notified = 0;
                for (int i = 0; synapse_notified < synapse_counts; i++) {
                        uint32_t iSyn = maxSynapses * idx + i;
                        if (allSynapsesDevice->in_use[iSyn] == true) {
                                uint32_t &delay_queue = allSynapsesDevice->delayQueue[iSyn];
                                int delayIdx = allSynapsesDevice->delayIdx[iSyn];
                                int ldelayQueue = allSynapsesDevice->ldelayQueue[iSyn];
                                int total_delay = allSynapsesDevice->total_delay[iSyn];

                                // Add to spike queue

                                // calculate index where to insert the spike into delayQueue
                                int idx = delayIdx +  total_delay;
                                if ( idx >= ldelayQueue ) {
                                        idx -= ldelayQueue;
                                }

                                // set a spike
                                //assert( !(delay_queue[0] & (0x1 << idx)) );
                                delay_queue |= (0x1 << idx);

                                synapse_notified++;
                        }
                }
        } else {
                r_sp += allNeuronsDevice->I0[idx]; // add IO

                // Random number alg. goes here
                r_sp += (randNoise[idx] * allNeuronsDevice->Inoise[idx]); // add cheap noise

                BGFLOAT Vint = r_vm * 1000;

                // Izhikevich model integration step
                BGFLOAT Vb = Vint + allNeuronsDevice->C3[idx] * (0.04 * Vint * Vint + 5 * Vint + 140 - u);
                u = r_u + allNeuronsDevice->C3[idx] * r_a * (r_b * Vint - r_u);

                vm = Vb * 0.001 + allNeuronsDevice->C2[idx] * r_sp;  // add inputs
        }

        // clear synaptic input for next time step
        sp = 0;
}

