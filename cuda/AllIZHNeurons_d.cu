/*
 * AllIZHNeurons.cu
 *
 */

#include "AllIZHNeurons.h"
#include "Book.h"

void AllIZHNeurons::allocNeuronDeviceStruct( void** allNeuronsDevice, SimulationInfo *sim_info ) {
	AllIZHNeurons allNeurons;

	allocDeviceStruct( allNeurons, sim_info );

        HANDLE_ERROR( hipMalloc( allNeuronsDevice, sizeof( AllIZHNeurons ) ) );
        HANDLE_ERROR( hipMemcpy ( *allNeuronsDevice, &allNeurons, sizeof( AllIZHNeurons ), hipMemcpyHostToDevice ) );
}

void AllIZHNeurons::allocDeviceStruct( AllIZHNeurons &allNeurons, SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;
	int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);

	AllIFNeurons::allocDeviceStruct( allNeurons, sim_info );
 
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Aconst, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Bconst, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Cconst, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Dconst, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.u, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.C3, count * sizeof( BGFLOAT ) ) );
}

void AllIZHNeurons::deleteNeuronDeviceStruct( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
	AllIZHNeurons allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allNeurons, sim_info );

	HANDLE_ERROR( hipFree( allNeuronsDevice ) );
}

void AllIZHNeurons::deleteDeviceStruct( AllIZHNeurons& allNeurons, const SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;

	HANDLE_ERROR( hipFree( allNeurons.Aconst ) );
	HANDLE_ERROR( hipFree( allNeurons.Bconst ) );
	HANDLE_ERROR( hipFree( allNeurons.Cconst ) );
	HANDLE_ERROR( hipFree( allNeurons.Dconst ) );
	HANDLE_ERROR( hipFree( allNeurons.u ) );
	HANDLE_ERROR( hipFree( allNeurons.C3 ) );

	AllIFNeurons::deleteDeviceStruct( allNeurons, sim_info );
}

void AllIZHNeurons::copyNeuronHostToDevice( void* allNeuronsDevice, const SimulationInfo *sim_info ) { 
	AllIZHNeurons allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allNeurons, sim_info );
}

void AllIZHNeurons::copyHostToDevice( AllIZHNeurons& allNeurons, const SimulationInfo *sim_info ) { 
	int count = sim_info->totalNeurons;

	AllIFNeurons::copyHostToDevice( allNeurons, sim_info );

	HANDLE_ERROR( hipMemcpy ( allNeurons.Aconst, Aconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Bconst, Bconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Cconst, Cconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Dconst, Dconst, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.u, u, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.C3, C3, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
}

void AllIZHNeurons::copyNeuronDeviceToHost( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
	AllIZHNeurons allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );
	copyDeviceToHost( allNeurons, sim_info );
}

void AllIZHNeurons::copyDeviceToHost( AllIZHNeurons& allNeurons, const SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;

	AllIFNeurons::copyDeviceToHost( allNeurons, sim_info );

	HANDLE_ERROR( hipMemcpy ( Aconst, allNeurons.Aconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Bconst, allNeurons.Bconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Cconst, allNeurons.Cconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Dconst, allNeurons.Dconst, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( u, allNeurons.u, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( C3, allNeurons.C3, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

/**
 *  Get spike history in AllIZHNeurons struct on device memory.
 *  @param  allNeuronsDevice      Reference to the allNeurons struct on device memory.
 *  @param  sim_info    SimulationInfo to refer from.
 */
void AllIZHNeurons::copyNeuronDeviceSpikeHistoryToHost( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
        AllIZHNeurons allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeHistoryToHost( allNeurons, sim_info );
}

/**
 *  Get spikeCount in AllIZHNeurons struct on device memory.
 *  @param  allNeuronsDevice      Reference to the allNeurons struct on device memory.
 *  @param  sim_info    SimulationInfo to refer from.
 */
void AllIZHNeurons::copyNeuronDeviceSpikeCountsToHost( void* allNeuronsDevice, const SimulationInfo *sim_info )
{
        AllIZHNeurons allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeCountsToHost( allNeurons, sim_info );
}

/** 
*  Clear the spike counts out of all Neurons.
 *  @param  allNeuronsDevice      Reference to the allNeurons struct on device memory.
 *  @param  sim_info    SimulationInfo to refer from.
*/
void AllIZHNeurons::clearNeuronSpikeCounts( void* allNeuronsDevice, const SimulationInfo *sim_info )
{
        AllIZHNeurons allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIZHNeurons ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::clearDeviceSpikeCounts( allNeurons, sim_info );
}

