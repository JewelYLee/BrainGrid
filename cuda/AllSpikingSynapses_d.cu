#include "hip/hip_runtime.h"
/*
 * AllSpikingSynapses.cu
 *
 */

#include "AllSpikingSynapses.h"
#include "Book.h"

/**
 *  Advance all the Synapses in the simulation.
 *  @param  sim_info    SimulationInfo class to read information from.
 */
void AllSpikingSynapses::advanceSynapses(AllSynapses* allSynapsesDevice, AllNeurons* allNeuronsDevice, void* synapseIndexMapDevice, const SimulationInfo *sim_info)
{
    unsigned long long fpChangePSR_h;
    getFpChangePSR(fpChangePSR_h);

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( total_synapse_counts + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance synapses ------------->
    advanceSynapsesDevice <<< blocksPerGrid, threadsPerBlock >>> ( total_synapse_counts, (SynapseIndexMap*)synapseIndexMapDevice, g_simulationStep, sim_info->deltaT, (AllSpikingSynapses*)allSynapsesDevice, (void (*)(AllSpikingSynapses*, const uint32_t, const uint64_t, const BGFLOAT))fpChangePSR_h );
}

void AllSpikingSynapses::getFpPreSpikeHit(unsigned long long& fpPreSpikeHit_h)
{
    unsigned long long *fpPreSpikeHit_d;

    HANDLE_ERROR( hipMalloc(&fpPreSpikeHit_d, sizeof(unsigned long long)) );

    getFpPreSpikeHitDevice<<<1,1>>>((void (**)(const uint32_t, AllSpikingSynapses*))fpPreSpikeHit_d);

    HANDLE_ERROR( hipMemcpy(&fpPreSpikeHit_h, fpPreSpikeHit_d, sizeof(unsigned long long), hipMemcpyDeviceToHost) );

    HANDLE_ERROR( hipFree( fpPreSpikeHit_d ) );
}

void AllSpikingSynapses::getFpPostSpikeHit(unsigned long long& fpPostSpikeHit_h)
{
    unsigned long long *fpPostSpikeHit_d;

    HANDLE_ERROR( hipMalloc(&fpPostSpikeHit_d, sizeof(unsigned long long)) );

    getFpPostSpikeHitDevice<<<1,1>>>((void (**)(const uint32_t, AllSpikingSynapses*))fpPostSpikeHit_d);

    HANDLE_ERROR( hipMemcpy(&fpPostSpikeHit_h, fpPostSpikeHit_d, sizeof(unsigned long long), hipMemcpyDeviceToHost) );

    HANDLE_ERROR( hipFree( fpPostSpikeHit_d ) );
}

void AllSpikingSynapses::getFpChangePSR(unsigned long long& fpChangePSR_h)
{
    unsigned long long *fpChangePSR_d;

    HANDLE_ERROR( hipMalloc(&fpChangePSR_d, sizeof(unsigned long long)) );

    getFpChangePSRDevice<<<1,1>>>((void (**)(AllSpikingSynapses*, const uint32_t, const uint64_t, const BGFLOAT))fpChangePSR_d);

    HANDLE_ERROR( hipMemcpy(&fpChangePSR_h, fpChangePSR_d, sizeof(unsigned long long), hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipFree( fpChangePSR_d ) );
}

/* ------------------*\
|* # Global Functions
\* ------------------*/

/** 
* @param[in] total_synapse_counts       Total number of synapses.
* @param[in] synapseIndexMap            Inverse map, which is a table indexed by an input neuron and maps to the synapses that provide input to that neuron.
* @param[in] simulationStep             The current simulation step.
* @param[in] deltaT                     Inner simulation step duration.
* @param[in] allSynapsesDevice  Pointer to Synapse structures in device memory.
*/
__global__ void advanceSynapsesDevice ( int total_synapse_counts, SynapseIndexMap* synapseIndexMapDevice, uint64_t simulationStep, const BGFLOAT deltaT, AllSpikingSynapses* allSynapsesDevice, void (*fpChangePSR)(AllSpikingSynapses*, const uint32_t, const uint64_t, const BGFLOAT) ) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if ( idx >= total_synapse_counts )
                return;

        uint32_t iSyn = synapseIndexMapDevice->activeSynapseIndex[idx];

        BGFLOAT &psr = allSynapsesDevice->psr[iSyn];
        BGFLOAT decay = allSynapsesDevice->decay[iSyn];

        // Checks if there is an input spike in the queue.
        bool isFired = isSpikeQueueDevice(allSynapsesDevice, iSyn);

        // is an input in the queue?
        if (isFired) {
                fpChangePSR(allSynapsesDevice, iSyn, simulationStep, deltaT);
        }
        // decay the post spike response
        psr *= decay;
}

__device__ bool isSpikeQueueDevice(AllSpikingSynapses* allSynapsesDevice, uint32_t iSyn)
{
    uint32_t &delay_queue = allSynapsesDevice->delayQueue[iSyn];
    int &delayIdx = allSynapsesDevice->delayIdx[iSyn];
    int ldelayQueue = allSynapsesDevice->ldelayQueue[iSyn];

    uint32_t delayMask = (0x1 << delayIdx);
    bool isFired = delay_queue & (delayMask);
    delay_queue &= ~(delayMask);
    if ( ++delayIdx >= ldelayQueue ) {
            delayIdx = 0;
    }

    return isFired;
}

__global__ void getFpPreSpikeHitDevice(void (**fpPreSpikeHit_d)(const uint32_t, AllSpikingSynapses*))
{
    *fpPreSpikeHit_d = preSpikeHitDevice;
}

__global__ void getFpPostSpikeHitDevice(void (**fpPostSpikeHit_d)(const uint32_t, AllSpikingSynapses*))
{
    *fpPostSpikeHit_d = postSpikeHitDevice;
}

__device__ void preSpikeHitDevice( const uint32_t iSyn, AllSpikingSynapses* allSynapsesDevice ) {
        uint32_t &delay_queue = allSynapsesDevice->delayQueue[iSyn];
        int delayIdx = allSynapsesDevice->delayIdx[iSyn];
        int ldelayQueue = allSynapsesDevice->ldelayQueue[iSyn];
        int total_delay = allSynapsesDevice->total_delay[iSyn];

        // Add to spike queue

        // calculate index where to insert the spike into delayQueue
        int idx = delayIdx +  total_delay;
        if ( idx >= ldelayQueue ) {
                idx -= ldelayQueue;
        }

        // set a spike
        //assert( !(delay_queue[0] & (0x1 << idx)) );
        delay_queue |= (0x1 << idx);
}

__device__ void postSpikeHitDevice( const uint32_t iSyn, AllSpikingSynapses* allSynapsesDevice ) {
}

__global__ void getFpChangePSRDevice(void (**fpChangePSR_d)(AllSpikingSynapses*, const uint32_t, const uint64_t, const BGFLOAT))
{
    *fpChangePSR_d = changePSR;
}

__device__ void changePSR(AllSpikingSynapses* allSynapsesDevice, const uint32_t iSyn, const uint64_t simulationStep, const BGFLOAT deltaT)
{
    BGFLOAT &psr = allSynapsesDevice->psr[iSyn];
    BGFLOAT &W = allSynapsesDevice->W[iSyn];
    BGFLOAT &decay = allSynapsesDevice->decay[iSyn];

    psr += ( W / decay );    // calculate psr
}

/**
* Adds a synapse to the network.  Requires the locations of the source and
* destination neurons.
* @param allSynapsesDevice      Pointer to the Synapse structures in device memory.
* @param type                   Type of the Synapse to create.
* @param src_neuron             Index of the source neuron.
* @param dest_neuron            Index of the destination neuron.
* @param source_x               X location of source.
* @param source_y               Y location of source.
* @param dest_x                 X location of destination.
* @param dest_y                 Y location of destination.
* @param sum_point              Pointer to the summation point.
* @param deltaT                 The time step size.
* @param W_d                    Array of synapse weight.
* @param num_neurons            The number of neurons.
*/
__device__ void addSynapse(AllSpikingSynapses* allSynapsesDevice, synapseType type, const int src_neuron, const int dest_neuron, int source_x, int source_y, int dest_x, int dest_y, BGFLOAT *sum_point, const BGFLOAT deltaT, BGFLOAT* W_d, int num_neurons, void (*fpCreateSynapse)(AllSpikingSynapses*, const int, const int, int, int, int, int, BGFLOAT*, const BGFLOAT, synapseType))
{
    if (allSynapsesDevice->synapse_counts[src_neuron] >= allSynapsesDevice->maxSynapsesPerNeuron) {
        return; // TODO: ERROR!
    }

    // add it to the list
    size_t synapse_index;
    size_t max_synapses = allSynapsesDevice->maxSynapsesPerNeuron;
    uint32_t iSync = max_synapses * src_neuron;
    for (synapse_index = 0; synapse_index < max_synapses; synapse_index++) {
        if (!allSynapsesDevice->in_use[iSync + synapse_index]) {
            break;
        }
    }

    allSynapsesDevice->synapse_counts[src_neuron]++;

    // create a synapse
    fpCreateSynapse(allSynapsesDevice, src_neuron, synapse_index, source_x, source_y, dest_x, dest_y, sum_point, deltaT, type );
    allSynapsesDevice->W[iSync + synapse_index] = W_d[src_neuron * num_neurons + dest_neuron] * synSign(type) * AllSynapses::SYNAPSE_STRENGTH_ADJUSTMENT;
}

/**
* Remove a synapse from the network.
* @param[in] allSynapsesDevice         Pointer to the Synapse structures in device memory.
* @param neuron_index   Index of a neuron.
* @param synapse_index  Index of a synapse.
* @param[in] maxSynapses        Maximum number of synapses per neuron.
*/
__device__ void eraseSynapse( AllSpikingSynapses* allSynapsesDevice, const int neuron_index, const int synapse_index, int maxSynapses )
{
    uint32_t iSync = maxSynapses * neuron_index + synapse_index;
    allSynapsesDevice->synapse_counts[neuron_index]--;
    allSynapsesDevice->in_use[iSync] = false;
    allSynapsesDevice->summationPoint[iSync] = NULL;
}

/**
* Returns the type of synapse at the given coordinates
* @param[in] allNeuronsDevice          Pointer to the Neuron structures in device memory.
* @param src_neuron             Index of the source neuron.
* @param dest_neuron            Index of the destination neuron.
*/
__device__ synapseType synType( AllSpikingNeurons* allNeuronsDevice, const int src_neuron, const int dest_neuron )
{
    if ( allNeuronsDevice->neuron_type_map[src_neuron] == INH && allNeuronsDevice->neuron_type_map[dest_neuron] == INH )
        return II;
    else if ( allNeuronsDevice->neuron_type_map[src_neuron] == INH && allNeuronsDevice->neuron_type_map[dest_neuron] == EXC )
        return IE;
    else if ( allNeuronsDevice->neuron_type_map[src_neuron] == EXC && allNeuronsDevice->neuron_type_map[dest_neuron] == INH )
        return EI;
    else if ( allNeuronsDevice->neuron_type_map[src_neuron] == EXC && allNeuronsDevice->neuron_type_map[dest_neuron] == EXC )
        return EE;

    return STYPE_UNDEF;

}

/**
* Return 1 if originating neuron is excitatory, -1 otherwise.
* @param[in] t  synapseType I to I, I to E, E to I, or E to E
* @return 1 or -1
*/
__device__ int synSign( synapseType t )
{
        switch ( t )
        {
        case II:
        case IE:
                return -1;
        case EI:
        case EE:
                return 1;
        }

        return 0;
}
