#include "hip/hip_runtime.h"
/** - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - **\ 
 * @authors Aaron Oziel, Sean Blackbourn 
 *
 * Fumitaka Kawasaki (5/3/14):
 * All functions were completed and working. Therefore, the followng comments
 * were removed. 
 *
 * Aaron Wrote (2/3/14):
 * All comments are now tracking progress in conversion from old GpuSim_struct.cu
 * file to the new one here. This is a quick key to keep track of their meanings. 
 *
 *	TODO = 	Needs work and/or is blank. Used to indicate possibly problematic 
 *				functions. 
 *	DONE = 	Likely complete functions. Will still need to be checked for
 *				variable continuity and proper arguments. 
 *   REMOVED =	Deleted, likely due to it becoming unnecessary or not necessary 
 *				for GPU implementation. These functions will likely have to be 
 *				removed from the Model super class.
 *    COPIED = 	These functions were in the original GpuSim_struct.cu file 
 *				and were directly copy-pasted across to this file. 
 *
\** - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - **/

#include "GPUSpikingModel.h"

extern "C" {
void normalMTGPU(float * randNoise_d);
void initMTGPU(unsigned int seed, unsigned int blocks, unsigned int threads, unsigned int nPerRng, unsigned int mt_rng_count);
}

__global__ void setSynapseSummationPointDevice(int num_neurons, AllSpikingNeurons* allNeuronsDevice, AllDSSynapses* allSynapsesDevice, int max_synapses, int width);

//! Perform updating synapses for one time step.
__global__ void advanceSynapsesDevice ( int total_synapse_counts, GPUSpikingModel::SynapseIndexMap* synapseIndexMapDevice, uint64_t simulationStep, const BGFLOAT deltaT, AllDSSynapses* allSynapsesDevice );

//! Calculate summation point.
__global__ void calcSummationMapDevice( int totalNeurons, GPUSpikingModel::SynapseIndexMap* synapseIndexMapDevice, AllDSSynapses* allSynapsesDevice );

//! Update the network.
__global__ void updateNetworkDevice( int num_neurons, int width, BGFLOAT deltaT, BGFLOAT* W_d, int maxSynapses, AllSpikingNeurons* allNeuronsDevice, AllDSSynapses* allSynapsesDevice );

//! Add a synapse to the network.
__device__ void addSynapse( AllDSSynapses* allSynapsesDevice, synapseType type, const int src_neuron, const int dest_neuron, int source_x, int source_y, int dest_x, int dest_y, BGFLOAT *sum_point, const BGFLOAT deltaT, BGFLOAT* W_d, int num_neurons );

//! Create a synapse.
__device__ void createSynapse( AllDSSynapses* allSynapsesDevice, const int neuron_index, const int synapse_index, int source_x, int source_y, int dest_x, int dest_y, BGFLOAT *sum_point, const BGFLOAT deltaT, synapseType type );

//! Remove a synapse from the network.
__device__ void eraseSynapse( AllDSSynapses* allSynapsesDevice, const int neuron_index, const int synapse_index, int maxSynapses );

//! Get the type of synapse.
__device__ synapseType synType( AllSpikingNeurons* allNeuronsDevice, const int src_neuron, const int dest_neuron );

//! Get the type of synapse (excitatory or inhibitory)
__device__ int synSign( synapseType t );

#ifdef PERFORMANCE_METRICS
float g_time;
hipEvent_t start, stop;
#endif // PERFORMANCE_METRICS

// ----------------------------------------------------------------------------

GPUSpikingModel::GPUSpikingModel(Connections *conns, AllNeurons *neurons, AllSynapses *synapses, Layout *layout) : 	
	Model::Model(conns, neurons, synapses, layout),
	synapseIndexMapDevice(NULL),
	randNoise_d(NULL),
	m_allNeuronsDevice(NULL),
	m_allSynapsesDevice(NULL)
{
}

GPUSpikingModel::~GPUSpikingModel() 
{
	//Let Model base class handle de-allocation
}

/**
* Allocates memories on CUDA device.
* @param[in] sim_info			Pointer to the simulation information.
*/
void GPUSpikingModel::allocDeviceStruct(void** allNeuronsDevice, void** allSynapsesDevice, SimulationInfo *sim_info)
{
	// Allocate Neurons and Synapses strucs on GPU device memory
	m_neurons->allocNeuronDeviceStruct( allNeuronsDevice, sim_info );
	m_synapses->allocSynapseDeviceStruct( allSynapsesDevice, sim_info );

	// Allocate memory for random noise array
	int neuron_count = sim_info->totalNeurons;
	size_t randNoise_d_size = neuron_count * sizeof (float);	// size of random noise array
	HANDLE_ERROR( hipMalloc ( ( void ** ) &randNoise_d, randNoise_d_size ) );

	// Copy host neuron and synapse arrays into GPU device
	m_neurons->copyNeuronHostToDevice( *allNeuronsDevice, sim_info );
	m_synapses->copySynapseHostToDevice( *allSynapsesDevice, sim_info );

	// allocate synapse inverse map
	allocSynapseImap( neuron_count );

	// create a synapse index map on device memory
	createSynapseImap(*m_synapses, sim_info);
}

void GPUSpikingModel::deleteDeviceStruct(void** allNeuronsDevice, void** allSynapsesDevice, SimulationInfo *sim_info)
{
    // copy device synapse and neuron structs to host memory
    m_neurons->copyNeuronDeviceToHost( *allNeuronsDevice, sim_info );

    // Deallocate device memory
    m_neurons->deleteNeuronDeviceStruct( *allNeuronsDevice, sim_info );

    // copy device synapse and neuron structs to host memory
    m_synapses->copySynapseDeviceToHost( *allSynapsesDevice, sim_info );

    // Deallocate device memory
    m_synapses->deleteSynapseDeviceStruct( *allSynapsesDevice, sim_info );

    deleteSynapseImap();

    HANDLE_ERROR( hipFree( randNoise_d ) );
}

/**
 *  Sets up the Simulation.
 *  @param  sim_info    SimulationInfo class to read information from.
 *  @param  simRecorder Pointer to the simulation recordig object.
 */
void GPUSpikingModel::setupSim(SimulationInfo *sim_info, IRecorder* simRecorder)
{
    // Set device ID
    HANDLE_ERROR( hipSetDevice( g_deviceId ) );

    Model::setupSim(sim_info, simRecorder);

    //initialize Mersenne Twister
    //assuming neuron_count >= 100 and is a multiple of 100. Note rng_mt_rng_count must be <= MT_RNG_COUNT
    int rng_blocks = 25; //# of blocks the kernel will use
    int rng_nPerRng = 4; //# of iterations per thread (thread granularity, # of rands generated per thread)
    int rng_mt_rng_count = sim_info->totalNeurons/rng_nPerRng; //# of threads to generate for neuron_count rand #s
    int rng_threads = rng_mt_rng_count/rng_blocks; //# threads per block needed
    initMTGPU(sim_info->seed, rng_blocks, rng_threads, rng_nPerRng, rng_mt_rng_count);

#ifdef PERFORMANCE_METRICS
    hipEventCreate( &start );
    hipEventCreate( &stop );

    t_gpu_rndGeneration = 0.0f;
    t_gpu_advanceNeurons = 0.0f;
    t_gpu_advanceSynapses = 0.0f;
    t_gpu_calcSummation = 0.0f;
#endif // PERFORMANCE_METRICS

    // allocates memories on CUDA device
    allocDeviceStruct((void **)&m_allNeuronsDevice, (void **)&m_allSynapsesDevice, sim_info);

    // set device summation points
    int neuron_count = sim_info->totalNeurons;
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( neuron_count + threadsPerBlock - 1 ) / threadsPerBlock;
    setSynapseSummationPointDevice <<< blocksPerGrid, threadsPerBlock >>> (neuron_count, m_allNeuronsDevice, m_allSynapsesDevice, sim_info->maxSynapsesPerNeuron, sim_info->width);
}

/** 
*  Begin terminating the simulator.
*  @param  sim_info    SimulationInfo to refer.
*/
void GPUSpikingModel::cleanupSim(SimulationInfo *sim_info)
{
    // deallocates memories on CUDA device
    deleteDeviceStruct((void**)&m_allNeuronsDevice, (void**)&m_allSynapsesDevice, sim_info);

#ifdef PERFORMANCE_METRICS
    hipEventDestroy( start );
    hipEventDestroy( stop );
#endif // PERFORMANCE_METRICS
}

/**
 *  Loads the simulation based on istream input.
 *  @param  input   istream to read from.
 *  @param  sim_info    used as a reference to set info for neurons and synapses.
 */
void GPUSpikingModel::loadMemory(istream& input, const SimulationInfo *sim_info)
{
    Model::loadMemory(input, sim_info);
   
    // create a synapse index map on device memory
    createSynapseImap(*m_synapses, sim_info);

    // Reinitialize device struct - Copy host neuron and synapse arrays into GPU device
    m_neurons->copyNeuronHostToDevice( m_allNeuronsDevice, sim_info );
    m_synapses->copySynapseHostToDevice( m_allSynapsesDevice, sim_info );

    // set summation points
    int neuron_count = sim_info->totalNeurons;
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( neuron_count + threadsPerBlock - 1 ) / threadsPerBlock;
    setSynapseSummationPointDevice <<< blocksPerGrid, threadsPerBlock >>> (neuron_count, m_allNeuronsDevice, m_allSynapsesDevice, sim_info->maxSynapsesPerNeuron, sim_info->width);
}

/** 
*  Advance everything in the model one time step. In this case, that
*  means calling all of the kernels that do the "micro step" updating
*  (i.e., NOT the stuff associated with growth).
*  @param  sim_info    SimulationInfo class to read information from.
*/
void GPUSpikingModel::advance(const SimulationInfo *sim_info)
{
	size_t total_synapse_counts = m_synapses->total_synapse_counts;

	// CUDA parameters
	const int threadsPerBlock = 256;
	int blocksPerGrid;

#ifdef PERFORMANCE_METRICS
	startTimer();
#endif // PERFORMANCE_METRICS

	normalMTGPU(randNoise_d);

#ifdef PERFORMANCE_METRICS
	lapTime(t_gpu_rndGeneration);
	startTimer();
#endif // PERFORMANCE_METRICS

	// display running info to console
	// Advance neurons ------------->
	advanceNeurons(sim_info);

#ifdef PERFORMANCE_METRICS
	lapTime(t_gpu_advanceNeurons);
	startTimer();
#endif // PERFORMANCE_METRICS

	// Advance synapses ------------->
	advanceSynapses(sim_info);

#ifdef PERFORMANCE_METRICS
	lapTime(t_gpu_advanceSynapses);
	startTimer();
#endif // PERFORMANCE_METRICS

	// calculate summation point
        calcSummationMap(sim_info);

#ifdef PERFORMANCE_METRICS
	lapTime(t_gpu_calcSummation);
#endif // PERFORMANCE_METRICS
}

/**
 *  Advance all the Synapses in the simulation.
 *  @param  sim_info    SimulationInfo class to read information from.
 */
void GPUSpikingModel::advanceSynapses(const SimulationInfo *sim_info)
{
    size_t total_synapse_counts = m_synapses->total_synapse_counts;

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( total_synapse_counts + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance synapses ------------->
    advanceSynapsesDevice <<< blocksPerGrid, threadsPerBlock >>> ( total_synapse_counts, synapseIndexMapDevice, g_simulationStep, sim_info->deltaT, m_allSynapsesDevice );
}

void GPUSpikingModel::calcSummationMap(const SimulationInfo *sim_info)
{
    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( sim_info->totalNeurons + threadsPerBlock - 1 ) / threadsPerBlock;

    calcSummationMapDevice <<< blocksPerGrid, threadsPerBlock >>> ( sim_info->totalNeurons, synapseIndexMapDevice, m_allSynapsesDevice );
}

/** 
*  Update the connection of all the Neurons and Synapses of the simulation.
*  @param  currentStep the current step of the simulation.
*  @param  sim_info    SimulationInfo class to read information from.
*  @param  simRecorder Pointer to the simulation recordig object.
*/
void GPUSpikingModel::updateConnections(const int currentStep, const SimulationInfo *sim_info, IRecorder* simRecorder)
{
	const int num_neurons = sim_info->totalNeurons;
	updateHistory(currentStep, sim_info->epochDuration, *m_neurons, sim_info, simRecorder);
	// Update the distance between frontiers of Neurons
	m_conns->updateFrontiers(num_neurons);
	// Update the areas of overlap in between Neurons
	m_conns->updateOverlap(num_neurons);
	updateWeights(sim_info->totalNeurons, *m_neurons, *m_synapses, sim_info);
}

/**
 *  Get synapse_counts in AllSynapses struct on device memory.
 *  @param  allSynapsesHost     Reference to the AllSynapses struct on host memory.
 *  @param  neuron_coun         The number of neurons.
 */
void GPUSpikingModel::copyDeviceSynapseCountsToHost(AllSynapses &allSynapsesHost, int neuron_count)
{
        AllDSSynapses allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, m_allSynapsesDevice, sizeof( AllDSSynapses ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.synapse_counts, allSynapses.synapse_counts, neuron_count * sizeof( size_t ), hipMemcpyDeviceToHost ) );
}

/** 
 *  Get summationCoord and in_use in AllSynapses struct on device memory.
 *  @param  allSynapsesHost     Reference to the AllSynapses struct on host memory.
 *  @param  neuron_coun         The number of neurons.
 *  @param  max_synapses        Maximum number of synapses per neuron.
 */
void GPUSpikingModel::copyDeviceSynapseSumCoordToHost(AllSynapses &allSynapsesHost, int neuron_count, int max_synapses)
{
        AllDSSynapses allSynapses_0;

        HANDLE_ERROR( hipMemcpy ( &allSynapses_0, m_allSynapsesDevice, sizeof( AllDSSynapses ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.summationCoord, allSynapses_0.summationCoord,
                max_synapses * neuron_count * sizeof( Coordinate ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( allSynapsesHost.in_use, allSynapses_0.in_use,
                max_synapses * neuron_count * sizeof( bool ), hipMemcpyDeviceToHost ) );
}

/** 
*  Update the weight of the Synapses in the simulation.
*  Note: Platform Dependent.
*  @param  num_neurons number of neurons to update.
*  @param  neurons the Neuron list to search from.
*  @param  synapses    the Synapse list to search from.
*  @param  sim_info    SimulationInfo to refer from.
*/
void GPUSpikingModel::updateWeights(const int num_neurons, AllNeurons &neurons, AllSynapses &synapses, const SimulationInfo *sim_info)
{
        // For now, we just set the weights to equal the areas. We will later
        // scale it and set its sign (when we index and get its sign).
        (*m_conns->W) = (*m_conns->area);

        int width = sim_info->width;
        BGFLOAT deltaT = sim_info->deltaT;

        // CUDA parameters
        const int threadsPerBlock = 256;
        int blocksPerGrid;

        // allocate memories
        size_t W_d_size = sim_info->totalNeurons * sim_info->totalNeurons * sizeof (BGFLOAT);
        BGFLOAT* W_h = new BGFLOAT[W_d_size];
        BGFLOAT* W_d;
        HANDLE_ERROR( hipMalloc ( ( void ** ) &W_d, W_d_size ) );

        // copy weight data to the device memory
        for ( int i = 0 ; i < sim_info->totalNeurons; i++ )
                for ( int j = 0; j < sim_info->totalNeurons; j++ )
                        W_h[i * sim_info->totalNeurons + j] = (*m_conns->W)(i, j);

        HANDLE_ERROR( hipMemcpy ( W_d, W_h, W_d_size, hipMemcpyHostToDevice ) );

        blocksPerGrid = ( sim_info->totalNeurons + threadsPerBlock - 1 ) / threadsPerBlock;
        updateNetworkDevice <<< blocksPerGrid, threadsPerBlock >>> ( sim_info->totalNeurons, width, deltaT, W_d, sim_info->maxSynapsesPerNeuron, m_allNeuronsDevice, m_allSynapsesDevice );

        // free memories
        HANDLE_ERROR( hipFree( W_d ) );
        delete[] W_h;

        // copy device synapse count to host memory
        copyDeviceSynapseCountsToHost(synapses, num_neurons);
        // copy device synapse summation coordinate to host memory
        copyDeviceSynapseSumCoordToHost(synapses, num_neurons, sim_info->maxSynapsesPerNeuron);
        // create synapse inverse map
        createSynapseImap( synapses, sim_info );
}

/* ------------------*\
|* # Helper Functions
\* ------------------*/

/**
 *  Allocate device memory for synapse inverse map.
 *  @param  count	The number of neurons.
 */
void GPUSpikingModel::allocSynapseImap( int count )
{
	SynapseIndexMap synapseIndexMap;

	HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIndexMap.incomingSynapse_begin, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIndexMap.synapseCount, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMemset(synapseIndexMap.incomingSynapse_begin, 0, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMemset(synapseIndexMap.synapseCount, 0, count * sizeof( int ) ) );

	HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIndexMapDevice, sizeof( SynapseIndexMap ) ) );
	HANDLE_ERROR( hipMemcpy( synapseIndexMapDevice, &synapseIndexMap, sizeof( SynapseIndexMap ), hipMemcpyHostToDevice ) );
}

/**
 *  Deallocate device memory for synapse inverse map.
 */
void GPUSpikingModel::deleteSynapseImap(  )
{
	SynapseIndexMap synapseIndexMap;

	HANDLE_ERROR( hipMemcpy ( &synapseIndexMap, synapseIndexMapDevice, sizeof( SynapseIndexMap ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipFree( synapseIndexMap.incomingSynapse_begin ) );
	HANDLE_ERROR( hipFree( synapseIndexMap.synapseCount ) );
	HANDLE_ERROR( hipFree( synapseIndexMap.inverseIndex ) );
	HANDLE_ERROR( hipFree( synapseIndexMap.activeSynapseIndex ) );
	HANDLE_ERROR( hipFree( synapseIndexMapDevice ) );
}

/** 
 *  Copy SynapseIndexMap in host memory to SynapseIndexMap in device memory.
 *  @param  synapseIndexMapHost		Reference to the SynapseIndexMap in host memory.
 *  @param  neuron_count		The number of neurons.
 *  @param  total_synapse_counts	The number of synapses.
 */
void GPUSpikingModel::copySynapseIndexMapHostToDevice(SynapseIndexMap &synapseIndexMapHost, int neuron_count, int total_synapse_counts)
{
	SynapseIndexMap synapseIndexMap;

	HANDLE_ERROR( hipMemcpy ( &synapseIndexMap, synapseIndexMapDevice, sizeof( SynapseIndexMap ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( synapseIndexMap.incomingSynapse_begin, synapseIndexMapHost.incomingSynapse_begin, neuron_count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( synapseIndexMap.synapseCount, synapseIndexMapHost.synapseCount, neuron_count * sizeof( int ), hipMemcpyHostToDevice ) );
	// the number of synapses may change, so we reallocate the memory
	if (synapseIndexMap.inverseIndex != NULL) {
		HANDLE_ERROR( hipFree( synapseIndexMap.inverseIndex ) );
	}
	HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIndexMap.inverseIndex, total_synapse_counts * sizeof( uint32_t ) ) );
	HANDLE_ERROR( hipMemcpy ( synapseIndexMap.inverseIndex, synapseIndexMapHost.inverseIndex, total_synapse_counts * sizeof( uint32_t ), hipMemcpyHostToDevice ) );

	if (synapseIndexMap.activeSynapseIndex != NULL) {
		HANDLE_ERROR( hipFree( synapseIndexMap.activeSynapseIndex ) );
	}
	HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIndexMap.activeSynapseIndex, total_synapse_counts * sizeof( uint32_t ) ) );
	HANDLE_ERROR( hipMemcpy ( synapseIndexMap.activeSynapseIndex, synapseIndexMapHost.activeSynapseIndex, total_synapse_counts * sizeof( uint32_t ), hipMemcpyHostToDevice ) );

	HANDLE_ERROR( hipMemcpy ( synapseIndexMapDevice, &synapseIndexMap, sizeof( SynapseIndexMap ), hipMemcpyHostToDevice ) );
}

/**
 *  Create a synapse index map on device memory.
 *  @param  synapses     Reference to the AllSynapses struct on host memory.
 *  @param] sim_info     Pointer to the simulation information.
 */
void GPUSpikingModel::createSynapseImap(AllSynapses &synapses, const SimulationInfo* sim_info )
{
	int neuron_count = sim_info->totalNeurons;
	int width = sim_info->width;
	int total_synapse_counts = 0;

	// count the total synapses
        for ( int i = 0; i < neuron_count; i++ )
        {
                assert( synapses.synapse_counts[i] < synapses.maxSynapsesPerNeuron );
                total_synapse_counts += synapses.synapse_counts[i];
        }

        DEBUG ( cout << "total_synapse_counts: " << total_synapse_counts << endl; )

        if ( total_synapse_counts == 0 )
        {
                return;
        }

        // allocate memories for inverse map
        vector<uint32_t>* rgSynapseSynapseIndexMap = new vector<uint32_t>[neuron_count];

        uint32_t syn_i = 0;
	int n_inUse = 0;

        // create synapse inverse map
	SynapseIndexMap synapseIndexMap(neuron_count, total_synapse_counts);
        for (int i = 0; i < neuron_count; i++)
        {
                for ( int j = 0; j < synapses.maxSynapsesPerNeuron; j++, syn_i++ )
                {
                        uint32_t iSyn = synapses.maxSynapsesPerNeuron * i + j;
                        if ( synapses.in_use[iSyn] == true )
                        {
                                int idx = synapses.summationCoord[iSyn].x
                                        + synapses.summationCoord[iSyn].y * width;
                                rgSynapseSynapseIndexMap[idx].push_back(syn_i);

				synapseIndexMap.activeSynapseIndex[n_inUse] = syn_i;
                                n_inUse++;
                        }
                }
        }

        assert( total_synapse_counts == n_inUse ); 
        synapses.total_synapse_counts = total_synapse_counts; 

        syn_i = 0;
        for (int i = 0; i < neuron_count; i++)
        {
                synapseIndexMap.incomingSynapse_begin[i] = syn_i;
                synapseIndexMap.synapseCount[i] = rgSynapseSynapseIndexMap[i].size();

                for ( int j = 0; j < rgSynapseSynapseIndexMap[i].size(); j++, syn_i++)
                {
                        synapseIndexMap.inverseIndex[syn_i] = rgSynapseSynapseIndexMap[i][j];
                }
        }

        // copy inverse map to the device memory
	copySynapseIndexMapHostToDevice(synapseIndexMap, neuron_count, total_synapse_counts);

        // delete memories
        delete[] rgSynapseSynapseIndexMap;
}

/**
 *  Update the Neuron's history.
 *  @param  currentStep current step of the simulation
 *  @param  epochDuration    duration of the 
 *  @param  neurons the list to update.
 *  @param  sim_info    SimulationInfo to refer from.
 *  @param  simRecorder Pointer to the simulation recordig object.
 */
void GPUSpikingModel::updateHistory(const int currentStep, BGFLOAT epochDuration, AllNeurons &neurons, const SimulationInfo *sim_info, IRecorder* simRecorder)
{
    // Calculate growth cycle firing rate for previous period
    neurons.copyNeuronDeviceSpikeCountsToHost(m_allNeuronsDevice, sim_info);
    neurons.copyNeuronDeviceSpikeHistoryToHost(m_allNeuronsDevice, sim_info);

    Model::updateHistory(currentStep, epochDuration, sim_info, simRecorder);

    // clear spike count
    neurons.clearNeuronSpikeCounts(m_allNeuronsDevice, sim_info);
}

/* ------------------*\
|* # Global Functions
\* ------------------*/

/**
 * Set the summation points in device memory
 * @param[in] num_neurons        Number of neurons.
 * @param[in] allNeuronsDevice   Pointer to the Neuron structures in device memory.
 * @param[in] allSynapsesDevice  Pointer to the Synapse structures in device memory.
 * @param[in] max_synapses       Maximum number of synapses per neuron.
 * @param[in] width              Width of neuron map (assumes square).
 */
__global__ void setSynapseSummationPointDevice(int num_neurons, AllSpikingNeurons* allNeuronsDevice, AllDSSynapses* allSynapsesDevice, int max_synapses, int width)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= num_neurons )
        return;

    int src_neuron = idx;
    int n_inUse = 0;
    for (int syn_index = 0; n_inUse < allSynapsesDevice->synapse_counts[src_neuron]; syn_index++) {
        if (allSynapsesDevice->in_use[max_synapses * src_neuron + syn_index] == true) {
            int dest_neuron = allSynapsesDevice->summationCoord[max_synapses * src_neuron + syn_index].x
                + allSynapsesDevice->summationCoord[max_synapses * src_neuron + syn_index].y * width;
            allSynapsesDevice->summationPoint[max_synapses * src_neuron + syn_index] = &( allNeuronsDevice->summation_map[dest_neuron] );
            n_inUse++;
        }
    }
}

/** 
* @param[in] total_synapse_counts       Total number of synapses.
* @param[in] synapseIndexMap            Inverse map, which is a table indexed by an input neuron and maps to the synapses that provide input to that neuron.
* @param[in] simulationStep             The current simulation step.
* @param[in] deltaT                     Inner simulation step duration.
* @param[in] allSynapsesDevice  Pointer to Synapse structures in device memory.
*/
__global__ void advanceSynapsesDevice ( int total_synapse_counts, GPUSpikingModel::SynapseIndexMap* synapseIndexMapDevice, uint64_t simulationStep, const BGFLOAT deltaT, AllDSSynapses* allSynapsesDevice ) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if ( idx >= total_synapse_counts )
                return;

        uint32_t iSyn = synapseIndexMapDevice->activeSynapseIndex[idx];

        BGFLOAT &psr = allSynapsesDevice->psr[iSyn];
        BGFLOAT decay = allSynapsesDevice->decay[iSyn];

        // Checks if there is an input spike in the queue.
        uint32_t &delay_queue = allSynapsesDevice->delayQueue[iSyn];
        int &delayIdx = allSynapsesDevice->delayIdx[iSyn];
        int ldelayQueue = allSynapsesDevice->ldelayQueue[iSyn];

        uint32_t delayMask = (0x1 << delayIdx);
        bool isFired = delay_queue & (delayMask);
        delay_queue &= ~(delayMask);
        if ( ++delayIdx >= ldelayQueue ) {
                delayIdx = 0;
        }

        // is an input in the queue?
        if (isFired) {
                uint64_t &lastSpike = allSynapsesDevice->lastSpike[iSyn];
                BGFLOAT &r = allSynapsesDevice->r[iSyn];
                BGFLOAT &u = allSynapsesDevice->u[iSyn];
                BGFLOAT D = allSynapsesDevice->D[iSyn];
                BGFLOAT F = allSynapsesDevice->F[iSyn];
                BGFLOAT U = allSynapsesDevice->U[iSyn];
                BGFLOAT W = allSynapsesDevice->W[iSyn];

                // adjust synapse parameters
                if (lastSpike != ULONG_MAX) {
                        BGFLOAT isi = (simulationStep - lastSpike) * deltaT ;
                        r = 1 + ( r * ( 1 - u ) - 1 ) * exp( -isi / D );
                        u = U + u * ( 1 - U ) * exp( -isi / F );
                }
                psr += ( ( W / decay ) * u * r );// calculate psr
                lastSpike = simulationStep; // record the time of the spike
        }

        // decay the post spike response
        psr *= decay;
}

/** 
* @param[in] totalNeurons       Number of neurons.
* @param[in] synapseIndexMap    Inverse map, which is a table indexed by an input neuron and maps to the synapses that provide input to that neuron.
* @param[in] allSynapsesDevice  Pointer to Synapse structures in device memory.
*/
__global__ void calcSummationMapDevice( int totalNeurons, GPUSpikingModel::SynapseIndexMap* synapseIndexMapDevice, AllDSSynapses* allSynapsesDevice ) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if ( idx >= totalNeurons )
                return;

        uint32_t iCount = synapseIndexMapDevice->synapseCount[idx];
        if (iCount != 0) {
                int beginIndex = synapseIndexMapDevice->incomingSynapse_begin[idx];
                uint32_t* inverseMap_begin = &( synapseIndexMapDevice->inverseIndex[beginIndex] );
                BGFLOAT sum = 0.0;
                uint32_t syn_i = inverseMap_begin[0];
                BGFLOAT &summationPoint = *( allSynapsesDevice->summationPoint[syn_i] );
                for ( uint32_t i = 0; i < iCount; i++ ) {
                        syn_i = inverseMap_begin[i];
                        sum += allSynapsesDevice->psr[syn_i];
                }
                summationPoint = sum;
        }
}

/**
* Adjust the strength of the synapse or remove it from the synapse map if it has gone below 
* zero.
* @param[in] num_neurons        Number of neurons.
* @param[in] width              Width of neuron map (assumes square).
* @param[in] deltaT             The time step size.
* @param[in] W_d                Array of synapse weight.
* @param[in] maxSynapses        Maximum number of synapses per neuron.
* @param[in] allNeuronsDevice          Pointer to the Neuron structures in device memory.
* @param[in] allSynapsesDevice         Pointer to the Synapse structures in device memory.
*/
__global__ void updateNetworkDevice( int num_neurons, int width, BGFLOAT deltaT, BGFLOAT* W_d, int maxSynapses, AllSpikingNeurons* allNeuronsDevice, AllDSSynapses* allSynapsesDevice )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= num_neurons )
        return;

    int adjusted = 0;
    //int could_have_been_removed = 0; // TODO: use this value
    int removed = 0;
    int added = 0;

    // Scale and add sign to the areas
    // visit each neuron 'a'
    int src_neuron = idx;
    int xa = src_neuron % width;
    int ya = src_neuron / width;

    // and each destination neuron 'b'
    for (int dest_neuron = 0; dest_neuron < num_neurons; dest_neuron++) {
        int xb = dest_neuron % width;
        int yb = dest_neuron / width;

        // visit each synapse at (xa,ya)
        bool connected = false;
        synapseType type = synType(allNeuronsDevice, src_neuron, dest_neuron);

        // for each existing synapse
        size_t synapse_counts = allSynapsesDevice->synapse_counts[src_neuron];
        int synapse_adjusted = 0;
        for (size_t synapse_index = 0; synapse_adjusted < synapse_counts; synapse_index++) {
            uint32_t iSyn = maxSynapses * src_neuron + synapse_index;
            if (allSynapsesDevice->in_use[iSyn] == true) {
                // if there is a synapse between a and b
                if (allSynapsesDevice->summationCoord[iSyn].x == xb &&
                    allSynapsesDevice->summationCoord[iSyn].y == yb) {
                    connected = true;
                    adjusted++;

                    // adjust the strength of the synapse or remove
                    // it from the synapse map if it has gone below
                    // zero.
                    if (W_d[src_neuron * num_neurons + dest_neuron] < 0) {
                        removed++;
                        eraseSynapse(allSynapsesDevice, src_neuron, synapse_index, maxSynapses);
                    } else {
                        // adjust
                        // g_synapseStrengthAdjustmentConstant is 1.0e-8;
                        allSynapsesDevice->W[iSyn] = W_d[src_neuron * num_neurons
                            + dest_neuron] * synSign(type) * SYNAPSE_STRENGTH_ADJUSTMENT;
                    }
                }
                synapse_adjusted++;
            }
        }

        // if not connected and weight(a,b) > 0, add a new synapse from a to b
        if (!connected && (W_d[src_neuron * num_neurons +  dest_neuron] > 0)) {
            // locate summation point
            BGFLOAT* sum_point = &( allNeuronsDevice->summation_map[dest_neuron] );
            added++;

            addSynapse(allSynapsesDevice, type, src_neuron, dest_neuron, xa, ya, xb, yb, sum_point, deltaT, W_d, num_neurons);

        }
    }
}

/** 
* Adds a synapse to the network.  Requires the locations of the source and
* destination neurons.
* @param allSynapsesDevice      Pointer to the Synapse structures in device memory.
* @param type                   Type of the Synapse to create.
* @param src_neuron             Index of the source neuron.
* @param dest_neuron            Index of the destination neuron.
* @param source_x               X location of source.
* @param source_y               Y location of source.
* @param dest_x                 X location of destination.
* @param dest_y                 Y location of destination.
* @param sum_point              Pointer to the summation point.
* @param deltaT                 The time step size.
* @param W_d                    Array of synapse weight.
* @param num_neurons            The number of neurons.
*/
__device__ void addSynapse(AllDSSynapses* allSynapsesDevice, synapseType type, const int src_neuron, const int dest_neuron, int source_x, int source_y, int dest_x, int dest_y, BGFLOAT *sum_point, const BGFLOAT deltaT, BGFLOAT* W_d, int num_neurons)
{
    if (allSynapsesDevice->synapse_counts[src_neuron] >= allSynapsesDevice->maxSynapsesPerNeuron) {
        return; // TODO: ERROR!
    }

    // add it to the list
    size_t synapse_index;
    size_t max_synapses = allSynapsesDevice->maxSynapsesPerNeuron;
    uint32_t iSync = max_synapses * src_neuron;
    for (synapse_index = 0; synapse_index < max_synapses; synapse_index++) {
        if (!allSynapsesDevice->in_use[iSync + synapse_index]) {
            break;
        }
    }

    allSynapsesDevice->synapse_counts[src_neuron]++;

    // create a synapse
    createSynapse(allSynapsesDevice, src_neuron, synapse_index, source_x, source_y, dest_x, dest_y, sum_point, deltaT, type );
    allSynapsesDevice->W[iSync + synapse_index] = W_d[src_neuron * num_neurons + dest_neuron] * synSign(type) * SYNAPSE_STRENGTH_ADJUSTMENT;
}

/**
 *  Create a Synapse and connect it to the model.
 *  @param allSynapsesDevice    Pointer to the Synapse structures in device memory.
 *  @param neuron_index         Index of the source neuron.
 *  @param synapse_index        Index of the Synapse to create.
 *  @param source_x             X location of source.
 *  @param source_y             Y location of source.
 *  @param dest_x               X location of destination.
 *  @param dest_y               Y location of destination.
 *  @param sum_point            Pointer to the summation point.
 *  @param deltaT               The time step size.
 *  @param type                 Type of the Synapse to create.
 */
__device__ void createSynapse(AllDSSynapses* allSynapsesDevice, const int neuron_index, const int synapse_index, int source_x, int source_y, int dest_x, int dest_y, BGFLOAT *sum_point, const BGFLOAT deltaT, synapseType type)
{
    BGFLOAT delay;
    size_t max_synapses = allSynapsesDevice->maxSynapsesPerNeuron;
    uint32_t iSyn = max_synapses * neuron_index + synapse_index;

    allSynapsesDevice->in_use[iSyn] = true;
    allSynapsesDevice->summationPoint[iSyn] = sum_point;
    allSynapsesDevice->summationCoord[iSyn].x = dest_x;
    allSynapsesDevice->summationCoord[iSyn].y = dest_y;
    allSynapsesDevice->synapseCoord[iSyn].x = source_x;
    allSynapsesDevice->synapseCoord[iSyn].y = source_y;
    allSynapsesDevice->W[iSyn] = 10.0e-9;

    allSynapsesDevice->delayQueue[iSyn] = 0;
    allSynapsesDevice->delayIdx[iSyn] = 0;
    allSynapsesDevice->ldelayQueue[iSyn] = LENGTH_OF_DELAYQUEUE;

    allSynapsesDevice->psr[iSyn] = 0.0;
    allSynapsesDevice->r[iSyn] = 1.0;
    allSynapsesDevice->u[iSyn] = 0.4;     // DEFAULT_U
    allSynapsesDevice->lastSpike[iSyn] = ULONG_MAX;
    allSynapsesDevice->type[iSyn] = type;

    allSynapsesDevice->U[iSyn] = DEFAULT_U;
    allSynapsesDevice->tau[iSyn] = DEFAULT_tau;

    BGFLOAT U;
    BGFLOAT D;
    BGFLOAT F;
    BGFLOAT tau;
    switch (type) {
        case II:
            U = 0.32;
            D = 0.144;
            F = 0.06;
            tau = 6e-3;
            delay = 0.8e-3;
            break;
        case IE:
            U = 0.25;
            D = 0.7;
            F = 0.02;
            tau = 6e-3;
            delay = 0.8e-3;
            break;
        case EI:
            U = 0.05;
            D = 0.125;
            F = 1.2;
            tau = 3e-3;
            delay = 0.8e-3;
            break;
        case EE:
            U = 0.5;
            D = 1.1;
            F = 0.05;
            tau = 3e-3;
            delay = 1.5e-3;
            break;
        default:
            break;
    }

    allSynapsesDevice->U[iSyn] = U;
    allSynapsesDevice->D[iSyn] = D;
    allSynapsesDevice->F[iSyn] = F;

    allSynapsesDevice->tau[iSyn] = tau;
    allSynapsesDevice->decay[iSyn] = exp( -deltaT / tau );
    allSynapsesDevice->total_delay[iSyn] = static_cast<int>( delay / deltaT ) + 1;

    size_t size = allSynapsesDevice->total_delay[iSyn] / ( sizeof(uint8_t) * 8 ) + 1;
    assert( size <= BYTES_OF_DELAYQUEUE );
}

/** 
* Remove a synapse from the network.
* @param[in] allSynapsesDevice         Pointer to the Synapse structures in device memory.
* @param neuron_index   Index of a neuron.
* @param synapse_index  Index of a synapse.
* @param[in] maxSynapses        Maximum number of synapses per neuron.
*/
__device__ void eraseSynapse( AllDSSynapses* allSynapsesDevice, const int neuron_index, const int synapse_index, int maxSynapses )
{
    uint32_t iSync = maxSynapses * neuron_index + synapse_index;
    allSynapsesDevice->synapse_counts[neuron_index]--;
    allSynapsesDevice->in_use[iSync] = false;
    allSynapsesDevice->summationPoint[iSync] = NULL;
}

/** 
* Returns the type of synapse at the given coordinates
* @param[in] allNeuronsDevice          Pointer to the Neuron structures in device memory.
* @param src_neuron             Index of the source neuron.
* @param dest_neuron            Index of the destination neuron.
*/
__device__ synapseType synType( AllSpikingNeurons* allNeuronsDevice, const int src_neuron, const int dest_neuron )
{
    if ( allNeuronsDevice->neuron_type_map[src_neuron] == INH && allNeuronsDevice->neuron_type_map[dest_neuron] == INH )
        return II;
    else if ( allNeuronsDevice->neuron_type_map[src_neuron] == INH && allNeuronsDevice->neuron_type_map[dest_neuron] == EXC )
        return IE;
    else if ( allNeuronsDevice->neuron_type_map[src_neuron] == EXC && allNeuronsDevice->neuron_type_map[dest_neuron] == INH )
        return EI;
    else if ( allNeuronsDevice->neuron_type_map[src_neuron] == EXC && allNeuronsDevice->neuron_type_map[dest_neuron] == EXC )
        return EE;

    return STYPE_UNDEF;

}

/** 
* Return 1 if originating neuron is excitatory, -1 otherwise.
* @param[in] t  synapseType I to I, I to E, E to I, or E to E
* @return 1 or -1
*/
__device__ int synSign( synapseType t )
{
        switch ( t )
        {
        case II:
        case IE:
                return -1;
        case EI:
        case EE:
                return 1;
        }

        return 0;
}

