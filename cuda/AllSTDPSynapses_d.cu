#include "hip/hip_runtime.h"
/*
 * AllSTDPSynapses_d.cu
 *
 */

#include "AllSTDPSynapses.h"
#include "AllSpikingSynapses.h"
#include "GPUSpikingModel.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSTDPSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, const SimulationInfo *sim_info ) {
	allocSynapseDeviceStruct( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice     Reference to the allSynapses struct on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) {
	AllSTDPSynapses allSynapses;

	allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

	HANDLE_ERROR( hipMalloc( allSynapsesDevice, sizeof( AllSTDPSynapses ) ) );
	HANDLE_ERROR( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllSTDPSynapses ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::allocDeviceStruct( AllSTDPSynapses &allSynapses, int num_neurons, int maxSynapsesPerNeuron ) {
        AllSpikingSynapses::allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

        uint32_t max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.total_delayPost, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.delayQueuePost, max_total_synapses * sizeof( uint32_t ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.delayIdxPost, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.ldelayQueuePost, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.tauspost, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.tauspre, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.taupos, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.tauneg, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.STDPgap, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.Wex, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.Aneg, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.Apos, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.mupos, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.muneg, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.useFroemkeDanSTDP, max_total_synapses * sizeof( bool ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSTDPSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
	AllSTDPSynapses allSynapses;

	HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSTDPSynapses ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allSynapses );

	HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 */
void AllSTDPSynapses::deleteDeviceStruct( AllSTDPSynapses& allSynapses ) {
        HANDLE_ERROR( hipFree( allSynapses.total_delayPost ) );
        HANDLE_ERROR( hipFree( allSynapses.delayQueuePost ) );
        HANDLE_ERROR( hipFree( allSynapses.delayIdxPost ) );
        HANDLE_ERROR( hipFree( allSynapses.tauspost ) );
        HANDLE_ERROR( hipFree( allSynapses.tauspre ) );
        HANDLE_ERROR( hipFree( allSynapses.taupos ) );
        HANDLE_ERROR( hipFree( allSynapses.tauneg ) );
        HANDLE_ERROR( hipFree( allSynapses.STDPgap ) );
        HANDLE_ERROR( hipFree( allSynapses.Wex ) );
        HANDLE_ERROR( hipFree( allSynapses.Aneg ) );
        HANDLE_ERROR( hipFree( allSynapses.Apos ) );
        HANDLE_ERROR( hipFree( allSynapses.mupos ) );
        HANDLE_ERROR( hipFree( allSynapses.muneg ) );
        HANDLE_ERROR( hipFree( allSynapses.useFroemkeDanSTDP ) );

        AllSpikingSynapses::deleteDeviceStruct( allSynapses );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::copySynapseHostToDevice( void* allSynapsesDevice, const SimulationInfo *sim_info ) { // copy everything necessary
	copySynapseHostToDevice( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary
	AllSTDPSynapses allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSTDPSynapses ), hipMemcpyDeviceToHost ) );

	copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::copyHostToDevice( void* allSynapsesDevice, AllSTDPSynapses& allSynapses, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        AllSpikingSynapses::copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );

        uint32_t max_total_synapses = maxSynapsesPerNeuron * num_neurons;
        
        HANDLE_ERROR( hipMemcpy ( allSynapses.total_delayPost, total_delayPost,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.delayQueuePost, delayQueuePost,
                max_total_synapses * sizeof( uint32_t ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.delayIdxPost, delayIdxPost,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.ldelayQueuePost, ldelayQueuePost,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.tauspost, tauspost,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.tauspre, tauspre,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.taupos, taupos,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.tauneg, tauneg,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.STDPgap, STDPgap,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.Wex, Wex,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.Aneg, Aneg,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.Apos, Apos,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.mupos, mupos,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.muneg, muneg,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.useFroemkeDanSTDP, useFroemkeDanSTDP,
                max_total_synapses * sizeof( bool ), hipMemcpyHostToDevice ) ); 
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSTDPSynapses::copySynapseDeviceToHost( void* allSynapsesDevice, const SimulationInfo *sim_info ) {
	// copy everything necessary
	AllSTDPSynapses allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSTDPSynapses ), hipMemcpyDeviceToHost ) );

	copyDeviceToHost( allSynapses, sim_info );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSTDPSynapses::copyDeviceToHost( AllSTDPSynapses& allSynapses, const SimulationInfo *sim_info ) {
        AllSpikingSynapses::copyDeviceToHost( allSynapses, sim_info ) ;

	int num_neurons = sim_info->totalNeurons;
	uint32_t max_total_synapses = sim_info->maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMemcpy ( delayQueuePost, allSynapses.delayQueuePost,
                max_total_synapses * sizeof( uint32_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayIdxPost, allSynapses.delayIdxPost,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( ldelayQueuePost, allSynapses.ldelayQueuePost,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauspost, allSynapses.tauspost,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauspre, allSynapses.tauspre,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( taupos, allSynapses.taupos,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauneg, allSynapses.tauneg,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( STDPgap, allSynapses.STDPgap,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( Wex, allSynapses.Wex,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( Aneg, allSynapses.Aneg,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( Apos, allSynapses.Apos,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( mupos, allSynapses.mupos,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( muneg, allSynapses.muneg,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( useFroemkeDanSTDP, allSynapses.useFroemkeDanSTDP,
                max_total_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
}

__device__ fpCreateSynapse_t fpCreateSTDPSynapse_d = (fpCreateSynapse_t)createSTDPSynapse;

/*
 *  Get a pointer to the device function createSTDPSynapse.
 *  The function will be called from updateSynapsesWeightsDevice device function.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *
 *  @param  fpCreateSynapse_h     Reference to the memory location 
 *                                where the function pointer will be set.
 */
void AllSTDPSynapses::getFpCreateSynapse(fpCreateSynapse_t& fpCreateSynapse_h)
{
    HANDLE_ERROR( hipMemcpyFromSymbol(&fpCreateSynapse_h, HIP_SYMBOL(fpCreateSTDPSynapse_d), sizeof(fpCreateSynapse_t)) );
}

/*
 *  Advance all the Synapses in the simulation.
 *  Update the state of all synapses for a time step.
 *
 *  @param  allSynapsesDevice      Reference to the allSynapses struct on device memory.
 *  @param  allNeuronsDevice       Reference to the allNeurons struct on device memory.
 *  @param  synapseIndexMapDevice  Reference to the SynapseIndexMap on device memory.
 *  @param  sim_info               SimulationInfo class to read information from.
 */
void AllSTDPSynapses::advanceSynapses(IAllSynapses* allSynapsesDevice, IAllNeurons* allNeuronsDevice, void* synapseIndexMapDevice, const SimulationInfo *sim_info)
{
    int max_spikes = (int) ((sim_info->epochDuration * sim_info->maxFiringRate));

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( total_synapse_counts + threadsPerBlock - 1 ) / threadsPerBlock;
    // Advance synapses ------------->
    advanceSTDPSynapsesDevice <<< blocksPerGrid, threadsPerBlock >>> ( total_synapse_counts, (SynapseIndexMap*)synapseIndexMapDevice, g_simulationStep, sim_info->deltaT, (AllSTDPSynapses*)allSynapsesDevice, (void (*)(AllSTDPSynapses*, const uint32_t, const uint64_t, const BGFLOAT))m_fpChangePSR_h, (AllSpikingNeurons*)allNeuronsDevice, max_spikes, sim_info->width );
}

__device__ fpPostSynapsesSpikeHit_t fpPostSTDPSynapsesSpikeHit_d = (fpPostSynapsesSpikeHit_t)postSTDPSynapseSpikeHitDevice;

/*
 *  Get a pointer to the device function ostSpikeHit.
 *  The function will be called from advanceNeuronsDevice device function.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *
 *  @param  fpostSpikeHit_h       Reference to the memory location
 *                                where the function pointer will be set.
 */
void AllSTDPSynapses::getFpPostSpikeHit(fpPostSynapsesSpikeHit_t& fpPostSpikeHit_h)
{
    HANDLE_ERROR( hipMemcpyFromSymbol(&fpPostSpikeHit_h, HIP_SYMBOL(fpPostSTDPSynapsesSpikeHit_d), sizeof(fpPostSynapsesSpikeHit_t)) );
}

/* ------------------*\
|* # Global Functions
\* ------------------*/

/*
 *  CUDA code for advancing STDP synapses.
 *  Perform updating synapses for one time step.
 *
 *  @param[in] total_synapse_counts  Number of synapses.
 *  @param  synapseIndexMapDevice    Reference to the SynapseIndexMap on device memory.
 *  @param[in] simulationStep        The current simulation step.
 *  @param[in] deltaT                Inner simulation step duration.
 *  @param[in] allSynapsesDevice     Pointer to Synapse structures in device memory.
 *  @param[in] fpChangePSR           Pointer to the device function changePSR() function.
 */
__global__ void advanceSTDPSynapsesDevice ( int total_synapse_counts, SynapseIndexMap* synapseIndexMapDevice, uint64_t simulationStep, const BGFLOAT deltaT, AllSTDPSynapses* allSynapsesDevice, void (*fpChangePSR)(AllSTDPSynapses*, const uint32_t, const uint64_t, const BGFLOAT), AllSpikingNeurons* allNeuronsDevice, int max_spikes, int width ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= total_synapse_counts )
            return;

    uint32_t iSyn = synapseIndexMapDevice->activeSynapseIndex[idx];

    BGFLOAT &decay = allSynapsesDevice->decay[iSyn];
    BGFLOAT &psr = allSynapsesDevice->psr[iSyn];

    // is an input in the queue?
    bool fPre = isSpikingSynapsesSpikeQueueDevice(allSynapsesDevice, iSyn);
    bool fPost = isSTDPSynapseSpikeQueuePostDevice(allSynapsesDevice, iSyn);
    if (fPre || fPost) {
        BGFLOAT &tauspre = allSynapsesDevice->tauspre[iSyn];
        BGFLOAT &tauspost = allSynapsesDevice->tauspost[iSyn];
        BGFLOAT &taupos = allSynapsesDevice->taupos[iSyn];
        BGFLOAT &tauneg = allSynapsesDevice->tauneg[iSyn];
        int &total_delay = allSynapsesDevice->total_delay[iSyn];
        bool &useFroemkeDanSTDP = allSynapsesDevice->useFroemkeDanSTDP[iSyn];

        // pre and post neurons index
        int idxPre = allSynapsesDevice->sourceNeuronIndex[iSyn];
        int idxPost = allSynapsesDevice->destNeuronIndex[iSyn];
        int64_t spikeHistory, spikeHistory2;
        BGFLOAT delta;
        BGFLOAT epre, epost;

        if (fPre) {     // preSpikeHit
            // spikeCount points to the next available position of spike_history,
            // so the getSpikeHistory w/offset = -2 will return the spike time 
            // just one before the last spike.
            spikeHistory = getSTDPSynapseSpikeHistoryDevice(allNeuronsDevice, idxPre, -2, max_spikes);
            if (spikeHistory > 0 && useFroemkeDanSTDP) {
                // delta will include the transmission delay
                delta = ((int64_t)simulationStep - spikeHistory) * deltaT;
                epre = 1.0 - exp(-delta / tauspre);
            } else {
                epre = 1.0;
            }

            // call the learning function stdpLearning() for each pair of
            // pre-post spikes
            int offIndex = -1;	// last spike
            while (true) {
                spikeHistory = getSTDPSynapseSpikeHistoryDevice(allNeuronsDevice, idxPost, offIndex, max_spikes);
                if (spikeHistory == ULONG_MAX)
                    break;
                // delta is the spike interval between pre-post spikes
                delta = (spikeHistory - (int64_t)simulationStep) * deltaT;

                DEBUG_SYNAPSE(
                    printf("advanceSTDPSynapsesDevice: fPre\n");
                    printf("          iSyn: %d\n", iSyn);
                    printf("          idxPre: %d\n", idxPre);
                    printf("          idxPost: %d\n", idxPost);
                    printf("          spikeHistory: %d\n", spikeHistory);
                    printf("          simulationStep: %d\n", simulationStep);
                    printf("          delta: %f\n\n", delta);
                );

                if (delta <= -3.0 * tauneg)
                    break;
                if (useFroemkeDanSTDP) {
                    spikeHistory2 = getSTDPSynapseSpikeHistoryDevice(allNeuronsDevice, idxPost, offIndex-1, max_spikes);
                    if (spikeHistory2 == ULONG_MAX)
                        break;
                    epost = 1.0 - exp(-((spikeHistory - spikeHistory2) * deltaT) / tauspost);
                } else {
                    epost = 1.0;
                }
                stdpLearningDevice(allSynapsesDevice, iSyn, delta, epost, epre);
                --offIndex;
            }

            fpChangePSR(allSynapsesDevice, iSyn, simulationStep, deltaT);
        }

        if (fPost) {    // postSpikeHit
            // spikeCount points to the next available position of spike_history,
            // so the getSpikeHistory w/offset = -2 will return the spike time
            // just one before the last spike.
            spikeHistory = getSTDPSynapseSpikeHistoryDevice(allNeuronsDevice, idxPost, -2, max_spikes);
            if (spikeHistory > 0 && useFroemkeDanSTDP) {
                // delta will include the transmission delay
                delta = ((int64_t)simulationStep - spikeHistory) * deltaT;
                epost = 1.0 - exp(-delta / tauspost);
            } else {
                epost = 1.0;
            }

            // call the learning function stdpLearning() for each pair of
            // post-pre spikes
            int offIndex = -1;	// last spike
            while (true) {
                spikeHistory = getSTDPSynapseSpikeHistoryDevice(allNeuronsDevice, idxPre, offIndex, max_spikes);
                if (spikeHistory == ULONG_MAX)
                    break;
                // delta is the spike interval between post-pre spikes
                delta = ((int64_t)simulationStep - spikeHistory - total_delay) * deltaT;

                DEBUG_SYNAPSE(
                    printf("advanceSTDPSynapsesDevice: fPost\n");
                    printf("          iSyn: %d\n", iSyn);
                    printf("          idxPre: %d\n", idxPre);
                    printf("          idxPost: %d\n", idxPost);
                    printf("          spikeHistory: %d\n", spikeHistory);
                    printf("          simulationStep: %d\n", simulationStep);
                    printf("          delta: %f\n\n", delta);
                );

                if (delta <= 0 || delta >= 3.0 * taupos)
                    break;
                if (useFroemkeDanSTDP) {
                    spikeHistory2 = getSTDPSynapseSpikeHistoryDevice(allNeuronsDevice, idxPre, offIndex-1, max_spikes);
                    if (spikeHistory2 == ULONG_MAX)
                        break;
                    epre = 1.0 - exp(-((spikeHistory - spikeHistory2) * deltaT) / tauspre);
                } else {
                    epre = 1.0;
                }
                stdpLearningDevice(allSynapsesDevice, iSyn, delta, epost, epre);
                --offIndex;
            }
        }
    }

    // decay the post spike response
    psr *= decay;
}

/* ------------------*\
|* # Device Functions
\* ------------------*/

/*
 *  Create a Synapse and connect it to the model.
 *
 *  @param allSynapsesDevice    Pointer to the Synapse structures in device memory.
 *  @param neuron_index         Index of the source neuron.
 *  @param synapse_index        Index of the Synapse to create.
 *  @param source_x             X location of source.
 *  @param source_y             Y location of source.
 *  @param dest_x               X location of destination.
 *  @param dest_y               Y location of destination.
 *  @param sum_point            Pointer to the summation point.
 *  @param deltaT               The time step size.
 *  @param type                 Type of the Synapse to create.
 */
__device__ void createSTDPSynapse(AllSTDPSynapses* allSynapsesDevice, const int neuron_index, const int synapse_index, int source_index, int dest_index, BGFLOAT *sum_point, const BGFLOAT deltaT, synapseType type)
{
    BGFLOAT delay;
    size_t max_synapses = allSynapsesDevice->maxSynapsesPerNeuron;
    uint32_t iSyn = max_synapses * neuron_index + synapse_index;

    allSynapsesDevice->in_use[iSyn] = true;
    allSynapsesDevice->summationPoint[iSyn] = sum_point;
    allSynapsesDevice->destNeuronIndex[iSyn] = dest_index;
    allSynapsesDevice->sourceNeuronIndex[iSyn] = source_index;
    allSynapsesDevice->W[iSyn] = synSign(type) * 10.0e-9;

    allSynapsesDevice->delayQueue[iSyn] = 0;
    allSynapsesDevice->delayIdx[iSyn] = 0;
    allSynapsesDevice->ldelayQueue[iSyn] = LENGTH_OF_DELAYQUEUE;

    allSynapsesDevice->psr[iSyn] = 0.0;
    allSynapsesDevice->type[iSyn] = type;

    allSynapsesDevice->tau[iSyn] = DEFAULT_tau;

    BGFLOAT tau;
    switch (type) {
        case II:
            tau = 6e-3;
            delay = 0.8e-3;
            break;
        case IE:
            tau = 6e-3;
            delay = 0.8e-3;
            break;
        case EI:
            tau = 3e-3;
            delay = 0.8e-3;
            break;
        case EE:
            tau = 3e-3;
            delay = 1.5e-3;
            break;
        default:
            break;
    }

    allSynapsesDevice->tau[iSyn] = tau;
    allSynapsesDevice->decay[iSyn] = exp( -deltaT / tau );
    allSynapsesDevice->total_delay[iSyn] = static_cast<int>( delay / deltaT ) + 1;

    size_t size = allSynapsesDevice->total_delay[iSyn] / ( sizeof(uint8_t) * 8 ) + 1;
    assert( size <= BYTES_OF_DELAYQUEUE );

    allSynapsesDevice->Apos[iSyn] = 0.5;
    allSynapsesDevice->Aneg[iSyn] = -0.5;
    allSynapsesDevice->STDPgap[iSyn] = 2e-3;

    allSynapsesDevice->total_delayPost[iSyn] = 0;

    allSynapsesDevice->tauspost[iSyn] = 0;
    allSynapsesDevice->tauspre[iSyn] = 0;

    allSynapsesDevice->taupos[iSyn] = 15e-3;
    allSynapsesDevice->tauneg[iSyn] = 35e-3;
    allSynapsesDevice->Wex[iSyn] = 1.0;

    allSynapsesDevice->mupos[iSyn] = 0;
    allSynapsesDevice->muneg[iSyn] = 0;

    allSynapsesDevice->useFroemkeDanSTDP[iSyn] = false;
}

/*     
 *  Adjust synapse weight according to the Spike-timing-dependent synaptic modification
 *  induced by natural spike trains
 *
 *  @param  allSynapsesDevice    Pointer to the Synapse structures in device memory.
 *  @param  iSyn                 Index of the synapse to set.
 *  @param  delta                Pre/post synaptic spike interval.
 *  @param  epost                Params for the rule given in Froemke and Dan (2002).
 *  @param  epre                 Params for the rule given in Froemke and Dan (2002).
 */
__device__ void stdpLearningDevice(AllSTDPSynapses* allSynapsesDevice, const uint32_t iSyn, double delta, double epost, double epre)
{
    BGFLOAT STDPgap = allSynapsesDevice->STDPgap[iSyn];
    BGFLOAT muneg = allSynapsesDevice->muneg[iSyn];
    BGFLOAT mupos = allSynapsesDevice->mupos[iSyn];
    BGFLOAT tauneg = allSynapsesDevice->tauneg[iSyn];
    BGFLOAT taupos = allSynapsesDevice->taupos[iSyn];
    BGFLOAT Aneg = allSynapsesDevice->Aneg[iSyn];
    BGFLOAT Apos = allSynapsesDevice->Apos[iSyn];
    BGFLOAT Wex = allSynapsesDevice->Wex[iSyn];
    BGFLOAT &W = allSynapsesDevice->W[iSyn];
    BGFLOAT dw;

    if (delta < -STDPgap) {
        // Depression
        dw = pow(W, muneg) * Aneg * exp(delta / tauneg);
    } else if (delta > STDPgap) {
        // Potentiation
        dw = pow(Wex - W, mupos) * Apos * exp(-delta / taupos);
    } else {
        return;
    }

    W += epost * epre * dw;

    // check the sign
    if ((Wex < 0 && W > 0) || (Wex > 0 && W < 0)) W = 0;

    // check for greater Wmax
    if (fabs(W) > fabs(Wex)) W = Wex;

    DEBUG_SYNAPSE(
        printf("AllSTDPSynapses::stdpLearning:\n");
        printf("          iSyn: %d\n", iSyn);
        printf("          delta: %f\n", delta);
        printf("          epre: %f\n", epre);
        printf("          epost: %f\n", epost);
        printf("          dw: %f\n", dw);
        printf("          W: %f\n\n", W);
    );
}

/*
 *  Checks if there is an input spike in the queue.
 *
 *  @param[in] allSynapsesDevice     Pointer to Synapse structures in device memory.
 *  @param[in] iSyn                  Index of the Synapse to check.
 *  @return true if there is an input spike event.
 */
__device__ bool isSTDPSynapseSpikeQueuePostDevice(AllSTDPSynapses* allSynapsesDevice, uint32_t iSyn)
{
    uint32_t &delay_queue = allSynapsesDevice->delayQueuePost[iSyn];
    int &delayIdx = allSynapsesDevice->delayIdxPost[iSyn];
    int ldelayQueue = allSynapsesDevice->ldelayQueuePost[iSyn];

    uint32_t delayMask = (0x1 << delayIdx);
    bool isFired = delay_queue & (delayMask);
    delay_queue &= ~(delayMask);
    if ( ++delayIdx >= ldelayQueue ) {
            delayIdx = 0;
    }

    return isFired;
}

/*
 *  Gets the spike history of the neuron.
 *
 *  @param  allNeuronsDevice       Reference to the allNeurons struct on device memory. 
 *  @param  index                  Index of the neuron to get spike history.
 *  @param  offIndex               Offset of the history beffer to get.
 *                                 -1 will return the last spike.
 *  @param  max_spikes             Maximum number of spikes per neuron per epoch.
 *  @return Spike history.
 */
__device__ uint64_t getSTDPSynapseSpikeHistoryDevice(AllSpikingNeurons* allNeuronsDevice, int index, int offIndex, int max_spikes)
{
    // offIndex is a minus offset
    int idxSp = (allNeuronsDevice->spikeCount[index] + allNeuronsDevice->spikeCountOffset[index] +  max_spikes + offIndex) % max_spikes;
    return allNeuronsDevice->spike_history[index][idxSp];
}

/*
 *  Prepares Synapse for a spike hit (for back propagation).
 *
 *  @param[in] iSyn                  Index of the Synapse to update.
 *  @param[in] allSynapsesDevice     Pointer to Synapse structures in device memory.
 */
__device__ void postSTDPSynapseSpikeHitDevice( const uint32_t iSyn, AllSTDPSynapses* allSynapsesDevice ) {
        uint32_t &delay_queue = allSynapsesDevice->delayQueuePost[iSyn];
        int delayIdx = allSynapsesDevice->delayIdxPost[iSyn];
        int ldelayQueue = allSynapsesDevice->ldelayQueuePost[iSyn];
        int total_delay = allSynapsesDevice->total_delayPost[iSyn];

        // Add to spike queue

        // calculate index where to insert the spike into delayQueue
        int idx = delayIdx +  total_delay;
        if ( idx >= ldelayQueue ) {
                idx -= ldelayQueue;
        }

        // set a spike
        //assert( !(delay_queue[0] & (0x1 << idx)) );
        delay_queue |= (0x1 << idx);
}
