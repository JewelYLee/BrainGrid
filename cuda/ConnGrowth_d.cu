#include "hip/hip_runtime.h"
#include "ConnGrowth.h"
#include "AllSpikingSynapses.h"
#include "Book.h"

/*
 *  Update the weight of the Synapses in the simulation.
 *  GETDONE: Figure out what is going on with the type map
 *  Note: Platform Dependent.
 *
 *  @param  num_neurons         number of neurons to update.
 *  @param  neurons             the Neuron list to search from.
 *  @param  synapses            the Synapse list to search from.
 *  @param  sim_info            SimulationInfo to refer from.
 *  @param  m_allNeuronsDevice  Reference to the allNeurons struct on device memory. 
 *  @param  m_allSynapsesDevice Reference to the allSynapses struct on device memory.
 *  @param  layout              Layout information of the neunal network.
 */
void ConnGrowth::updateSynapsesWeights(const int num_neurons, IAllNeurons &neurons, IAllSynapses &synapses, const SimulationInfo *sim_info, AllSpikingNeurons** m_allNeuronsDevice, AllSpikingSynapses** m_allSynapsesDevice, Layout *layout)
{
   // For now, we just set the weights to equal the areas. We will later
   // scale it and set its sign (when we index and get its sign).
   (*W) = (*area);

   BGFLOAT deltaT = sim_info->deltaT;

   // CUDA parameters
   const int threadsPerBlock = 256;
   int blocksPerGrid;

   size_t W_h_size = sim_info->totalNeurons * sim_info->totalNeurons * sizeof (BGFLOAT);
   BGFLOAT* W_h = new BGFLOAT[W_h_size];
   
   // copy weight data to the device memory
   for ( int i = 0 ; i < sim_info->totalNeurons; i++ ){
      for ( int j = 0; j < sim_info->totalNeurons; j++ ){
         W_h[i * sim_info->totalNeurons + j] = (*W)(i, j);
      }
   }
   
   BGFLOAT* W_current = W_h;
   
   // allocate device memories
   for(int i = 0; i < sim_info->numGPU; i++){
      hipSetDevice(i);
      size_t W_elements = sim_info->individualGPUInfo[i].totalNeurons * sim_info->individualGPUInfo[i].totalNeurons;
      size_t W_d_size =  W_elements * sizeof (BGFLOAT);
      BGFLOAT* W_d;
      HANDLE_ERROR( hipMalloc ( ( void ** ) &W_d, W_d_size ) );

      neuronType* neuron_type_map_d;
      HANDLE_ERROR( hipMalloc( ( void ** ) &neuron_type_map_d, sim_info->totalNeurons * sizeof( neuronType ) ) );

      HANDLE_ERROR( hipMemcpy ( W_d, W_current, W_d_size, hipMemcpyHostToDevice ) );
      W_current += W_elements;

      HANDLE_ERROR( hipMemcpy ( neuron_type_map_d, layout->neuron_type_map, sim_info->totalNeurons * sizeof( neuronType ), hipMemcpyHostToDevice ) );

      unsigned long long fpCreateSynapse_h;
      synapses.getFpCreateSynapse(fpCreateSynapse_h);

      blocksPerGrid = ( sim_info->totalNeurons + threadsPerBlock - 1 ) / threadsPerBlock;
      updateSynapsesWeightsDevice <<< blocksPerGrid, threadsPerBlock >>> ( sim_info->individualGPUInfo[i].totalNeurons, deltaT, W_d, sim_info->maxSynapsesPerNeuron, m_allNeuronsDevice[i], m_allSynapsesDevice[i], (void (*)(AllSpikingSynapses*, const int, const int, int, int, BGFLOAT*, const BGFLOAT, synapseType))fpCreateSynapse_h, neuron_type_map_d );
      
      // free memories
      HANDLE_ERROR( hipFree( W_d ) );
      HANDLE_ERROR( hipFree( neuron_type_map_d ) );

   }
   
   // free memories
   delete[] W_h;

   // copy device synapse count to host memory
   synapses.copyDeviceSynapseCountsToHost(m_allSynapsesDevice, sim_info);
   // copy device synapse summation coordinate to host memory
   synapses.copyDeviceSynapseSumIdxToHost(m_allSynapsesDevice, sim_info);
}

/*
 * Adjust the strength of the synapse or remove it from the synapse map if it has gone below
 * zero.
 * //GETDONE Figure out how to piece out this function for the simulation.
 * @param[in] num_neurons        Number of neurons.
 * @param[in] deltaT             The time step size.
 * @param[in] W_d                Array of synapse weight.
 * @param[in] maxSynapses        Maximum number of synapses per neuron.
 * @param[in] allNeuronsDevice   Pointer to the Neuron structures in device memory.
 * @param[in] allSynapsesDevice  Pointer to the Synapse structures in device memory.
 * @param[in] fpCreateSynapse    Function pointer to the createSynapse device function.
 */
__global__ void updateSynapsesWeightsDevice( int num_neurons, BGFLOAT deltaT, BGFLOAT* W_d, int maxSynapses, AllSpikingNeurons* allNeuronsDevice, AllSpikingSynapses* allSynapsesDevice, void (*fpCreateSynapse)(AllSpikingSynapses*, const int, const int, int, int, BGFLOAT*, const BGFLOAT, synapseType), neuronType* neuron_type_map_d )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= num_neurons )
        return;

    int adjusted = 0;
    //int could_have_been_removed = 0; // TODO: use this value
    int removed = 0;
    int added = 0;

    // Scale and add sign to the areas
    // visit each neuron 'a'
    int src_neuron = idx;

    // and each destination neuron 'b'
    for (int dest_neuron = 0; dest_neuron < num_neurons; dest_neuron++) {
        // visit each synapse at (xa,ya)
        bool connected = false;
        //GETDONE: figure exactly what this synType comes from
        synapseType type = synType(neuron_type_map_d, src_neuron, dest_neuron);

        // for each existing synapse
        size_t synapse_counts = allSynapsesDevice->synapse_counts[src_neuron];
        int synapse_adjusted = 0;
        for (size_t synapse_index = 0; synapse_adjusted < synapse_counts; synapse_index++) {
            uint32_t iSyn = maxSynapses * src_neuron + synapse_index;
            if (allSynapsesDevice->in_use[iSyn] == true) {
                // if there is a synapse between a and b
                if (allSynapsesDevice->destNeuronIndex[iSyn] == dest_neuron) {
                    connected = true;
                    adjusted++;

                    // adjust the strength of the synapse or remove
                    // it from the synapse map if it has gone below
                    // zero.
                    if (W_d[src_neuron * num_neurons + dest_neuron] < 0) {
                        removed++;
                        eraseSpikingSynapse(allSynapsesDevice, src_neuron, synapse_index, maxSynapses);
                    } else {
                        // adjust
                        // g_synapseStrengthAdjustmentConstant is 1.0e-8;
                        allSynapsesDevice->W[iSyn] = W_d[src_neuron * num_neurons
                            + dest_neuron] * synSign(type) * AllSynapses::SYNAPSE_STRENGTH_ADJUSTMENT;
                    }
                }
                synapse_adjusted++;
            }
        }

        // if not connected and weight(a,b) > 0, add a new synapse from a to b
        if (!connected && (W_d[src_neuron * num_neurons +  dest_neuron] > 0)) {
            // locate summation point
            BGFLOAT* sum_point = &( allNeuronsDevice->summation_map[dest_neuron] );
            added++;

            addSpikingSynapse(allSynapsesDevice, type, src_neuron, dest_neuron, src_neuron, dest_neuron, sum_point, deltaT, W_d, num_neurons, fpCreateSynapse);
        }
    }
}
