#include "hip/hip_runtime.h"
/*
 * AllDSSynapses_d.cu
 *
 */

#include "AllDSSynapses.h"
#include "GPUSpikingModel.h"
#include "Book.h"

void AllDSSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, const SimulationInfo *sim_info ) {
	allocSynapseDeviceStruct( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );
}

void AllDSSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) {
	AllDSSynapses allSynapses;

	allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

	HANDLE_ERROR( hipMalloc( allSynapsesDevice, sizeof( AllDSSynapses ) ) );
	HANDLE_ERROR( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllDSSynapses ), hipMemcpyHostToDevice ) );
}

void AllDSSynapses::allocDeviceStruct( AllDSSynapses &allSynapses, int num_neurons, int maxSynapsesPerNeuron ) {
        uint32_t max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.destNeuronIndex, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.W, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.summationPoint, max_total_synapses * sizeof( BGFLOAT* ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.sourceNeuronIndex, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.psr, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.decay, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.total_delay, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.delayQueue, max_total_synapses * sizeof( uint32_t ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.delayIdx, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.ldelayQueue, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.type, max_total_synapses * sizeof( synapseType ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.tau, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.lastSpike, max_total_synapses * sizeof( uint64_t ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.in_use, max_total_synapses * sizeof( bool ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.synapse_counts, num_neurons * sizeof( size_t ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.r, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.u, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.D, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.U, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.F, max_total_synapses * sizeof( BGFLOAT ) ) );
}

void AllDSSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
	AllDSSynapses allSynapses;

	HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDSSynapses ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allSynapses );

	HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

void AllDSSynapses::deleteDeviceStruct( AllDSSynapses& allSynapses ) {
        HANDLE_ERROR( hipFree( allSynapses.destNeuronIndex ) );
        HANDLE_ERROR( hipFree( allSynapses.W ) );
        HANDLE_ERROR( hipFree( allSynapses.summationPoint ) );
        HANDLE_ERROR( hipFree( allSynapses.sourceNeuronIndex ) );
        HANDLE_ERROR( hipFree( allSynapses.psr ) );
        HANDLE_ERROR( hipFree( allSynapses.decay ) );
        HANDLE_ERROR( hipFree( allSynapses.total_delay ) );
        HANDLE_ERROR( hipFree( allSynapses.delayQueue ) );
        HANDLE_ERROR( hipFree( allSynapses.delayIdx ) );
        HANDLE_ERROR( hipFree( allSynapses.ldelayQueue ) );
        HANDLE_ERROR( hipFree( allSynapses.type ) );
        HANDLE_ERROR( hipFree( allSynapses.tau ) );
        HANDLE_ERROR( hipFree( allSynapses.lastSpike ) );
        HANDLE_ERROR( hipFree( allSynapses.in_use ) );
        HANDLE_ERROR( hipFree( allSynapses.synapse_counts ) );
	HANDLE_ERROR( hipFree( allSynapses.r ) );
	HANDLE_ERROR( hipFree( allSynapses.u ) );
	HANDLE_ERROR( hipFree( allSynapses.D ) );
	HANDLE_ERROR( hipFree( allSynapses.U ) );
	HANDLE_ERROR( hipFree( allSynapses.F ) );
}

void AllDSSynapses::copySynapseHostToDevice( void* allSynapsesDevice, const SimulationInfo *sim_info ) { // copy everything necessary
	copySynapseHostToDevice( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );	
}

void AllDSSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary
	AllDSSynapses allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDSSynapses ), hipMemcpyDeviceToHost ) );

	copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );	
}

void AllDSSynapses::copyHostToDevice( void* allSynapsesDevice, AllDSSynapses& allSynapses, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        uint32_t max_total_synapses = maxSynapsesPerNeuron * num_neurons;
        
        HANDLE_ERROR( hipMemcpy ( allSynapses.synapse_counts, synapse_counts,
                        num_neurons * sizeof( size_t ), hipMemcpyHostToDevice ) );
        allSynapses.maxSynapsesPerNeuron = maxSynapsesPerNeuron;
        allSynapses.total_synapse_counts = total_synapse_counts;
        HANDLE_ERROR( hipMemcpy ( allSynapsesDevice, &allSynapses, sizeof( AllSpikingSynapses ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.destNeuronIndex, destNeuronIndex, 
                max_total_synapses * sizeof( int ),  hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.W, W,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.sourceNeuronIndex, sourceNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.psr, psr,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.decay, decay,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.total_delay, total_delay,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.delayQueue, delayQueue,
                max_total_synapses * sizeof( uint32_t ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.delayIdx, delayIdx,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.ldelayQueue, ldelayQueue,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.type, type,
                max_total_synapses * sizeof( synapseType ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.tau, tau, 
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.lastSpike, lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.in_use, in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allSynapses.r, r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.u, u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.D, D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.U, U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.F, F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
}

void AllDSSynapses::copySynapseDeviceToHost( void* allSynapsesDevice, const SimulationInfo *sim_info ) {
	// copy everything necessary
	AllDSSynapses allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDSSynapses ), hipMemcpyDeviceToHost ) );

	copyDeviceToHost( allSynapses, sim_info );
}

void AllDSSynapses::copyDeviceToHost( AllDSSynapses& allSynapses, const SimulationInfo *sim_info ) {
	int num_neurons = sim_info->totalNeurons;
	uint32_t max_total_synapses = sim_info->maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMemcpy ( synapse_counts, allSynapses.synapse_counts,
                num_neurons * sizeof( size_t ), hipMemcpyDeviceToHost ) );
        maxSynapsesPerNeuron = allSynapses.maxSynapsesPerNeuron;
        total_synapse_counts = allSynapses.total_synapse_counts;

        HANDLE_ERROR( hipMemcpy ( destNeuronIndex, allSynapses.destNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( W, allSynapses.W,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( sourceNeuronIndex, allSynapses.sourceNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( psr, allSynapses.psr,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( decay, allSynapses.decay,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( total_delay, allSynapses.total_delay,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayQueue, allSynapses.delayQueue,
                max_total_synapses * sizeof( uint32_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayIdx, allSynapses.delayIdx,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( ldelayQueue, allSynapses.ldelayQueue,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( type, allSynapses.type,
                max_total_synapses * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tau, allSynapses.tau,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( lastSpike, allSynapses.lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( in_use, allSynapses.in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( r, allSynapses.r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( u, allSynapses.u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( D, allSynapses.D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( U, allSynapses.U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( F, allSynapses.F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

/**
 *  Get synapse_counts in AllSynapses struct on device memory.
 *  @param  sim_info    SimulationInfo to refer from.
 */
void AllDSSynapses::copyDeviceSynapseCountsToHost(void* allSynapsesDevice, const SimulationInfo *sim_info)
{
        AllDSSynapses allSynapses;
        int neuron_count = sim_info->totalNeurons;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDSSynapses ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( synapse_counts, allSynapses.synapse_counts, neuron_count * sizeof( size_t ), hipMemcpyDeviceToHost ) );
}

/** 
 *  Get summationCoord and in_use in AllSynapses struct on device memory.
 *  @param  sim_info    SimulationInfo to refer from.
 */
void AllDSSynapses::copyDeviceSynapseSumIdxToHost(void* allSynapsesDevice, const SimulationInfo *sim_info)
{
        AllDSSynapses allSynapses;
	uint32_t max_total_synapses = sim_info->maxSynapsesPerNeuron * sim_info->totalNeurons;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDSSynapses ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( destNeuronIndex, allSynapses.destNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( in_use, allSynapses.in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
}

void AllDSSynapses::getFpCreateSynapse(unsigned long long& fpCreateSynapse_h)
{
    unsigned long long *fpCreateSynapse_d;

    HANDLE_ERROR( hipMalloc(&fpCreateSynapse_d, sizeof(unsigned long long)) );

    getFpCreateSynapseDevice<<<1,1>>>((void (**)(AllDSSynapses*, const int, const int, int, int, BGFLOAT*, const BGFLOAT, synapseType))fpCreateSynapse_d);

    HANDLE_ERROR( hipMemcpy(&fpCreateSynapse_h, fpCreateSynapse_d, sizeof(unsigned long long), hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipFree( fpCreateSynapse_d ) );
}

void AllDSSynapses::getFpChangePSR(unsigned long long& fpChangePSR_h)
{
    unsigned long long *fpChangePSR_d;

    HANDLE_ERROR( hipMalloc(&fpChangePSR_d, sizeof(unsigned long long)) );

    getFpChangePSRDevice<<<1,1>>>((void (**)(AllDSSynapses*, const uint32_t, const uint64_t, const BGFLOAT))fpChangePSR_d);

    HANDLE_ERROR( hipMemcpy(&fpChangePSR_h, fpChangePSR_d, sizeof(unsigned long long), hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipFree( fpChangePSR_d ) );
}

/* ------------------*\
|* # Global Functions
\* ------------------*/

__global__ void getFpCreateSynapseDevice(void (**fpCreateSynapse_d)(AllDSSynapses*, const int, const int, int, int, BGFLOAT*, const BGFLOAT, synapseType))
{
    *fpCreateSynapse_d = createSynapse;
}

__global__ void getFpChangePSRDevice(void (**fpChangePSR_d)(AllDSSynapses*, const uint32_t, const uint64_t, const BGFLOAT))
{
    *fpChangePSR_d = changePSR;
}

/**
 *  Create a Synapse and connect it to the model.
 *  @param allSynapsesDevice    Pointer to the Synapse structures in device memory.
 *  @param neuron_index         Index of the source neuron.
 *  @param synapse_index        Index of the Synapse to create.
 *  @param source_x             X location of source.
 *  @param source_y             Y location of source.
 *  @param dest_x               X location of destination.
 *  @param dest_y               Y location of destination.
 *  @param sum_point            Pointer to the summation point.
 *  @param deltaT               The time step size.
 *  @param type                 Type of the Synapse to create.
 */
__device__ void createSynapse(AllDSSynapses* allSynapsesDevice, const int neuron_index, const int synapse_index, int source_index, int dest_index, BGFLOAT *sum_point, const BGFLOAT deltaT, synapseType type)
{
    BGFLOAT delay;
    size_t max_synapses = allSynapsesDevice->maxSynapsesPerNeuron;
    uint32_t iSyn = max_synapses * neuron_index + synapse_index;

    allSynapsesDevice->in_use[iSyn] = true;
    allSynapsesDevice->summationPoint[iSyn] = sum_point;
    allSynapsesDevice->destNeuronIndex[iSyn] = dest_index;
    allSynapsesDevice->sourceNeuronIndex[iSyn] = source_index;
    allSynapsesDevice->W[iSyn] = 10.0e-9;

    allSynapsesDevice->delayQueue[iSyn] = 0;
    allSynapsesDevice->delayIdx[iSyn] = 0;
    allSynapsesDevice->ldelayQueue[iSyn] = LENGTH_OF_DELAYQUEUE;

    allSynapsesDevice->psr[iSyn] = 0.0;
    allSynapsesDevice->r[iSyn] = 1.0;
    allSynapsesDevice->u[iSyn] = 0.4;     // DEFAULT_U
    allSynapsesDevice->lastSpike[iSyn] = ULONG_MAX;
    allSynapsesDevice->type[iSyn] = type;

    allSynapsesDevice->U[iSyn] = DEFAULT_U;
    allSynapsesDevice->tau[iSyn] = DEFAULT_tau;

    BGFLOAT U;
    BGFLOAT D;
    BGFLOAT F;
    BGFLOAT tau;
    switch (type) {
        case II:
            U = 0.32;
            D = 0.144;
            F = 0.06;
            tau = 6e-3;
            delay = 0.8e-3;
            break;
        case IE:
            U = 0.25;
            D = 0.7;
            F = 0.02;
            tau = 6e-3;
            delay = 0.8e-3;
            break;
        case EI:
            U = 0.05;
            D = 0.125;
            F = 1.2;
            tau = 3e-3;
            delay = 0.8e-3;
            break;
        case EE:
            U = 0.5;
            D = 1.1;
            F = 0.05;
            tau = 3e-3;
            delay = 1.5e-3;
            break;
        default:
            break;
    }

    allSynapsesDevice->U[iSyn] = U;
    allSynapsesDevice->D[iSyn] = D;
    allSynapsesDevice->F[iSyn] = F;

    allSynapsesDevice->tau[iSyn] = tau;
    allSynapsesDevice->decay[iSyn] = exp( -deltaT / tau );
    allSynapsesDevice->total_delay[iSyn] = static_cast<int>( delay / deltaT ) + 1;

    size_t size = allSynapsesDevice->total_delay[iSyn] / ( sizeof(uint8_t) * 8 ) + 1;
    assert( size <= BYTES_OF_DELAYQUEUE );
}

__device__ void changePSR(AllDSSynapses* allSynapsesDevice, const uint32_t iSyn, const uint64_t simulationStep, const BGFLOAT deltaT)
{
    uint64_t &lastSpike = allSynapsesDevice->lastSpike[iSyn];
    BGFLOAT &r = allSynapsesDevice->r[iSyn];
    BGFLOAT &u = allSynapsesDevice->u[iSyn];
    BGFLOAT D = allSynapsesDevice->D[iSyn];
    BGFLOAT F = allSynapsesDevice->F[iSyn];
    BGFLOAT U = allSynapsesDevice->U[iSyn];
    BGFLOAT W = allSynapsesDevice->W[iSyn];
    BGFLOAT &psr = allSynapsesDevice->psr[iSyn];
    BGFLOAT decay = allSynapsesDevice->decay[iSyn];

    // adjust synapse parameters
    if (lastSpike != ULONG_MAX) {
            BGFLOAT isi = (simulationStep - lastSpike) * deltaT ;
            r = 1 + ( r * ( 1 - u ) - 1 ) * exp( -isi / D );
            u = U + u * ( 1 - U ) * exp( -isi / F );
    }
    psr += ( ( W / decay ) * u * r );// calculate psr
    lastSpike = simulationStep; // record the time of the spike
}
