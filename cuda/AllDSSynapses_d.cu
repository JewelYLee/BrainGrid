/*
 * AllDSSynapses_d.cu
 *
 */

#include "AllDSSynapses.h"
#include "Book.h"

void AllDSSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, const SimulationInfo *sim_info ) {
	int num_neurons = sim_info->totalNeurons;
	int max_synapses = sim_info->maxSynapsesPerNeuron;

	allocSynapseDeviceStruct( allSynapsesDevice, num_neurons, max_synapses );
}

void AllDSSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int num_neurons, int max_synapses ) {
	AllDSSynapses allSynapses;
	uint32_t max_total_synapses = max_synapses * num_neurons;

	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.summationCoord, max_total_synapses * sizeof( Coordinate ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.W, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.summationPoint, max_total_synapses * sizeof( BGFLOAT* ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.synapseCoord, max_total_synapses * sizeof( Coordinate ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.psr, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.decay, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.total_delay, max_total_synapses * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.delayQueue, max_total_synapses * sizeof( uint32_t ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.delayIdx, max_total_synapses * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.ldelayQueue, max_total_synapses * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.type, max_total_synapses * sizeof( synapseType ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.tau, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.r, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.u, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.D, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.U, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.F, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.lastSpike, max_total_synapses * sizeof( uint64_t ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.in_use, max_total_synapses * sizeof( bool ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.synapse_counts, num_neurons * sizeof( size_t ) ) );

	HANDLE_ERROR( hipMalloc( allSynapsesDevice, sizeof( AllDSSynapses ) ) );
	HANDLE_ERROR( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllDSSynapses ), hipMemcpyHostToDevice ) );
}

void AllDSSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice, const SimulationInfo *sim_info ) {
        int num_neurons = sim_info->totalNeurons;
        int max_synapses = sim_info->maxSynapsesPerNeuron;

	deleteSynapseDeviceStruct( allSynapsesDevice, num_neurons, max_synapses );
}

void AllDSSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice, int num_neurons, int max_synapses ) {
	AllDSSynapses allSynapses;

	HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDSSynapses ), hipMemcpyDeviceToHost ) );

	HANDLE_ERROR( hipFree( allSynapses.summationCoord ) );
	HANDLE_ERROR( hipFree( allSynapses.W ) );
	HANDLE_ERROR( hipFree( allSynapses.summationPoint ) );
	HANDLE_ERROR( hipFree( allSynapses.synapseCoord ) );
	HANDLE_ERROR( hipFree( allSynapses.psr ) );
	HANDLE_ERROR( hipFree( allSynapses.decay ) );
	HANDLE_ERROR( hipFree( allSynapses.total_delay ) );
	HANDLE_ERROR( hipFree( allSynapses.delayQueue ) );
	HANDLE_ERROR( hipFree( allSynapses.delayIdx ) );
	HANDLE_ERROR( hipFree( allSynapses.ldelayQueue ) );
	HANDLE_ERROR( hipFree( allSynapses.type ) );
	HANDLE_ERROR( hipFree( allSynapses.tau ) );
	HANDLE_ERROR( hipFree( allSynapses.r ) );
	HANDLE_ERROR( hipFree( allSynapses.u ) );
	HANDLE_ERROR( hipFree( allSynapses.D ) );
	HANDLE_ERROR( hipFree( allSynapses.U ) );
	HANDLE_ERROR( hipFree( allSynapses.F ) );
	HANDLE_ERROR( hipFree( allSynapses.lastSpike ) );
	HANDLE_ERROR( hipFree( allSynapses.in_use ) );
	HANDLE_ERROR( hipFree( allSynapses.synapse_counts ) );

	HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

void AllDSSynapses::copySynapseHostToDevice( void* allSynapsesDevice, const SimulationInfo *sim_info ) { // copy everything necessary
	int num_neurons = sim_info->totalNeurons;
	int max_synapses =  sim_info->maxSynapsesPerNeuron;

	copySynapseHostToDevice( allSynapsesDevice, num_neurons, max_synapses );	
}

void AllDSSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int num_neurons, int max_synapses ) { // copy everything necessary
	uint32_t max_total_synapses = max_synapses * num_neurons;
	AllDSSynapses allSynapses_0;

        HANDLE_ERROR( hipMemcpy ( &allSynapses_0, allSynapsesDevice, sizeof( AllDSSynapses ), hipMemcpyDeviceToHost ) );

	HANDLE_ERROR( hipMemcpy ( allSynapses_0.synapse_counts, synapse_counts, 
			num_neurons * sizeof( size_t ), hipMemcpyHostToDevice ) );
	allSynapses_0.maxSynapsesPerNeuron = maxSynapsesPerNeuron;	
	allSynapses_0.total_synapse_counts = total_synapse_counts;	
	HANDLE_ERROR( hipMemcpy ( allSynapsesDevice, &allSynapses_0, sizeof( AllDSSynapses ), hipMemcpyHostToDevice ) );

        HANDLE_ERROR( hipMemcpy ( allSynapses_0.summationCoord, summationCoord,
                max_total_synapses * sizeof( Coordinate ),  hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.W, allSynapses_0.W,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.synapseCoord, synapseCoord,
                max_total_synapses * sizeof( Coordinate ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.psr, allSynapses_0.psr,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.decay, decay,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.total_delay, total_delay,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.delayQueue, delayQueue,
                max_total_synapses * sizeof( uint32_t ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.delayIdx, delayIdx,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.ldelayQueue, ldelayQueue,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.type, type,
                max_total_synapses * sizeof( synapseType ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.tau, tau,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.r, r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.u, u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.D, D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.U, U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.F, F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.lastSpike, lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses_0.in_use, in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyHostToDevice ) );
}

void AllDSSynapses::copySynapseDeviceToHost( void* allSynapsesDevice, const SimulationInfo *sim_info ) {
	// copy everything necessary
	AllDSSynapses allSynapses_0;
	int num_neurons = sim_info->totalNeurons;
	int max_synapses = sim_info->maxSynapsesPerNeuron;
	uint32_t max_total_synapses = max_synapses * num_neurons;

        HANDLE_ERROR( hipMemcpy ( &allSynapses_0, allSynapsesDevice, sizeof( AllDSSynapses ), hipMemcpyDeviceToHost ) );

	HANDLE_ERROR( hipMemcpy ( synapse_counts, allSynapses_0.synapse_counts, 
		num_neurons * sizeof( size_t ), hipMemcpyDeviceToHost ) );
	maxSynapsesPerNeuron = allSynapses_0.maxSynapsesPerNeuron;
	total_synapse_counts = allSynapses_0.total_synapse_counts;

        HANDLE_ERROR( hipMemcpy ( summationCoord, allSynapses_0.summationCoord,
                max_total_synapses * sizeof( Coordinate ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( W, allSynapses_0.W,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( synapseCoord, allSynapses_0.synapseCoord,
                max_total_synapses * sizeof( Coordinate ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( psr, allSynapses_0.psr,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( decay, allSynapses_0.decay,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( total_delay, allSynapses_0.total_delay,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayQueue, allSynapses_0.delayQueue,
                max_total_synapses * sizeof( uint32_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayIdx, allSynapses_0.delayIdx,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( ldelayQueue, allSynapses_0.ldelayQueue,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( type, allSynapses_0.type,
                max_total_synapses * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tau, allSynapses_0.tau,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( r, allSynapses_0.r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( u, allSynapses_0.u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( D, allSynapses_0.D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( U, allSynapses_0.U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( F, allSynapses_0.F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( lastSpike, allSynapses_0.lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( in_use, allSynapses_0.in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
}
