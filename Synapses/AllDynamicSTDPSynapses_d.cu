#include "hip/hip_runtime.h"
/*
 * AllDynamicSTDPSynapses_d.cu
 *
 */

#include "AllDynamicSTDPSynapses.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDynamicSTDPSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, const SimulationInfo *sim_info ) {
	allocSynapseDeviceStruct( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice     Reference to the allSynapses struct on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) {
	AllDynamicSTDPSynapses allSynapses;

	allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

	HANDLE_ERROR( hipMalloc( allSynapsesDevice, sizeof( AllDynamicSTDPSynapses ) ) );
	HANDLE_ERROR( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllDynamicSTDPSynapses ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::allocDeviceStruct( AllDynamicSTDPSynapses &allSynapses, int num_neurons, int maxSynapsesPerNeuron ) {
        AllSTDPSynapses::allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.lastSpike, max_total_synapses * sizeof( uint64_t ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.r, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.u, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.D, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.U, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.F, max_total_synapses * sizeof( BGFLOAT ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDynamicSTDPSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
	AllDynamicSTDPSynapses allSynapses;

	HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDynamicSTDPSynapses ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allSynapses );

	HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 */
void AllDynamicSTDPSynapses::deleteDeviceStruct( AllDynamicSTDPSynapses& allSynapses ) {
        HANDLE_ERROR( hipFree( allSynapses.lastSpike ) );
	HANDLE_ERROR( hipFree( allSynapses.r ) );
	HANDLE_ERROR( hipFree( allSynapses.u ) );
	HANDLE_ERROR( hipFree( allSynapses.D ) );
	HANDLE_ERROR( hipFree( allSynapses.U ) );
	HANDLE_ERROR( hipFree( allSynapses.F ) );

        AllSTDPSynapses::deleteDeviceStruct( allSynapses );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDynamicSTDPSynapses::copySynapseHostToDevice( void* allSynapsesDevice, const SimulationInfo *sim_info ) { // copy everything necessary
	copySynapseHostToDevice( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary
	AllDynamicSTDPSynapses allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDynamicSTDPSynapses ), hipMemcpyDeviceToHost ) );

	copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::copyHostToDevice( void* allSynapsesDevice, AllDynamicSTDPSynapses& allSynapses, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        AllSTDPSynapses::copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );

        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;
        
        HANDLE_ERROR( hipMemcpy ( allSynapses.lastSpike, lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.r, r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.u, u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.D, D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.U, U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.F, F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDynamicSTDPSynapses::copySynapseDeviceToHost( void* allSynapsesDevice, const SimulationInfo *sim_info ) {
	// copy everything necessary
	AllDynamicSTDPSynapses allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDynamicSTDPSynapses ), hipMemcpyDeviceToHost ) );

	copyDeviceToHost( allSynapses, sim_info );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::copyDeviceToHost( AllDynamicSTDPSynapses& allSynapses, const SimulationInfo *sim_info ) {
        AllSTDPSynapses::copyDeviceToHost( allSynapses, sim_info ) ;

	int num_neurons = sim_info->totalNeurons;
	BGSIZE max_total_synapses = sim_info->maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMemcpy ( lastSpike, allSynapses.lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( r, allSynapses.r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( u, allSynapses.u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( D, allSynapses.D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( U, allSynapses.U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( F, allSynapses.F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

__device__ fpCreateSynapse_t fpCreateDynamicSTDPSynapse_d = (fpCreateSynapse_t)createDynamicSTDPSSynapse;

/*
 *  Get a pointer to the device function createDynamicSTDPSSynapse.
 *  The function will be called from updateSynapsesWeightsDevice device function.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *
 *  @param  fpCreateSynapse_h     Reference to the memory location 
 *                                where the function pointer will be set.
 */
void AllDynamicSTDPSynapses::getFpCreateSynapse(fpCreateSynapse_t& fpCreateSynapse_h)
{
    HANDLE_ERROR( hipMemcpyFromSymbol(&fpCreateSynapse_h, HIP_SYMBOL(fpCreateDynamicSTDPSynapse_d), sizeof(fpCreateSynapse_t)) );
}

__device__ fpChangeSynapsesPSR_t fpChangeDynamicSTDPSynapsesPSR_d = (fpChangeSynapsesPSR_t)changeDynamicSTDPSynapsePSR;

/*
 *  Get a pointer to the device function changeDynamicSTDPSynapsePSR.
 *  The function will be called from advanceSynapsesDevice device function.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *
 *  @param  fpChangePSR_h         Reference to the memory location
 *                                where the function pointer will be set.
 */
void AllDynamicSTDPSynapses::getFpChangePSR(fpChangeSynapsesPSR_t& fpChangePSR_h)
{
    HANDLE_ERROR( hipMemcpyFromSymbol(&fpChangePSR_h, HIP_SYMBOL(fpChangeDynamicSTDPSynapsesPSR_d), sizeof(fpChangeSynapsesPSR_t)) );
}

/* ------------------*\
|* # Global Functions
\* ------------------*/

/* ------------------*\
|* # Device Functions
\* ------------------*/

/*
 *  Create a Synapse and connect it to the model.
 *
 *  @param allSynapsesDevice    Pointer to the Synapse structures in device memory.
 *  @param neuron_index         Index of the source neuron.
 *  @param synapse_index        Index of the Synapse to create.
 *  @param source_x             X location of source.
 *  @param source_y             Y location of source.
 *  @param dest_x               X location of destination.
 *  @param dest_y               Y location of destination.
 *  @param sum_point            Pointer to the summation point.
 *  @param deltaT               The time step size.
 *  @param type                 Type of the Synapse to create.
 */
__device__ void createDynamicSTDPSSynapse(AllDynamicSTDPSynapses* allSynapsesDevice, const int neuron_index, const int synapse_index, int source_index, int dest_index, BGFLOAT *sum_point, const BGFLOAT deltaT, synapseType type)
{
    BGFLOAT delay;
    BGSIZE max_synapses = allSynapsesDevice->maxSynapsesPerNeuron;
    BGSIZE iSyn = max_synapses * neuron_index + synapse_index;

    allSynapsesDevice->in_use[iSyn] = true;
    allSynapsesDevice->summationPoint[iSyn] = sum_point;
    allSynapsesDevice->destNeuronIndex[iSyn] = dest_index;
    allSynapsesDevice->sourceNeuronIndex[iSyn] = source_index;
    allSynapsesDevice->W[iSyn] = synSign(type) * 10.0e-9;

    allSynapsesDevice->delayQueue[iSyn] = 0;
    allSynapsesDevice->delayIdx[iSyn] = 0;
    allSynapsesDevice->ldelayQueue[iSyn] = LENGTH_OF_DELAYQUEUE;

    allSynapsesDevice->psr[iSyn] = 0.0;
    allSynapsesDevice->r[iSyn] = 1.0;
    allSynapsesDevice->u[iSyn] = 0.4;     // DEFAULT_U
    allSynapsesDevice->lastSpike[iSyn] = ULONG_MAX;
    allSynapsesDevice->type[iSyn] = type;

    allSynapsesDevice->U[iSyn] = DEFAULT_U;
    allSynapsesDevice->tau[iSyn] = DEFAULT_tau;

    BGFLOAT U;
    BGFLOAT D;
    BGFLOAT F;
    BGFLOAT tau;
    switch (type) {
        case II:
            U = 0.32;
            D = 0.144;
            F = 0.06;
            tau = 6e-3;
            delay = 0.8e-3;
            break;
        case IE:
            U = 0.25;
            D = 0.7;
            F = 0.02;
            tau = 6e-3;
            delay = 0.8e-3;
            break;
        case EI:
            U = 0.05;
            D = 0.125;
            F = 1.2;
            tau = 3e-3;
            delay = 0.8e-3;
            break;
        case EE:
            U = 0.5;
            D = 1.1;
            F = 0.05;
            tau = 3e-3;
            delay = 1.5e-3;
            break;
        default:
            break;
    }

    allSynapsesDevice->U[iSyn] = U;
    allSynapsesDevice->D[iSyn] = D;
    allSynapsesDevice->F[iSyn] = F;

    allSynapsesDevice->tau[iSyn] = tau;
    allSynapsesDevice->decay[iSyn] = exp( -deltaT / tau );
    allSynapsesDevice->total_delay[iSyn] = static_cast<int>( delay / deltaT ) + 1;

    uint32_t size = allSynapsesDevice->total_delay[iSyn] / ( sizeof(uint8_t) * 8 ) + 1;
    assert( size <= BYTES_OF_DELAYQUEUE );

    allSynapsesDevice->Apos[iSyn] = 0.5;
    allSynapsesDevice->Aneg[iSyn] = -0.5;
    allSynapsesDevice->STDPgap[iSyn] = 2e-3;

    allSynapsesDevice->total_delayPost[iSyn] = 0;

    allSynapsesDevice->tauspost[iSyn] = 0;
    allSynapsesDevice->tauspre[iSyn] = 0;

    allSynapsesDevice->taupos[iSyn] = 15e-3;
    allSynapsesDevice->tauneg[iSyn] = 35e-3;
    allSynapsesDevice->Wex[iSyn] = 1.0;

    allSynapsesDevice->mupos[iSyn] = 0;
    allSynapsesDevice->muneg[iSyn] = 0;

    allSynapsesDevice->useFroemkeDanSTDP[iSyn] = false;
}

/*
 *  Update PSR (post synapse response)
 *
 *  @param  allSynapsesDevice  Reference to the allSynapses struct on device memory.
 *  @param  iSyn               Index of the synapse to set.
 *  @param  simulationStep     The current simulation step.
 *  @param  deltaT             Inner simulation step duration.
 */
__device__ void changeDynamicSTDPSynapsePSR(AllDynamicSTDPSynapses* allSynapsesDevice, const BGSIZE iSyn, const uint64_t simulationStep, const BGFLOAT deltaT)
{
    uint64_t &lastSpike = allSynapsesDevice->lastSpike[iSyn];
    BGFLOAT &r = allSynapsesDevice->r[iSyn];
    BGFLOAT &u = allSynapsesDevice->u[iSyn];
    BGFLOAT D = allSynapsesDevice->D[iSyn];
    BGFLOAT F = allSynapsesDevice->F[iSyn];
    BGFLOAT U = allSynapsesDevice->U[iSyn];
    BGFLOAT W = allSynapsesDevice->W[iSyn];
    BGFLOAT &psr = allSynapsesDevice->psr[iSyn];
    BGFLOAT decay = allSynapsesDevice->decay[iSyn];

    // adjust synapse parameters
    if (lastSpike != ULONG_MAX) {
            BGFLOAT isi = (simulationStep - lastSpike) * deltaT ;
            r = 1 + ( r * ( 1 - u ) - 1 ) * exp( -isi / D );
            u = U + u * ( 1 - U ) * exp( -isi / F );
    }
    psr += ( ( W / decay ) * u * r );// calculate psr
    lastSpike = simulationStep; // record the time of the spike
}
