#include "hip/hip_runtime.h"
/*
 * AllDSSynapses_d.cu
 *
 */

#include "AllDSSynapses.h"
#include "GPUSpikingModel.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllDSSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDSSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, const SimulationInfo *sim_info ) {
	allocSynapseDeviceStruct( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice     Reference to the AllDSSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDSSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) {
	AllDSSynapsesDeviceProperties allSynapses;

	allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

	HANDLE_ERROR( hipMalloc( allSynapsesDevice, sizeof( AllDSSynapsesDeviceProperties ) ) );
	HANDLE_ERROR( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllDSSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice     Reference to the AllDSSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDSSynapses::allocDeviceStruct( AllDSSynapsesDeviceProperties &allSynapses, int num_neurons, int maxSynapsesPerNeuron ) {
        AllSpikingSynapses::allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.lastSpike, max_total_synapses * sizeof( uint64_t ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.r, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.u, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.D, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.U, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.F, max_total_synapses * sizeof( BGFLOAT ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDevice  Reference to the AllDSSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDSSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
	AllDSSynapsesDeviceProperties allSynapses;

	HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDSSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allSynapses );

	HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  Reference to the AllDSSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllDSSynapses::deleteDeviceStruct( AllDSSynapsesDeviceProperties& allSynapses ) {
        HANDLE_ERROR( hipFree( allSynapses.lastSpike ) );
	HANDLE_ERROR( hipFree( allSynapses.r ) );
	HANDLE_ERROR( hipFree( allSynapses.u ) );
	HANDLE_ERROR( hipFree( allSynapses.D ) );
	HANDLE_ERROR( hipFree( allSynapses.U ) );
	HANDLE_ERROR( hipFree( allSynapses.F ) );

        AllSpikingSynapses::deleteDeviceStruct( allSynapses );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice  Reference to the AllDSSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDSSynapses::copySynapseHostToDevice( void* allSynapsesDevice, const SimulationInfo *sim_info ) { // copy everything necessary
	copySynapseHostToDevice( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice     Reference to the AllDSSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDSSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary
	AllDSSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDSSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDevice     Reference to the AllDSSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDSSynapses::copyHostToDevice( void* allSynapsesDevice, AllDSSynapsesDeviceProperties& allSynapses, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        AllSpikingSynapses::copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );

        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMemcpy ( allSynapses.lastSpike, lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.r, r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.u, u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.D, D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.U, U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.F, F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDevice  Reference to the AllDSSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDSSynapses::copySynapseDeviceToHost( void* allSynapsesDevice, const SimulationInfo *sim_info ) {
	// copy everything necessary
	AllDSSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDSSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	copyDeviceToHost( allSynapses, sim_info );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesDevice     Reference to the AllDSSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDSSynapses::copyDeviceToHost( AllDSSynapsesDeviceProperties& allSynapses, const SimulationInfo *sim_info ) {
        AllSpikingSynapses::copyDeviceToHost( allSynapses, sim_info ) ;

	int num_neurons = sim_info->totalNeurons;
	BGSIZE max_total_synapses = sim_info->maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMemcpy ( lastSpike, allSynapses.lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( r, allSynapses.r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( u, allSynapses.u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( D, allSynapses.D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( U, allSynapses.U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( F, allSynapses.F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

/*
 *  Advance all the Synapses in the simulation.
 *  Update the state of all synapses for a time step.
 *
 *  @param  allSynapsesDevice      Reference to the AllSynapsesDeviceProperties struct 
 *                                 on device memory.
 *  @param  allNeuronsDevice       Reference to the allNeurons struct on device memory.
 *  @param  synapseIndexMapDevice  Reference to the SynapseIndexMap on device memory.
 *  @param  sim_info               SimulationInfo class to read information from.
 */
void AllDSSynapses::advanceSynapses(void* allSynapsesDevice, IAllNeurons* allNeuronsDevice, void* synapseIndexMapDevice, const SimulationInfo *sim_info)
{
    if (total_synapse_counts == 0)
        return;

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( total_synapse_counts + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance synapses ------------->
    advanceDSSynapsesDevice <<< blocksPerGrid, threadsPerBlock >>> ( total_synapse_counts, (SynapseIndexMap*)synapseIndexMapDevice, g_simulationStep, sim_info->deltaT, (AllDSSynapsesDeviceProperties*)allSynapsesDevice, (void (*)(AllDSSynapsesDeviceProperties*, const BGSIZE, const uint64_t, const BGFLOAT))m_fpChangePSR_h );
}

__device__ fpCreateSynapse_t fpCreateDSSynapse_d = (fpCreateSynapse_t)createDSSynapse;

/*
 *  Get a pointer to the device function createSynapse.
 *  The function will be called from updateSynapsesWeightsDevice device function.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *
 *  @param  fpCreateSynapse_h     Reference to the memory location 
 *                                where the function pointer will be set.
 */
void AllDSSynapses::getFpCreateSynapse(fpCreateSynapse_t& fpCreateSynapse_h)
{
    HANDLE_ERROR( hipMemcpyFromSymbol(&fpCreateSynapse_h, HIP_SYMBOL(fpCreateDSSynapse_d), sizeof(fpCreateSynapse_t)) );
}

__device__ fpChangeSynapsesPSR_t fpChangeDSSynapsePSR_d = (fpChangeSynapsesPSR_t)changeDSSynapsePSR;

/*
 *  Get a pointer to the device function changeDSSynapsePSR.
 *  The function will be called from advanceSynapsesDevice device function.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *
 *  @param  fpChangePSR_h         Reference to the memory location
 *                                where the function pointer will be set.
 */
void AllDSSynapses::getFpChangePSR(fpChangeSynapsesPSR_t& fpChangePSR_h)
{
    HANDLE_ERROR( hipMemcpyFromSymbol(&fpChangePSR_h, HIP_SYMBOL(fpChangeDSSynapsePSR_d), sizeof(fpChangeSynapsesPSR_t)) );
}

/* ------------------*\
|* # Global Functions
\* ------------------*/

/*
 *  CUDA code for advancing spiking synapses.
 *  Perform updating synapses for one time step.
 *
 *  @param[in] total_synapse_counts  Number of synapses.
 *  @param  synapseIndexMapDevice    Reference to the SynapseIndexMap on device memory.
 *  @param[in] simulationStep        The current simulation step.
 *  @param[in] deltaT                Inner simulation step duration.
 *  @param[in] allSynapsesDevice     Pointer to AllSpikingSynapsesDeviceProperties structures 
 *                                   on device memory.
 *  @param[in] fpChangePSR           Pointer to the device function changePSR() function.
 */
__global__ void advanceDSSynapsesDevice ( int total_synapse_counts, SynapseIndexMap* synapseIndexMapDevice, uint64_t simulationStep, const BGFLOAT deltaT, AllDSSynapsesDeviceProperties* allSynapsesDevice, void (*fpChangePSR)(AllDSSynapsesDeviceProperties*, const BGSIZE, const uint64_t, const BGFLOAT) ) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if ( idx >= total_synapse_counts )
                return;

        BGSIZE iSyn = synapseIndexMapDevice->activeSynapseIndex[idx];

        BGFLOAT &psr = allSynapsesDevice->psr[iSyn];
        BGFLOAT decay = allSynapsesDevice->decay[iSyn];

        // Checks if there is an input spike in the queue.
        bool isFired = isDSSynapsesSpikeQueueDevice(allSynapsesDevice, iSyn);

        // is an input in the queue?
        if (isFired) {
                //fpChangePSR(allSynapsesDevice, iSyn, simulationStep, deltaT);
                changeDSSynapsePSR(allSynapsesDevice, iSyn, simulationStep, deltaT);
        }
        // decay the post spike response
        psr *= decay;
}

/* ------------------*\
|* # Device Functions
\* ------------------*/

/*
 *  Create a DS Synapse and connect it to the model.
 *
 *  @param allSynapsesDevice    Pointer to the AllDSSynapsesDeviceProperties structures 
 *                              on device memory.
 *  @param neuron_index         Index of the source neuron.
 *  @param synapse_index        Index of the Synapse to create.
 *  @param source_x             X location of source.
 *  @param source_y             Y location of source.
 *  @param dest_x               X location of destination.
 *  @param dest_y               Y location of destination.
 *  @param sum_point            Pointer to the summation point.
 *  @param deltaT               The time step size.
 *  @param type                 Type of the Synapse to create.
 */
__device__ void createDSSynapse(AllDSSynapsesDeviceProperties* allSynapsesDevice, const int neuron_index, const int synapse_index, int source_index, int dest_index, BGFLOAT *sum_point, const BGFLOAT deltaT, synapseType type)
{
    BGFLOAT delay;
    BGSIZE max_synapses = allSynapsesDevice->maxSynapsesPerNeuron;
    BGSIZE iSyn = max_synapses * neuron_index + synapse_index;

    allSynapsesDevice->in_use[iSyn] = true;
    allSynapsesDevice->summationPoint[iSyn] = sum_point;
    allSynapsesDevice->destNeuronIndex[iSyn] = dest_index;
    allSynapsesDevice->sourceNeuronIndex[iSyn] = source_index;
    allSynapsesDevice->W[iSyn] = synSign(type) * 10.0e-9;

    allSynapsesDevice->delayQueue[iSyn] = 0;
    allSynapsesDevice->delayIdx[iSyn] = 0;
    allSynapsesDevice->ldelayQueue[iSyn] = LENGTH_OF_DELAYQUEUE;

    allSynapsesDevice->psr[iSyn] = 0.0;
    allSynapsesDevice->r[iSyn] = 1.0;
    allSynapsesDevice->u[iSyn] = 0.4;     // DEFAULT_U
    allSynapsesDevice->lastSpike[iSyn] = ULONG_MAX;
    allSynapsesDevice->type[iSyn] = type;

    allSynapsesDevice->U[iSyn] = DEFAULT_U;
    allSynapsesDevice->tau[iSyn] = DEFAULT_tau;

    BGFLOAT U;
    BGFLOAT D;
    BGFLOAT F;
    BGFLOAT tau;
    switch (type) {
        case II:
            U = 0.32;
            D = 0.144;
            F = 0.06;
            tau = 6e-3;
            delay = 0.8e-3;
            break;
        case IE:
            U = 0.25;
            D = 0.7;
            F = 0.02;
            tau = 6e-3;
            delay = 0.8e-3;
            break;
        case EI:
            U = 0.05;
            D = 0.125;
            F = 1.2;
            tau = 3e-3;
            delay = 0.8e-3;
            break;
        case EE:
            U = 0.5;
            D = 1.1;
            F = 0.05;
            tau = 3e-3;
            delay = 1.5e-3;
            break;
        default:
            break;
    }

    allSynapsesDevice->U[iSyn] = U;
    allSynapsesDevice->D[iSyn] = D;
    allSynapsesDevice->F[iSyn] = F;

    allSynapsesDevice->tau[iSyn] = tau;
    allSynapsesDevice->decay[iSyn] = exp( -deltaT / tau );
    allSynapsesDevice->total_delay[iSyn] = static_cast<int>( delay / deltaT ) + 1;

    uint32_t size = allSynapsesDevice->total_delay[iSyn] / ( sizeof(uint8_t) * 8 ) + 1;
    assert( size <= BYTES_OF_DELAYQUEUE );
}

/*     
 *  Checks if there is an input spike in the queue.
 *
 *  @param[in] allSynapsesDevice     Pointer to AllSpikingSynapsesDeviceProperties structures 
 *                                   on device memory.
 *  @param[in] iSyn                  Index of the Synapse to check.
 *  @return true if there is an input spike event.
 */
__device__ bool isDSSynapsesSpikeQueueDevice(AllDSSynapsesDeviceProperties* allSynapsesDevice, BGSIZE iSyn)
{
    uint32_t &delay_queue = allSynapsesDevice->delayQueue[iSyn];
    int &delayIdx = allSynapsesDevice->delayIdx[iSyn];
    int ldelayQueue = allSynapsesDevice->ldelayQueue[iSyn];

    uint32_t delayMask = (0x1 << delayIdx);
    bool isFired = delay_queue & (delayMask);
    delay_queue &= ~(delayMask);
    if ( ++delayIdx >= ldelayQueue ) {
            delayIdx = 0;
    }

    return isFired;
}

/*
 *  Update PSR (post synapse response)
 *
 *  @param  allSynapsesDevice  Reference to the AllDSSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  iSyn               Index of the synapse to set.
 *  @param  simulationStep     The current simulation step.
 *  @param  deltaT             Inner simulation step duration.
 */
__device__ void changeDSSynapsePSR(AllDSSynapsesDeviceProperties* allSynapsesDevice, const BGSIZE iSyn, const uint64_t simulationStep, const BGFLOAT deltaT)
{
    //assert( iSyn < allSynapsesDevice->maxSynapsesPerNeuron * allSynapsesDevice->count_neurons );

    uint64_t &lastSpike = allSynapsesDevice->lastSpike[iSyn];
    BGFLOAT &r = allSynapsesDevice->r[iSyn];
    BGFLOAT &u = allSynapsesDevice->u[iSyn];
    BGFLOAT D = allSynapsesDevice->D[iSyn];
    BGFLOAT F = allSynapsesDevice->F[iSyn];
    BGFLOAT U = allSynapsesDevice->U[iSyn];
    BGFLOAT W = allSynapsesDevice->W[iSyn];
    BGFLOAT &psr = allSynapsesDevice->psr[iSyn];
    BGFLOAT decay = allSynapsesDevice->decay[iSyn];

    // adjust synapse parameters
    if (lastSpike != ULONG_MAX) {
            BGFLOAT isi = (simulationStep - lastSpike) * deltaT ;
            r = 1 + ( r * ( 1 - u ) - 1 ) * exp( -isi / D );
            u = U + u * ( 1 - U ) * exp( -isi / F );
    }
    psr += ( ( W / decay ) * u * r );// calculate psr
    lastSpike = simulationStep; // record the time of the spike
}
