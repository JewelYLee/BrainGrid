#include "hip/hip_runtime.h"
/*
 * AllSpikingSynapses.cu
 *
 */

#include "AllSpikingSynapses.h"
#include "AllSynapsesPolyFuncs.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, const SimulationInfo *sim_info ) {
        allocSynapseDeviceStruct( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) {
        AllSpikingSynapsesDeviceProperties allSynapses;

        allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

        HANDLE_ERROR( hipMalloc( allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ) ) );
        HANDLE_ERROR( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::allocDeviceStruct( AllSpikingSynapsesDeviceProperties &allSynapses, int num_neurons, int maxSynapsesPerNeuron ) {
        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.sourceNeuronIndex, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.destNeuronIndex, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.W, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.summationPoint, max_total_synapses * sizeof( BGFLOAT* ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.type, max_total_synapses * sizeof( synapseType ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.psr, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.in_use, max_total_synapses * sizeof( bool ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.synapse_counts, num_neurons * sizeof( BGSIZE ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.decay, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.tau, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.total_delay, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.delayQueue, max_total_synapses * sizeof( uint32_t ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.delayIdx, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.ldelayQueue, max_total_synapses * sizeof( int ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
        AllSpikingSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        deleteDeviceStruct( allSynapses );

        HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::deleteDeviceStruct( AllSpikingSynapsesDeviceProperties& allSynapses ) {
        HANDLE_ERROR( hipFree( allSynapses.sourceNeuronIndex ) );
        HANDLE_ERROR( hipFree( allSynapses.destNeuronIndex ) );
        HANDLE_ERROR( hipFree( allSynapses.W ) );
        HANDLE_ERROR( hipFree( allSynapses.summationPoint ) );
        HANDLE_ERROR( hipFree( allSynapses.type ) );
        HANDLE_ERROR( hipFree( allSynapses.psr ) );
        HANDLE_ERROR( hipFree( allSynapses.in_use ) );
        HANDLE_ERROR( hipFree( allSynapses.synapse_counts ) );
        HANDLE_ERROR( hipFree( allSynapses.decay ) );
        HANDLE_ERROR( hipFree( allSynapses.tau ) );
        HANDLE_ERROR( hipFree( allSynapses.total_delay ) );
        HANDLE_ERROR( hipFree( allSynapses.delayQueue ) );
        HANDLE_ERROR( hipFree( allSynapses.delayIdx ) );
        HANDLE_ERROR( hipFree( allSynapses.ldelayQueue ) );

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.count_neurons = 0;
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::copySynapseHostToDevice( void* allSynapsesDevice, const SimulationInfo *sim_info ) { // copy everything necessary
        copySynapseHostToDevice( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary
        AllSpikingSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copyHostToDevice( void* allSynapsesDevice, AllSpikingSynapsesDeviceProperties& allSynapses, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        allSynapses.maxSynapsesPerNeuron = maxSynapsesPerNeuron;
        allSynapses.total_synapse_counts = total_synapse_counts;
        allSynapses.count_neurons = count_neurons;
        HANDLE_ERROR( hipMemcpy ( allSynapsesDevice, &allSynapses, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        allSynapses.count_neurons = 0;

        HANDLE_ERROR( hipMemcpy ( allSynapses.sourceNeuronIndex, sourceNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.destNeuronIndex, destNeuronIndex,
                max_total_synapses * sizeof( int ),  hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.W, W,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.type, type,
                max_total_synapses * sizeof( synapseType ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.psr, psr,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.in_use, in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.synapse_counts, synapse_counts,
                        num_neurons * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.decay, decay,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.tau, tau,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.total_delay, total_delay,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.delayQueue, delayQueue,
                max_total_synapses * sizeof( uint32_t ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.delayIdx, delayIdx,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.ldelayQueue, ldelayQueue,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::copySynapseDeviceToHost( void* allSynapsesDevice, const SimulationInfo *sim_info ) {
        // copy everything necessary
        AllSpikingSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        copyDeviceToHost( allSynapses, sim_info );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copyDeviceToHost( AllSpikingSynapsesDeviceProperties& allSynapses, const SimulationInfo *sim_info ) {
        int num_neurons = sim_info->totalNeurons;
        BGSIZE max_total_synapses = sim_info->maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMemcpy ( synapse_counts, allSynapses.synapse_counts,
                num_neurons * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );
        maxSynapsesPerNeuron = allSynapses.maxSynapsesPerNeuron;
        total_synapse_counts = allSynapses.total_synapse_counts;
        count_neurons = allSynapses.count_neurons;

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        allSynapses.count_neurons = 0;

        HANDLE_ERROR( hipMemcpy ( sourceNeuronIndex, allSynapses.sourceNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( destNeuronIndex, allSynapses.destNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( W, allSynapses.W,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( type, allSynapses.type,
                max_total_synapses * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( psr, allSynapses.psr,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( in_use, allSynapses.in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( decay, allSynapses.decay,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tau, allSynapses.tau,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( total_delay, allSynapses.total_delay,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayQueue, allSynapses.delayQueue,
                max_total_synapses * sizeof( uint32_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayIdx, allSynapses.delayIdx,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( ldelayQueue, allSynapses.ldelayQueue,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
}

/*
 *  Get synapse_counts in AllSynapses struct on device memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::copyDeviceSynapseCountsToHost(void* allSynapsesDevice, const SimulationInfo *sim_info)
{
        AllSpikingSynapsesDeviceProperties allSynapses;
        int neuron_count = sim_info->totalNeurons;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( synapse_counts, allSynapses.synapse_counts, neuron_count * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.count_neurons = 0;
}

/* 
 *  Get summationCoord and in_use in AllSynapses struct on device memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::copyDeviceSynapseSumIdxToHost(void* allSynapsesDevice, const SimulationInfo *sim_info)
{
        AllSpikingSynapsesDeviceProperties allSynapses;
        BGSIZE max_total_synapses = sim_info->maxSynapsesPerNeuron * sim_info->totalNeurons;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( destNeuronIndex, allSynapses.destNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( in_use, allSynapses.in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
       
        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.count_neurons = 0;
}

__device__ fpCreateSynapse_t fpCreateSpikingSynapse_d = (fpCreateSynapse_t)createSpikingSynapse;

/*
 *  Get a pointer to the device function createSynapse.
 *  The function will be called from updateSynapsesWeightsDevice device function.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *
 *  @param  fpCreateSynapse_h     Reference to the memory location
 *                                where the function pointer will be set.
 */
void AllSpikingSynapses::getFpCreateSynapse(fpCreateSynapse_t& fpCreateSynapse_h)
{
    HANDLE_ERROR( hipMemcpyFromSymbol(&fpCreateSynapse_h, HIP_SYMBOL(fpCreateSpikingSynapse_d), sizeof(fpCreateSynapse_t)) );
}

/*
 *  Set some parameters used for advanceSynapsesDevice.
 */
void AllSpikingSynapses::setAdvanceSynapsesDeviceParams()
{
    setSynapseClassID();
}

/**
 *  Set synapse class ID defined by enumClassSynapses for the caller's Synapse class.
 *  The class ID will be set to classSynapses_d in device memory,
 *  and the classSynapses_d will be referred to call a device function for the
 *  particular synapse class.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *  Note: we used to use a function pointer; however, it caused the growth_cuda crash
 *  (see issue#137).
 */
void AllSpikingSynapses::setSynapseClassID()
{
    enumClassSynapses classSynapses_h = classAllSpikingSynapses;

    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(classSynapses_d), &classSynapses_h, sizeof(enumClassSynapses)) );
}

/*
 *  Advance all the Synapses in the simulation.
 *  Update the state of all synapses for a time step.
 *
 *  @param  allSynapsesDevice      Reference to the AllSynapsesDeviceProperties struct 
 *                                 on device memory.
 *  @param  allNeuronsDevice       Reference to the allNeurons struct on device memory.
 *  @param  synapseIndexMapDevice  Reference to the SynapseIndexMap on device memory.
 *  @param  sim_info               SimulationInfo class to read information from.
 */
void AllSpikingSynapses::advanceSynapses(void* allSynapsesDevice, void* allNeuronsDevice, void* synapseIndexMapDevice, const SimulationInfo *sim_info)
{
    if (total_synapse_counts == 0)
        return;

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( total_synapse_counts + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance synapses ------------->
    advanceSpikingSynapsesDevice <<< blocksPerGrid, threadsPerBlock >>> ( total_synapse_counts, (SynapseIndexMap*)synapseIndexMapDevice, g_simulationStep, sim_info->deltaT, (AllSpikingSynapsesDeviceProperties*)allSynapsesDevice );
}

__device__ fpPreSynapsesSpikeHit_t fpPreSpikingSynapsesSpikeHit_d = (fpPreSynapsesSpikeHit_t)preSpikingSynapsesSpikeHitDevice;

/*
 *  Get a pointer to the device function preSpikeHit.
 *  The function will be called from advanceNeuronsDevice device function.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *
 *  @param  fpPreSpikeHit_h       Reference to the memory location
 *                                where the function pointer will be set.
 */
void AllSpikingSynapses::getFpPreSpikeHit(fpPreSynapsesSpikeHit_t& fpPreSpikeHit_h)
{
    HANDLE_ERROR( hipMemcpyFromSymbol(&fpPreSpikeHit_h, HIP_SYMBOL(fpPreSpikingSynapsesSpikeHit_d), sizeof(fpPreSynapsesSpikeHit_t)) );
}

__device__ fpPostSynapsesSpikeHit_t fpPostSpikingSynapsesSpikeHit_d = (fpPostSynapsesSpikeHit_t)postSpikingSynapsesSpikeHitDevice;

/*
 *  Get a pointer to the device function ostSpikeHit.
 *  The function will be called from advanceNeuronsDevice device function.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *
 *  @param  fpostSpikeHit_h       Reference to the memory location
 *                                where the function pointer will be set.
 */
void AllSpikingSynapses::getFpPostSpikeHit(fpPostSynapsesSpikeHit_t& fpPostSpikeHit_h)
{
    HANDLE_ERROR( hipMemcpyFromSymbol(&fpPostSpikeHit_h, HIP_SYMBOL(fpPostSpikingSynapsesSpikeHit_d), sizeof(fpPostSynapsesSpikeHit_t)) );
}

/* ------------------*\
|* # Global Functions
\* ------------------*/

/* ------------------*\
|* # Device Functions
\* ------------------*/

/*
 *  Create a Spiking Synapse and connect it to the model.
 *
 *  @param allSynapsesDevice    Pointer to the AllSpikingSynapsesDeviceProperties structures 
 *                              on device memory.
 *  @param neuron_index         Index of the source neuron.
 *  @param synapse_index        Index of the Synapse to create.
 *  @param source_x             X location of source.
 *  @param source_y             Y location of source.
 *  @param dest_x               X location of destination.
 *  @param dest_y               Y location of destination.
 *  @param sum_point            Pointer to the summation point.
 *  @param deltaT               The time step size.
 *  @param type                 Type of the Synapse to create.
 */
__device__ void createSpikingSynapse(AllSpikingSynapsesDeviceProperties* allSynapsesDevice, const int neuron_index, const int synapse_index, int source_index, int dest_index, BGFLOAT *sum_point, const BGFLOAT deltaT, synapseType type)
{
    BGFLOAT delay;
    BGSIZE max_synapses = allSynapsesDevice->maxSynapsesPerNeuron;
    BGSIZE iSyn = max_synapses * neuron_index + synapse_index;

    allSynapsesDevice->in_use[iSyn] = true;
    allSynapsesDevice->summationPoint[iSyn] = sum_point;
    allSynapsesDevice->destNeuronIndex[iSyn] = dest_index;
    allSynapsesDevice->sourceNeuronIndex[iSyn] = source_index;
    allSynapsesDevice->W[iSyn] = synSign(type) * 10.0e-9;

    allSynapsesDevice->delayQueue[iSyn] = 0;
    allSynapsesDevice->delayIdx[iSyn] = 0;
    allSynapsesDevice->ldelayQueue[iSyn] = LENGTH_OF_DELAYQUEUE;

    allSynapsesDevice->psr[iSyn] = 0.0;
    allSynapsesDevice->type[iSyn] = type;

    allSynapsesDevice->tau[iSyn] = DEFAULT_tau;

    BGFLOAT tau;
    switch (type) {
        case II:
            tau = 6e-3;
            delay = 0.8e-3;
            break;
        case IE:
            tau = 6e-3;
            delay = 0.8e-3;
            break;
        case EI:
            tau = 3e-3;
            delay = 0.8e-3;
            break;
        case EE:
            tau = 3e-3;
            delay = 1.5e-3;
            break;
        default:
            break;
    }

    allSynapsesDevice->tau[iSyn] = tau;
    allSynapsesDevice->decay[iSyn] = exp( -deltaT / tau );
    allSynapsesDevice->total_delay[iSyn] = static_cast<int>( delay / deltaT ) + 1;

    uint32_t size = allSynapsesDevice->total_delay[iSyn] / ( sizeof(uint8_t) * 8 ) + 1;
    assert( size <= BYTES_OF_DELAYQUEUE );
}

/*
 *  Prepares Synapse for a spike hit.
 *
 *  @param[in] iSyn                  Index of the Synapse to update.
 *  @param[in] allSynapsesDevice     Pointer to AllSpikingSynapsesDeviceProperties structures 
 *                                   on device memory.
 */
__device__ void preSpikingSynapsesSpikeHitDevice( const BGSIZE iSyn, AllSpikingSynapsesDeviceProperties* allSynapsesDevice ) {
        uint32_t &delay_queue = allSynapsesDevice->delayQueue[iSyn];
        int delayIdx = allSynapsesDevice->delayIdx[iSyn];
        int ldelayQueue = allSynapsesDevice->ldelayQueue[iSyn];
        int total_delay = allSynapsesDevice->total_delay[iSyn];

        // Add to spike queue

        // calculate index where to insert the spike into delayQueue
        int idx = delayIdx +  total_delay;
        if ( idx >= ldelayQueue ) {
                idx -= ldelayQueue;
        }

        // set a spike
        //assert( !(delay_queue[0] & (0x1 << idx)) );
        delay_queue |= (0x1 << idx);
}

/*
 *  Prepares Synapse for a spike hit (for back propagation).
 *
 *  @param[in] iSyn                  Index of the Synapse to update.
 *  @param[in] allSynapsesDevice     Pointer to AllSpikingSynapsesDeviceProperties structures 
 *                                   on device memory.
 */
__device__ void postSpikingSynapsesSpikeHitDevice( const BGSIZE iSyn, AllSpikingSynapsesDeviceProperties* allSynapsesDevice ) {
}

/*
 * Adds a synapse to the network.  Requires the locations of the source and
 * destination neurons.
 *
 * @param allSynapsesDevice      Pointer to the AllSpikingSynapsesDeviceProperties structures 
 *                               on device memory.
 * @param type                   Type of the Synapse to create.
 * @param src_neuron             Index of the source neuron.
 * @param dest_neuron            Index of the destination neuron.
 * @param source_x               X location of source.
 * @param source_y               Y location of source.
 * @param dest_x                 X location of destination.
 * @param dest_y                 Y location of destination.
 * @param sum_point              Pointer to the summation point.
 * @param deltaT                 The time step size.
 * @param W_d                    Array of synapse weight.
 * @param num_neurons            The number of neurons.
 */
__device__ void addSpikingSynapse(AllSpikingSynapsesDeviceProperties* allSynapsesDevice, synapseType type, const int src_neuron, const int dest_neuron, int source_index, int dest_index, BGFLOAT *sum_point, const BGFLOAT deltaT, BGFLOAT* W_d, int num_neurons, void (*fpCreateSynapse)(AllSpikingSynapsesDeviceProperties*, const int, const int, int, int, BGFLOAT*, const BGFLOAT, synapseType))
{
    if (allSynapsesDevice->synapse_counts[src_neuron] >= allSynapsesDevice->maxSynapsesPerNeuron) {
        return; // TODO: ERROR!
    }

    // add it to the list
    BGSIZE synapse_index;
    BGSIZE max_synapses = allSynapsesDevice->maxSynapsesPerNeuron;
    BGSIZE iSync = max_synapses * src_neuron;
    for (synapse_index = 0; synapse_index < max_synapses; synapse_index++) {
        if (!allSynapsesDevice->in_use[iSync + synapse_index]) {
            break;
        }
    }

    allSynapsesDevice->synapse_counts[src_neuron]++;

    // create a synapse
    fpCreateSynapse(allSynapsesDevice, src_neuron, synapse_index, source_index, dest_index, sum_point, deltaT, type );
    allSynapsesDevice->W[iSync + synapse_index] = W_d[src_neuron * num_neurons + dest_neuron] * synSign(type) * AllSynapses::SYNAPSE_STRENGTH_ADJUSTMENT;
}

/*
 * Remove a synapse from the network.
 *
 * @param[in] allSynapsesDevice      Pointer to the AllSpikingSynapsesDeviceProperties structures 
 *                                   on device memory.
 * @param neuron_index               Index of a neuron.
 * @param synapse_index              Index of a synapse.
 * @param[in] maxSynapses            Maximum number of synapses per neuron.
 */
__device__ void eraseSpikingSynapse( AllSpikingSynapsesDeviceProperties* allSynapsesDevice, const int neuron_index, const int synapse_index, int maxSynapses )
{
    BGSIZE iSync = maxSynapses * neuron_index + synapse_index;
    allSynapsesDevice->synapse_counts[neuron_index]--;
    allSynapsesDevice->in_use[iSync] = false;
    allSynapsesDevice->summationPoint[iSync] = NULL;
}

/*
 * Returns the type of synapse at the given coordinates
 *
 * @param[in] allNeuronsDevice          Pointer to the Neuron structures in device memory.
 * @param src_neuron             Index of the source neuron.
 * @param dest_neuron            Index of the destination neuron.
 */
__device__ synapseType synType( neuronType* neuron_type_map_d, const int src_neuron, const int dest_neuron )
{
    if ( neuron_type_map_d[src_neuron] == INH && neuron_type_map_d[dest_neuron] == INH )
        return II;
    else if ( neuron_type_map_d[src_neuron] == INH && neuron_type_map_d[dest_neuron] == EXC )
        return IE;
    else if ( neuron_type_map_d[src_neuron] == EXC && neuron_type_map_d[dest_neuron] == INH )
        return EI;
    else if ( neuron_type_map_d[src_neuron] == EXC && neuron_type_map_d[dest_neuron] == EXC )
        return EE;

    return STYPE_UNDEF;

}

/*
 * Return 1 if originating neuron is excitatory, -1 otherwise.
 *
 * @param[in] t  synapseType I to I, I to E, E to I, or E to E
 * @return 1 or -1
 */
__device__ int synSign( synapseType t )
{
        switch ( t )
        {
        case II:
        case IE:
                return -1;
        case EI:
        case EE:
                return 1;
        }

        return 0;
}
