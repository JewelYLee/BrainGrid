#include "hip/hip_runtime.h"
#include "AllLIFNeurons.h"
#include "Book.h"

/*
 *  Update the state of all neurons for a time step
 *  Notify outgoing synapses if neuron has fired.
 *
 *  @param  synapses               Reference to the allSynapses struct on host memory.
 *  @param  allNeuronsDevice       Reference to the allNeurons struct on device memory.
 *  @param  allSynapsesDevice      Reference to the allSynapses struct on device memory.
 *  @param  sim_info               SimulationInfo to refer from.
 *  @param  randNoise              Reference to the random noise array.
 *  @param  synapseIndexMapDevice  Reference to the SynapseIndexMap on device memory.
 */
void AllLIFNeurons::advanceNeurons( IAllSynapses &synapses, IAllNeurons* allNeuronsDevice, void* allSynapsesDevice, const SimulationInfo *sim_info, float* randNoise, SynapseIndexMap* synapseIndexMapDevice )
{
    int neuron_count = sim_info->totalNeurons;
    int maxSpikes = (int)((sim_info->epochDuration * sim_info->maxFiringRate));

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( neuron_count + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance neurons ------------->
    advanceLIFNeuronsDevice <<< blocksPerGrid, threadsPerBlock >>> ( neuron_count, sim_info->maxSynapsesPerNeuron, maxSpikes, sim_info->deltaT, g_simulationStep, randNoise, (AllIFNeurons *)allNeuronsDevice, (AllSpikingSynapsesDeviceProperties*)allSynapsesDevice, synapseIndexMapDevice, (void (*)(const BGSIZE, AllSpikingSynapsesDeviceProperties*))m_fpPreSpikeHit_h, (void (*)(const BGSIZE, AllSpikingSynapsesDeviceProperties*))m_fpPostSpikeHit_h, m_fAllowBackPropagation );
}

/* ------------------*\
|* # Global Functions
\* ------------------*/

/*
 *  CUDA code for advancing LIF neurons
 * 
 *  @param[in] totalNeurons          Number of neurons.
 *  @param[in] maxSynapses           Maximum number of synapses per neuron.
 *  @param[in] maxSpikes             Maximum number of spikes per neuron per epoch.
 *  @param[in] deltaT                Inner simulation step duration.
 *  @param[in] simulationStep        The current simulation step.
 *  @param[in] randNoise             Pointer to device random noise array.
 *  @param[in] allNeuronsDevice      Pointer to Neuron structures in device memory.
 *  @param[in] allSynapsesDevice     Pointer to Synapse structures in device memory.
 *  @param[in] synapseIndexMap       Inverse map, which is a table indexed by an input neuron and maps to the synapses that provide input to that neuron.
 *  @param[in] fpPreSpikeHit         Pointer to the device function preSpikeHit() function.
 *  @param[in] fpPostSpikeHit        Pointer to the device function postSpikeHit() function.
 *  @param[in] fAllowBackPropagation True if back propagaion is allowed.
 */
__global__ void advanceLIFNeuronsDevice( int totalNeurons, int maxSynapses, int maxSpikes, const BGFLOAT deltaT, uint64_t simulationStep, float* randNoise, AllIFNeurons* allNeuronsDevice, AllSpikingSynapsesDeviceProperties* allSynapsesDevice, SynapseIndexMap* synapseIndexMapDevice, void (*fpPreSpikeHit)(const BGSIZE, AllSpikingSynapsesDeviceProperties*), void (*fpPostSpikeHit)(const BGSIZE, AllSpikingSynapsesDeviceProperties*), bool fAllowBackPropagation ) {
        // determine which neuron this thread is processing
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if ( idx >= totalNeurons )
                return;

        allNeuronsDevice->hasFired[idx] = false;
        BGFLOAT& sp = allNeuronsDevice->summation_map[idx];
        BGFLOAT& vm = allNeuronsDevice->Vm[idx];
        BGFLOAT r_sp = sp;
        BGFLOAT r_vm = vm;

        if ( allNeuronsDevice->nStepsInRefr[idx] > 0 ) { // is neuron refractory?
                --allNeuronsDevice->nStepsInRefr[idx];
        } else if ( r_vm >= allNeuronsDevice->Vthresh[idx] ) { // should it fire?
                int& spikeCount = allNeuronsDevice->spikeCount[idx];
                int& spikeCountOffset = allNeuronsDevice->spikeCountOffset[idx];

                // Note that the neuron has fired!
                allNeuronsDevice->hasFired[idx] = true;

                // record spike time
                int idxSp = (spikeCount + spikeCountOffset) % maxSpikes;
                allNeuronsDevice->spike_history[idx][idxSp] = simulationStep;
                spikeCount++;

                DEBUG_SYNAPSE(
                    printf("advanceLIFNeuronsDevice\n");
                    printf("          index: %d\n", idx);
                    printf("          simulationStep: %d\n\n", simulationStep);
                );

                // calculate the number of steps in the absolute refractory period
                allNeuronsDevice->nStepsInRefr[idx] = static_cast<int> ( allNeuronsDevice->Trefract[idx] / deltaT + 0.5 );

                // reset to 'Vreset'
                vm = allNeuronsDevice->Vreset[idx];

                // notify outgoing synapses of spike
                BGSIZE synapse_counts = allSynapsesDevice->synapse_counts[idx];
                int synapse_notified = 0;
                for (BGSIZE i = 0; synapse_notified < synapse_counts; i++) {
                        BGSIZE iSyn = maxSynapses * idx + i;
                        if (allSynapsesDevice->in_use[iSyn] == true) {
                                fpPreSpikeHit(iSyn, allSynapsesDevice); 
                                synapse_notified++;
                        }
                }

                // notify incomming synapses of spike
                synapse_counts = synapseIndexMapDevice->synapseCount[idx];
                if (fAllowBackPropagation && synapse_counts != 0) {
                        BGSIZE beginIndex = synapseIndexMapDevice->incomingSynapse_begin[idx];
                        BGSIZE* inverseMap_begin = &( synapseIndexMapDevice->inverseIndex[beginIndex] );
                        BGSIZE iSyn = inverseMap_begin[0];
                        for ( BGSIZE i = 0; i < synapse_counts; i++ ) {
                                iSyn = inverseMap_begin[i];
                                fpPostSpikeHit(iSyn, allSynapsesDevice);
                                synapse_notified++;
                        }
                }
        } else {
                r_sp += allNeuronsDevice->I0[idx]; // add IO

                // Random number alg. goes here
                r_sp += (randNoise[idx] * allNeuronsDevice->Inoise[idx]); // add cheap noise
                vm = allNeuronsDevice->C1[idx] * r_vm + allNeuronsDevice->C2[idx] * ( r_sp ); // decay Vm and add inputs
        }

        // clear synaptic input for next time step
        sp = 0;
}
